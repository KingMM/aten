#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

// NOTE
// ddx, ddy
// http://mosapui.blog116.fc2.com/blog-entry-35.html
// https://www.gamedev.net/forums/topic/478820-derivative-instruction-details-ddx-ddy-or-dfdx-dfdy-etc/
// http://d.hatena.ne.jp/umonist/20110616/p1
// http://monsho.blog63.fc2.com/blog-entry-105.html

inline __device__ float ddx(
	int x, int y,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov)
{
	// NOTE
	// 2x2 pixel���ƂɌv�Z����.

	int leftX = x; 
	int rightX = x + 1;

#if 0
	if ((x & 0x01) == 1) {
		leftX = x - 1;
		rightX = x;
	}
#else
	int offset = (x & 0x01);
	leftX -= offset;
	rightX -= offset;
#endif

	rightX = min(rightX, w - 1);

	const int idxL = getIdx(leftX, y, w);
	const int idxR = getIdx(rightX, y, w);

#if 0
	float left = aov[idxL].depth;
	float right = aov[idxR].depth;
#else
	auto l_v0 = ((float4*)aov)[idxL * idaten::SVGFPathTracing::AOV_float4_size + 0];
	auto r_v0 = ((float4*)aov)[idxR * idaten::SVGFPathTracing::AOV_float4_size + 0];

	float left = l_v0.w;
	float right = r_v0.w;
#endif

	return right - left;
}

inline __device__ float ddy(
	int x, int y,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov)
{
	// NOTE
	// 2x2 pixel���ƂɌv�Z����.

	int topY = y;
	int bottomY = y + 1;

#if 0
	if ((y & 0x01) == 1) {
		topY = y - 1;
		bottomY = y;
	}
#else
	int offset = (y & 0x01);
	topY -= offset;
	bottomY -= offset;
#endif

	bottomY = min(bottomY, h - 1);

	int idxT = getIdx(x, topY, w);
	int idxB = getIdx(x, bottomY, w);

#if 0
	float top = aov[idxT].depth;
	float bottom = aov[idxB].depth;
#else
	auto t_v0 = ((float4*)aov)[idxT * idaten::SVGFPathTracing::AOV_float4_size + 0];
	auto b_v0 = ((float4*)aov)[idxB * idaten::SVGFPathTracing::AOV_float4_size + 0];

	float top = t_v0.w;
	float bottom = b_v0.w;
#endif

	return bottom - top;
}

inline __device__ float gaussFilter3x3(
	int ix, int iy,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov)
{
	static const float kernel[] = {
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
		1.0 / 8.0,  1.0 / 4.0, 1.0 / 8.0,
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
	};

	static const int offsetx[] = {
		-1, 0, 1,
		-1, 0, 1,
		-1, 0, 1,
	};

	static const int offsety[] = {
		-1, -1, -1,
		0, 0, 0,
		1, 1, 1,
	};

	float sum = 0;

	int pos = 0;

#pragma unroll
	for (int i = 0; i < 9; i++) {
		int xx = clamp(ix + offsetx[i], 0, w - 1);
		int yy = clamp(iy + offsety[i], 0, h - 1);

		int idx = getIdx(xx, yy, w);

#if 0
		float tmp = aov[idx].var;
#else
		auto v = aov[idx].v2;
		float tmp = v.w;
#endif

		sum += kernel[pos] * tmp;

		pos++;
	}

	return sum;
}

inline __device__ float gaussFilter3x3(
	int ix, int iy,
	int w, int h,
	const float* __restrict__ var)
{
	static const float kernel[] = {
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
		1.0 / 8.0,  1.0 / 4.0, 1.0 / 8.0,
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
	};

	static const int offsetx[] = {
		-1, 0, 1,
		-1, 0, 1,
		-1, 0, 1,
	};

	static const int offsety[] = {
		-1, -1, -1,
		0, 0, 0,
		1, 1, 1,
	};

	float sum = 0;

	int pos = 0;

#pragma unroll
	for (int i = 0; i < 9; i++) {
		int xx = clamp(ix + offsetx[i], 0, w - 1);
		int yy = clamp(iy + offsety[i], 0, h - 1);

		int idx = getIdx(xx, yy, w);

		float tmp = var[idx];

		sum += kernel[pos] * tmp;

		pos++;
	}

	return sum;
}

template <bool isFirstIter, bool isFinalIter>
__global__ void atrousFilter(
	hipSurfaceObject_t dst,
	float4* tmpBuffer,
	idaten::SVGFPathTracing::AOV* aovs,
	const float4* __restrict__ clrBuffer,
	float4* nextClrBuffer,
	const float* __restrict__ varBuffer,
	float* nextVarBuffer,
	int stepScale,
	float thresholdTemporalWeight,
	int radiusScale,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	auto centerNormal = aovs[idx].normal;

	float centerDepth = aovs[idx].depth;
	int centerMeshId = aovs[idx].meshid;

	float tmpDdzX = ddx(ix, iy, width, height, aovs);
	float tmpDdzY = ddy(ix, iy, width, height, aovs);
	float2 ddZ = make_float2(tmpDdzX, tmpDdzY);

	float4 centerColor;

	if (isFirstIter) {
#if 0
		centerColor = make_float4(aovs[idx].color, 1);
#else
		auto v2 = ((float4*)aovs)[idx * idaten::SVGFPathTracing::AOV_float4_size + 2];
		centerColor = v2;
#endif
	}
	else {
		centerColor = clrBuffer[idx];
	}

	auto v1 = ((float4*)aovs)[idx * idaten::SVGFPathTracing::AOV_float4_size + 1];

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���̂܂܏o�͂��ďI��.
		nextClrBuffer[idx] = centerColor;

		if (isFinalIter) {
#if 0
			centerColor *= make_float4(aovs[idx].texclr, 1);
#else
			centerColor *= v1;
#endif

			surf2Dwrite(
				centerColor,
				dst,
				ix * sizeof(float4), iy,
				hipBoundaryModeTrap);
		}

		return;
	}

	float centerLum = AT_NAME::color::luminance(centerColor.x, centerColor.y, centerColor.z);

	// �K�E�X�t�B���^3x3
	float gaussedVarLum;
	
	if (isFirstIter) {
		gaussedVarLum = gaussFilter3x3(ix, iy, width, height, aovs);
	}
	else {
		gaussedVarLum = gaussFilter3x3(ix, iy, width, height, varBuffer);
	}

	float sqrGaussedVarLum = sqrt(gaussedVarLum);

	static const float sigmaZ = 1.0f;
	static const float sigmaN = 128.0f;
	static const float sigmaL = 4.0f;

	float2 p = make_float2(ix, iy);

	// NOTE
	// 5x5

	float4 sumC = make_float4(0, 0, 0, 0);
	float weightC = 0;

	float sumV = 0;
	float weightV = 0;

	int pos = 0;

	static const float h[] = {
		1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
		1.0 / 64.0,  1.0 / 16.0, 3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
		3.0 / 128.0, 3.0 / 32.0, 9.0 / 64.0,  3.0 / 32.0, 3.0 / 128.0,
		1.0 / 64.0,  1.0 / 16.0, 3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
		1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
	};

	int R = 2;

#if 0
	if (isFirstIter) {
		if (aovs[idx].temporalWeight < thresholdTemporalWeight) {
			R *= radiusScale;
		}
	}

	for (int y = -R; y <= R; y++) {
		for (int x = -R; x <= R; x++) {
			int xx = clamp(ix + x * stepScale, 0, width - 1);
			int yy = clamp(iy + y * stepScale, 0, height - 1);
#else
	static const int offsetx[] = {
		-2, -1, 0, 1, 2,
		-2, -1, 0, 1, 2,
		-2, -1, 0, 1, 2,
		-2, -1, 0, 1, 2,
		-2, -1, 0, 1, 2,
	};
	static const int offsety[] = {
		-2, -2, -2, -2, -2,
		-1, -1, -1, -1, -1,
		 0,  0,  0,  0,  0,
		 1,  1,  1,  1,  1,
		 2,  2,  2,  2,  2,
	};

#pragma unroll
	for (int i = 0; i < 25; i++) {
	{
			int xx = clamp(ix + offsetx[i] * stepScale, 0, width - 1);
			int yy = clamp(iy + offsety[i] * stepScale, 0, height - 1);
#endif

			float2 q = make_float2(xx, yy);

			const int qidx = getIdx(xx, yy, width);

#if 0
			float3 normal = aovs[qidx].normal;
			float depth = aovs[qidx].depth;
			int meshid = aovs[qidx].meshid;
#else
			auto v0 = ((float4*)aovs)[qidx * idaten::SVGFPathTracing::AOV_float4_size + 0];
			auto v3 = ((float4*)aovs)[qidx * idaten::SVGFPathTracing::AOV_float4_size + 3];

			float3 normal = make_float3(v0.x, v0.y, v0.z);

			float depth = v0.w;
			int meshid = __float_as_int(v3.w);
#endif

			float4 color;
			float variance;

			if (isFirstIter) {
#if 0
				color = make_float4(aovs[qidx].color, 1);
				variance = aovs[qidx].var;
#else
				auto v2 = ((float4*)aovs)[qidx * idaten::SVGFPathTracing::AOV_float4_size + 2];
				color = v2;
				variance = v2.w;
#endif
			}
			else {
				color = clrBuffer[qidx];
				variance = varBuffer[qidx];
			}

			float lum = AT_NAME::color::luminance(color.x, color.y, color.z);

			float Wz = min(expf(-abs(centerDepth - depth) / (sigmaZ * abs(dot(ddZ, p - q)) + 0.000001f)), 1.0f);

			float Wn = powf(max(0.0f, dot(centerNormal, normal)), sigmaN);

			float Wl = min(expf(-abs(centerLum - lum) / (sigmaL * sqrGaussedVarLum + 0.000001f)), 1.0f);

			float Wm = meshid == centerMeshId ? 1.0f : 0.0f;

			float W = Wz * Wn * Wl * Wm;
			
			sumC += h[pos] * W * color;
			weightC += h[pos] * W;

			sumV += (h[pos] * h[pos]) * (W * W) * variance;
			weightV += h[pos] * W;

			pos++;
		}
	}

	if (weightC > 0.0) {
		sumC /= weightC;
	}
	if (weightV > 0.0) {
		sumV /= (weightV * weightV);
	}

	nextClrBuffer[idx] = sumC;
	nextVarBuffer[idx] = sumV;

	if (isFirstIter) {
		// Store color temporary.
		tmpBuffer[idx] = sumC;
	}
	
	if (isFinalIter) {
#if 0
		sumC *= make_float4(aovs[idx].texclr, 1);
#else
		sumC *= v1;
#endif

		surf2Dwrite(
			sumC,
			dst,
			ix * sizeof(float4), iy,
			hipBoundaryModeTrap);
	}
}

__global__ void copyFromBufferToAov(
	float4* src,
	idaten::SVGFPathTracing::AOV* aovs,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	float4 s = src[idx];
	aovs[idx].color = make_float3(s.x, s.y, s.z);
}

namespace idaten
{
	void SVGFPathTracing::onAtrousFilter(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		static const int ITER = 5;

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		int cur = 0;
		int next = 1;

		for (int i = 0; i < ITER; i++) {
			int stepScale = 1 << i;

			if (i == 0) {
				// First.
				atrousFilter<true, false> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
#if 1
			else if (i == ITER - 1) {
				// Final.
				atrousFilter<false, true> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
			else {
				atrousFilter<false, false> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
#endif

			cur = next;
			next = 1 - cur;
		}
	}

	void SVGFPathTracing::copyFromTmpBufferToAov(int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		// Copy color from temporary buffer to AOV buffer for next temporal reprojection.
		copyFromBufferToAov << <grid, block >> > (
			m_tmpBuf.ptr(),
			curaov.ptr(),
			width, height);
		checkCudaKernel(copyFromBufferToAov);
	}
}
