#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

// NOTE
// ddx, ddy
// http://mosapui.blog116.fc2.com/blog-entry-35.html
// https://www.gamedev.net/forums/topic/478820-derivative-instruction-details-ddx-ddy-or-dfdx-dfdy-etc/
// http://d.hatena.ne.jp/umonist/20110616/p1
// http://monsho.blog63.fc2.com/blog-entry-105.html

inline __device__ float ddx(
	int x, int y,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov)
{
	// NOTE
	// 2x2 pixel���ƂɌv�Z����.

	int leftX = x; 
	int rightX = x + 1;
	if ((x & 0x01) == 1) {
		leftX = x - 1;
		rightX = x;
	}

	rightX = min(rightX, w - 1);

	const int idxL = getIdx(leftX, y, w);
	const int idxR = getIdx(rightX, y, w);

	float left = aov[idxL].depth;
	float right = aov[idxR].depth;

	return right - left;
}

inline __device__ float ddy(
	int x, int y,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov)
{
	// NOTE
	// 2x2 pixel���ƂɌv�Z����.

	int topY = y;
	int bottomY = y + 1;
	if ((y & 0x01) == 1) {
		topY = y - 1;
		bottomY = y;
	}

	bottomY = min(bottomY, h - 1);

	int idxT = getIdx(x, topY, w);
	int idxB = getIdx(x, bottomY, w);

	float top = aov[idxT].depth;
	float bottom = aov[idxB].depth;

	return bottom - top;
}

template <bool isReferAOV, int Target>
inline __device__ float gaussFilter3x3(
	int ix, int iy,
	int w, int h,
	idaten::SVGFPathTracing::AOV* aov,
	const float2* __restrict__ var)
{
	static const float kernel[] = {
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
		1.0 / 8.0,  1.0 / 4.0, 1.0 / 8.0,
		1.0 / 16.0, 1.0 / 8.0, 1.0 / 16.0,
	};

	float sum = 0;

	int pos = 0;

	for (int y = -1; y <= 1; y++) {
		for (int x = -1; x <= 1; x++) {
			int xx = clamp(ix + x, 0, w - 1);
			int yy = clamp(iy + y, 0, h - 1);

			int idx = getIdx(xx, yy, w);

			float tmp;
			if (isReferAOV) {
				tmp = aov[idx].var[Target];
			}
			else {
				tmp = Target == 0 ? var[idx].x : var[idx].y;
			}

			sum += kernel[pos] * tmp;

			pos++;
		}
	}

	return sum;
}

template <bool isFirstIter, bool isFinalIter>
__global__ void atrousFilter(
	hipSurfaceObject_t dst,
	idaten::SVGFPathTracing::Store* tmpBuffer,
	idaten::SVGFPathTracing::AOV* aovs,
	const idaten::SVGFPathTracing::Store* __restrict__ clrBuffer,
	idaten::SVGFPathTracing::Store* nextClrBuffer,
	const float2* __restrict__ varBuffer,
	float2* nextVarBuffer,
	int stepScale,
	float thresholdTemporalWeight,
	int radiusScale,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	auto centerNormal = aovs[idx].normal;

	float centerDepth = aovs[idx].depth;
	int centerMeshId = aovs[idx].meshid;

	float tmpDdzX = ddx(ix, iy, width, height, aovs);
	float tmpDdzY = ddy(ix, iy, width, height, aovs);
	float2 ddZ = make_float2(tmpDdzX, tmpDdzY);

	float4 centerColorDirect;
	float4 centerColorIndirect;

	if (isFirstIter) {
		centerColorDirect = aovs[idx].color[idaten::SVGFPathTracing::LightType::Direct];
		centerColorIndirect = aovs[idx].color[idaten::SVGFPathTracing::LightType::Indirect];
	}
	else {
		centerColorDirect = clrBuffer[idx].f[idaten::SVGFPathTracing::LightType::Direct];
		centerColorIndirect = clrBuffer[idx].f[idaten::SVGFPathTracing::LightType::Indirect];
	}

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���̂܂܏o�͂��ďI��.
		nextClrBuffer[idx].f[idaten::SVGFPathTracing::LightType::Direct] = centerColorDirect;

		if (isFinalIter) {
			centerColorDirect *= aovs[idx].texclr;

			surf2Dwrite(
				centerColorDirect,
				dst,
				ix * sizeof(float4), iy,
				hipBoundaryModeTrap);
		}

		return;
	}

	float centerLumDirect = AT_NAME::color::luminance(centerColorDirect.x, centerColorDirect.y, centerColorDirect.z);
	float centerLumIndirect = AT_NAME::color::luminance(centerColorIndirect.x, centerColorIndirect.y, centerColorIndirect.z);

	// �K�E�X�t�B���^3x3
	float gaussedVarLumDirect;
	float gaussedVarLumIndirect;
	
	if (isFirstIter) {
		gaussedVarLumDirect = gaussFilter3x3<true, idaten::SVGFPathTracing::LightType::Direct>(ix, iy, width, height, aovs, varBuffer);
		gaussedVarLumIndirect = gaussFilter3x3<true, idaten::SVGFPathTracing::LightType::Indirect>(ix, iy, width, height, aovs, varBuffer);
	}
	else {
		gaussedVarLumDirect = gaussFilter3x3<false, idaten::SVGFPathTracing::LightType::Direct>(ix, iy, width, height, aovs, varBuffer);
		gaussedVarLumIndirect = gaussFilter3x3<false, idaten::SVGFPathTracing::LightType::Indirect>(ix, iy, width, height, aovs, varBuffer);
	}

	float sqrGaussedVarLumDirect = sqrt(gaussedVarLumDirect);
	float sqrGaussedVarLumIndirect = sqrt(gaussedVarLumIndirect);

	static const float sigmaZ = 1.0f;
	static const float sigmaN = 128.0f;
	static const float sigmaL = 4.0f;

	float2 p = make_float2(ix, iy);

	// NOTE
	// 5x5

	float4 sumClrDirect = make_float4(0, 0, 0, 0);
	float weightClrDirect = 0;

	float4 sumClrIndirect = make_float4(0, 0, 0, 0);
	float weightClrIndirect = 0;

	float sumVarDirect = 0;
	float weightVarDirect = 0;

	float sumVarIndirect = 0;
	float weightVarIndirect = 0;

	int pos = 0;

	static const float h[] = {
		1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
		1.0 / 64.0,  1.0 / 16.0, 3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
		3.0 / 128.0, 3.0 / 32.0, 9.0 / 64.0,  3.0 / 32.0, 3.0 / 128.0,
		1.0 / 64.0,  1.0 / 16.0, 3.0 / 32.0,  1.0 / 16.0, 1.0 / 64.0,
		1.0 / 256.0, 1.0 / 64.0, 3.0 / 128.0, 1.0 / 64.0, 1.0 / 256.0,
	};

	int R = 2;

	if (isFirstIter) {
		if (aovs[idx].temporalWeight < thresholdTemporalWeight) {
			R *= radiusScale;
		}
	}

	for (int y = -R; y <= R; y++) {
		for (int x = -R; x <= R; x++) {
			int xx = clamp(ix + x * stepScale, 0, width - 1);
			int yy = clamp(iy + y * stepScale, 0, height - 1);

			float2 q = make_float2(xx, yy);

			const int qidx = getIdx(xx, yy, width);

			float depth = aovs[qidx].depth;
			int meshid = aovs[qidx].meshid;

			if (meshid != centerMeshId) {
				continue;
			}

			auto normal = aovs[qidx].normal;

			float4 colorDirect;
			float4 colorIndirect;

			float varDirect;
			float varIndirect;

			if (isFirstIter) {
				colorDirect = aovs[qidx].color[idaten::SVGFPathTracing::LightType::Direct];
				varDirect = aovs[qidx].var[idaten::SVGFPathTracing::LightType::Direct];

				colorIndirect = aovs[qidx].color[idaten::SVGFPathTracing::LightType::Indirect];
				varIndirect = aovs[qidx].var[idaten::SVGFPathTracing::LightType::Indirect];
			}
			else {
				colorDirect = clrBuffer[qidx].f[idaten::SVGFPathTracing::LightType::Direct];
				varDirect = varBuffer[qidx].x;

				colorIndirect = clrBuffer[qidx].f[idaten::SVGFPathTracing::LightType::Indirect];
				varIndirect = varBuffer[qidx].y;
			}

			float lumDirect = AT_NAME::color::luminance(colorDirect.x, colorDirect.y, colorDirect.z);
			float lumIndirect = AT_NAME::color::luminance(colorIndirect.x, colorIndirect.y, colorIndirect.z);

			float Wz = min(exp(-abs(centerDepth - depth) / (sigmaZ * abs(dot(ddZ, p - q)) + 0.000001f)), 1.0f);

			float Wn = pow(max(0.0f, dot(centerNormal, normal)), sigmaN);

			float Wl_Direct = min(exp(-abs(centerLumDirect - lumDirect) / (sigmaL * sqrGaussedVarLumDirect + 0.000001f)), 1.0f);
			float Wl_Indirect = min(exp(-abs(centerLumIndirect - lumIndirect) / (sigmaL * sqrGaussedVarLumIndirect + 0.000001f)), 1.0f);

			float Wm = meshid == centerMeshId ? 1.0f : 0.0f;

			float W_Direct = Wz * Wn * Wm * Wl_Direct;
			float W_Indirect = Wz * Wn * Wm * Wl_Indirect;
			
			sumClrDirect += h[pos] * W_Direct * colorDirect;
			weightClrDirect += h[pos] * W_Direct;

			sumVarDirect += (h[pos] * h[pos]) * (W_Direct * W_Direct) * varDirect;
			weightVarDirect += h[pos] * W_Direct;

			sumClrIndirect += h[pos] * W_Indirect * colorIndirect;
			weightClrIndirect += h[pos] * W_Indirect;

			sumVarIndirect += (h[pos] * h[pos]) * (W_Indirect * W_Indirect) * varIndirect;
			weightVarIndirect += h[pos] * W_Indirect;

			pos++;
		}
	}

	if (weightClrDirect > 0.0) {
		sumClrDirect /= weightClrDirect;
	}
	if (weightVarDirect > 0.0) {
		sumVarDirect /= (weightVarDirect * weightVarDirect);
	}

	if (weightClrIndirect > 0.0) {
		sumClrIndirect /= weightClrIndirect;
	}
	if (weightVarIndirect > 0.0) {
		sumVarIndirect /= (weightVarIndirect * weightVarIndirect);
	}

	nextClrBuffer[idx].f[idaten::SVGFPathTracing::LightType::Direct] = sumClrDirect;
	nextVarBuffer[idx].x = sumVarDirect;

	nextClrBuffer[idx].f[idaten::SVGFPathTracing::LightType::Indirect] = sumClrIndirect;
	nextVarBuffer[idx].y = sumVarIndirect;

	if (isFirstIter) {
		// Store color temporary.
		tmpBuffer[idx].f[idaten::SVGFPathTracing::LightType::Direct] = sumClrDirect;
		tmpBuffer[idx].f[idaten::SVGFPathTracing::LightType::Indirect] = sumClrIndirect;
	}	
}

__global__ void copyFromBufferToAov(
	const idaten::SVGFPathTracing::Store* __restrict__ src,
	idaten::SVGFPathTracing::AOV* aovs,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	aovs[idx].color[0] = src[idx].f[0];
	aovs[idx].color[1] = src[idx].f[1];
}

__global__ void modulateTexColor(
	hipSurfaceObject_t dst,
	const idaten::SVGFPathTracing::Store* __restrict__ buffer,
	const idaten::SVGFPathTracing::AOV* __restrict__ aovs,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	auto direct = buffer[idx].f[idaten::SVGFPathTracing::LightType::Direct];
	auto indirect = buffer[idx].f[idaten::SVGFPathTracing::LightType::Indirect];

	auto clr = (direct + indirect) * aovs[idx].texclr;

	surf2Dwrite(
		clr,
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::onAtrousFilter(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		static const int ITER = 5;

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		int cur = 0;
		int next = 1;

		for (int i = 0; i < ITER; i++) {
			int stepScale = 1 << i;

			if (i == 0) {
				// First.
				atrousFilter<true, false> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
#if 1
			else if (i == ITER - 1) {
				// Final.
				atrousFilter<false, true> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
			else {
				atrousFilter<false, false> << <grid, block >> > (
					outputSurf,
					m_tmpBuf.ptr(),
					curaov.ptr(),
					m_atrousClr[cur].ptr(), m_atrousClr[next].ptr(),
					m_atrousVar[cur].ptr(), m_atrousVar[next].ptr(),
					stepScale,
					m_thresholdTemporalWeight, m_atrousTapRadiusScale,
					width, height);
				checkCudaKernel(atrousFilter);
			}
#endif

			cur = next;
			next = 1 - cur;
		}

		modulateTexColor << < grid, block >> > (
			outputSurf,
			m_atrousClr[cur].ptr(),
			curaov.ptr(),
			width, height);
	}

	void SVGFPathTracing::copyFromTmpBufferToAov(int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		// Copy color from temporary buffer to AOV buffer for next temporal reprojection.
		copyFromBufferToAov << <grid, block >> > (
			m_tmpBuf.ptr(),
			curaov.ptr(),
			width, height);
		checkCudaKernel(copyFromBufferToAov);
	}
}
