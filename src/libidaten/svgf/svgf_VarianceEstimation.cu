#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

inline __device__ idaten::SVGFPathTracing::AOV* sampleAov(
	idaten::SVGFPathTracing::AOV* aovs,
	int ix, int iy,
	int width, int height)
{
	ix = clamp(ix, 0, width - 1);
	iy = clamp(iy, 0, height - 1);

	const int idx = getIdx(ix, iy, width);

	return &aovs[idx];
}

__global__ void varianceEstimation(
	hipSurfaceObject_t dst,
	idaten::SVGFPathTracing::AOV* aovs,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	float centerDepth = aovs[idx].depth;
	int centerMeshId = aovs[idx].meshid;

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���U�̓[��.
		aovs[idx].moments[0] = aovs[idx].moments[1] = make_float4(0, 0, 0, 1);

		surf2Dwrite(
			make_float4(0),
			dst,
			ix * sizeof(float4), iy,
			hipBoundaryModeTrap);
	}

	float4 centerMomentDirect = aovs[idx].moments[idaten::SVGFPathTracing::LightType::Direct];
	float4 centerMomentIndirect = aovs[idx].moments[idaten::SVGFPathTracing::LightType::Indirect];

	int frame = (int)centerMomentDirect.w;

	centerMomentDirect /= centerMomentDirect.w;
	centerMomentIndirect /= centerMomentIndirect.w;

	// ���U���v�Z.
	float varDirect = centerMomentDirect.x - centerMomentDirect.y * centerMomentDirect.y;
	float varIndirect = centerMomentIndirect.x - centerMomentIndirect.y * centerMomentIndirect.y;

	if (frame < 4) {
		// �ώZ�t���[�������S���� or Disoccluded����Ă���.
		// 7x7birateral filter�ŋP�x���v�Z.

		static const int radius = 3;
		static const float sigmaN = 0.005f;
		static const float sigmaD = 0.005f;
		static const float sigmaS = 8;

		auto centerNormal = aovs[idx].normal;

		float4 sumDirect = make_float4(0, 0, 0, 0);
		float4 sumIndirect = make_float4(0, 0, 0, 0);

		float weight = 0.0f;

		for (int v = -radius; v <= radius; v++)
		{
			for (int u = -radius; u <= radius; u++)
			{
				auto sampleaov = sampleAov(aovs, ix + u, iy + v, width, height);

				auto sampleNml = sampleaov->normal;

				float sampleDepth = sampleaov->depth;
				int sampleMeshId = sampleaov->meshid;

				float n = 1 - dot(sampleNml, centerNormal);
				float Wn = exp(-0.5f * n * n / (sigmaN * sigmaN));

				float d = 1 - min(centerDepth, sampleDepth) / max(centerDepth, sampleDepth);
				float Wd = exp(-0.5f * d * d / (sigmaD * sigmaD));

				float Ws = exp(-0.5f * (u * u + v * v) / (sigmaS * sigmaS));

				float Wm = centerMeshId == sampleMeshId ? 1.0f : 0.0f;

				float W = Ws * Wn * Wd * Wm;

				auto momentDirect = sampleaov->moments[idaten::SVGFPathTracing::LightType::Direct];
				momentDirect /= momentDirect.w;

				auto momentIndirect = sampleaov->moments[idaten::SVGFPathTracing::LightType::Indirect];
				momentIndirect /= momentIndirect.w;

				sumDirect += momentDirect * W;
				sumIndirect += momentIndirect * W;

				weight += W;
			}
		}

		if (weight > 0.0f) {
			sumDirect /= weight;
			sumIndirect /= weight;
		}

		varDirect = sumDirect.x - sumDirect.y * sumDirect.y;
		varIndirect = sumIndirect.x - sumIndirect.y * sumIndirect.y;
	}

	// TODO
	// ���U�̓}�C�i�X�ɂȂ�Ȃ����E�E�E�E
	varDirect = abs(varDirect);
	varIndirect = abs(varIndirect);

	aovs[idx].var[idaten::SVGFPathTracing::LightType::Direct] = varDirect;
	aovs[idx].var[idaten::SVGFPathTracing::LightType::Indirect] = varIndirect;

	surf2Dwrite(
		make_float4(varDirect, varIndirect, 0, 1),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::onVarianceEstimation(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		varianceEstimation << <grid, block >> > (
		//varianceEstimation << <1, 1 >> > (
			outputSurf,
			curaov.ptr(),
			width, height);

		checkCudaKernel(varianceEstimation);
	}
}
