#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

inline __device__ idaten::SVGFPathTracing::AOV* sampleAov(
	idaten::SVGFPathTracing::AOV* aovs,
	int ix, int iy,
	int width, int height)
{
	ix = clamp(ix, 0, width - 1);
	iy = clamp(iy, 0, height - 1);

	const int idx = getIdx(ix, iy, width);

	return &aovs[idx];
}

__global__ void varianceEstimation(
	hipSurfaceObject_t dst,
	idaten::SVGFPathTracing::AOV* aovs,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	float centerDepth = aovs[idx].depth;
	int centerMeshId = aovs[idx].meshid;

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���U�̓[��.
		aovs[idx].moments = make_float4(0, 0, 0, 1);

		surf2Dwrite(
			make_float4(0),
			dst,
			ix * sizeof(float4), iy,
			hipBoundaryModeTrap);
	}

	float4 centerMoment = aovs[idx].moments;

	int frame = (int)centerMoment.w;

	centerMoment /= centerMoment.w;

	// ���U���v�Z.
	float var = centerMoment.x - centerMoment.y * centerMoment.y;

	if (frame < 4) {
		// �ώZ�t���[�������S���� or Disoccluded����Ă���.
		// 7x7birateral filter�ŋP�x���v�Z.

		static const int radius = 3;
		static const float sigmaN = 0.005f;
		static const float sigmaD = 0.005f;
		static const float sigmaS = 8;

		float4 centerNormal = aovs[idx].normal;

		float4 sum = make_float4(0, 0, 0, 0);
		float weight = 0.0f;

		for (int v = -radius; v <= radius; v++)
		{
			for (int u = -radius; u <= radius; u++)
			{
				auto sampleaov = sampleAov(aovs, ix + u, iy + v, width, height);
				
				auto moment = sampleaov->moments;
				moment /= moment.w;

				auto sampleNml = sampleaov->normal;

				float sampleDepth = sampleaov->depth;
				int sampleMeshId = sampleaov->meshid;

				float n = 1 - dot(sampleNml, centerNormal);
				float Wn = exp(-0.5f * n * n / (sigmaN * sigmaN));

				float d = 1 - min(centerDepth, sampleDepth) / max(centerDepth, sampleDepth);
				float Wd = exp(-0.5f * d * d / (sigmaD * sigmaD));

				float Ws = exp(-0.5f * (u * u + v * v) / (sigmaS * sigmaS));

				float Wm = centerMeshId == sampleMeshId ? 1.0f : 0.0f;

				float W = Ws * Wn * Wd * Wm;
				sum += moment * W;
				weight += W;
			}
		}

		if (weight > 0.0f) {
			sum /= weight;
		}

		var = sum.x - sum.y * sum.y;
	}

	// TODO
	// ���U�̓}�C�i�X�ɂȂ�Ȃ����E�E�E�E
	var = abs(var);

	aovs[idx].var = make_float4(var, var, var, var);

	surf2Dwrite(
		make_float4(var, var, var, var),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::onVarianceEstimation(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		varianceEstimation << <grid, block >> > (
		//varianceEstimation << <1, 1 >> > (
			outputSurf,
			curaov.ptr(),
			width, height);

		checkCudaKernel(varianceEstimation);
	}
}
