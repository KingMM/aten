#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

inline __device__ float3 computeViewSpace(
	int ix, int iy,
	float centerDepth,
	int width, int height,
	const aten::mat4* mtxC2V)
{
	// NOTE
	// Pview = (Xview, Yview, Zview, 1)
	// mtxV2C = W 0 0  0
	//          0 H 0  0
	//          0 0 A  B
	//          0 0 -1 0
	// mtxV2C * Pview = (Xclip, Yclip, Zclip, Wclip) = (Xclip, Yclip, Zclip, Zview)
	//  Wclip = Zview = depth
	// Xscr = Xclip / Wclip = Xclip / Zview = Xclip / depth
	// Yscr = Yclip / Wclip = Yclip / Zview = Yclip / depth
	//
	// Xscr * depth = Xclip
	// Xview = mtxC2V * Xclip

	float2 uv = make_float2(ix + 0.5, iy + 0.5);
	uv /= make_float2(width - 1, height - 1);	// [0, 1]
	uv = uv * 2.0f - 1.0f;	// [0, 1] -> [-1, 1]

	aten::vec4 pos(uv.x, uv.y, 0, 0);

	// Screen-space -> Clip-space.
	pos.x *= centerDepth;
	pos.y *= centerDepth;

	// Clip-space -> View-space
	pos = mtxC2V->apply(pos);
	pos.z = -centerDepth;
	pos.w = 1.0;

	return make_float3(pos.x, pos.y, pos.z);
}

inline __device__ float C(float3 x1, float3 x2, float sigma)
{
	float a = length(x1 - x2) / sigma;
	a *= a;
	return expf(-0.5f * a);
}

inline __device__ float C(float x1, float x2, float sigma)
{
	float a = abs(x1 - x2) / sigma;
	a *= a;
	return expf(-0.5f * a);
}

#define IS_IN_BOUND(x, a, b)	((a) <= (x) && (x) < (b))

__global__ void varianceEstimation(
	hipSurfaceObject_t dst,
	const float4* __restrict__ aovNormalDepth,
	const float4* __restrict__ aovTexclrTemporalWeight,
	float4* aovColorVariance,
	float4* aovMomentMeshid,
	aten::mat4 mtxC2V,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const int idx = getIdx(ix, iy, width);

	auto normalDepth = aovNormalDepth[idx];
	auto momentMeshid = aovMomentMeshid[idx];

	float centerDepth = aovNormalDepth[idx].w;
	int centerMeshId = (int)momentMeshid.w;

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���U�̓[��.
		aovMomentMeshid[idx].x = 0;
		aovMomentMeshid[idx].y = 0;
		aovMomentMeshid[idx].z = 1;

		surf2Dwrite(
			make_float4(0),
			dst,
			ix * sizeof(float4), iy,
			hipBoundaryModeTrap);
	}

	float3 centerViewPos = computeViewSpace(ix, iy, centerDepth, width, height, &mtxC2V);

	float3 centerMoment = make_float3(momentMeshid.x, momentMeshid.y, momentMeshid.z);

	int frame = (int)centerMoment.z;

	centerMoment /= centerMoment.z;

	// ���U���v�Z.
	float var = centerMoment.x - centerMoment.y * centerMoment.y;

	if (frame < 4) {
		// �ώZ�t���[�������S���� or Disoccluded����Ă���.
		// 7x7birateral filter�ŋP�x���v�Z.

		static const int radius = 3;
		static const float sigmaN = 0.005f;
		static const float sigmaD = 0.005f;
		static const float sigmaS = 0.965f;

		float3 centerNormal = make_float3(normalDepth.x, normalDepth.y, normalDepth.z);

		float3 sum = make_float3(0);
		float weight = 0.0f;

#if 0
		for (int v = -radius; v <= radius; v++)
		{
			for (int u = -radius; u <= radius; u++)
			{
#else
		static const int offsetx[] = {
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
			-3, -2, -1, 0, 1, 2, 3,
		};

		static const int offsety[] = {
			-3, -3, -3, -3, -3, -3, -3,
			-2,	-2,	-2,	-2,	-2,	-2,	-2,
			-1,	-1,	-1,	-1,	-1,	-1,	-1,
			 0,	 0,	 0,	 0,	 0,	 0,	 0,
			 1,	 1,	 1,	 1,	 1,	 1,	 1,
			 2,	 2,	 2,	 2,	 2,	 2,	 2,
			 3,	 3,	 3,	 3,	 3,	 3,	 3,
		};

#pragma unroll
		for (int i = 0; i < 49; i++) {
		{
				int u = offsetx[i];
				int v = offsety[i];
#endif
				if (IS_IN_BOUND(ix + u, 0, width)
					&& IS_IN_BOUND(iy + v, 0, height))
				{
					int xx = clamp(ix + u, 0, width - 1);
					int yy = clamp(iy + v, 0, height - 1);

					int pidx = getIdx(xx, yy, width);
					normalDepth = aovNormalDepth[pidx];
					momentMeshid = aovMomentMeshid[pidx];

					float3 sampleNml = make_float3(normalDepth.x, normalDepth.y, normalDepth.z);
					float sampleDepth = normalDepth.w;
					int sampleMeshId = (int)momentMeshid.w;

					float3 moment = make_float3(momentMeshid.x, momentMeshid.y, momentMeshid.z);
					//moment /= moment.z;

#if 0
					float n = 1 - dot(sampleNml, centerNormal);
					float Wn = exp(-0.5f * n * n / (sigmaN * sigmaN));

					float d = 1 - min(centerDepth, sampleDepth) / max(centerDepth, sampleDepth);
					float Wd = exp(-0.5f * d * d / (sigmaD * sigmaD));

					float Ws = exp(-0.5f * (u * u + v * v) / (sigmaS * sigmaS));
#elif 0
					float Wn = 1.0f;
					{
						float normalCloseness = dot(sampleNml, centerNormal);
						normalCloseness = normalCloseness * normalCloseness;
						normalCloseness = normalCloseness * normalCloseness;
						float normalError = (1.0f - normalCloseness);
						Wn = max((1.0f - normalError), 0.0f);
					}

					float Wd = max(0.0f, 1.0f - abs(centerDepth - sampleDepth));

					float Ws = 1.0f;
					{
						auto sampleViewPos = computeViewSpace(ix + u, iy + v, sampleDepth, width, height, &mtxC2V);

						// Change in position in camera space.
						auto dq = centerViewPos - sampleViewPos;

						// How far away is this point from the original sample in camera space? (Max value is unbounded).
						auto dist2 = dot(dq, dq);

						// How far off the expected plane (on the perpendicular) is this point?  Max value is unbounded.
						float err = max(abs(dot(dq, sampleNml)), abs(dot(dq, centerNormal)));

						Ws = (dist2 < 0.001f)
							? 1.0
							: pow(max(0.0, 1.0 - 2.0 * err / sqrt(dist2)), 2.0);
					}
#else
					float3 sampleViewPos = computeViewSpace(ix + u, iy + v, sampleDepth, width, height, &mtxC2V);

					float Wn = C(centerNormal, sampleNml, 0.1f);
					float Ws = C(centerViewPos, sampleViewPos, 0.1f);
					float Wd = C(centerDepth, sampleDepth, 0.1f);
#endif

					float Wm = centerMeshId == sampleMeshId ? 1.0f : 0.0f;

					float W = Ws * Wn * Wd * Wm;
					sum += moment * W;
					weight += W;
				}
			}
		}

		if (weight > 0.0f) {
			sum /= weight;
		}

		var = sum.x - sum.y * sum.y;
	}

	// TODO
	// ���U�̓}�C�i�X�ɂȂ�Ȃ����E�E�E�E
	var = abs(var);

	aovColorVariance[idx].w = var;

	surf2Dwrite(
		make_float4(var, var, var, var),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::onVarianceEstimation(
		Resolution resType,
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int curaov = getCurAovs();

		varianceEstimation << <grid, block >> > (
		//varianceEstimation << <1, 1 >> > (
			outputSurf,
			m_aovNormalDepth[resType][curaov].ptr(),
			m_aovTexclrTemporalWeight[resType][curaov].ptr(),
			m_aovColorVariance[resType][curaov].ptr(),
			m_aovMomentMeshid[resType][curaov].ptr(),
			m_mtxC2V,
			width, height);

		checkCudaKernel(varianceEstimation);
	}
}
