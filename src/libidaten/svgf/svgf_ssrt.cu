#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void hitTestPrimaryRayInScreenSpace(
	hipSurfaceObject_t gbuffer,
	idaten::SVGFPathTracing::Path* paths,
	aten::Intersection* isects,
	int* hitbools,
	int width, int height,
	const aten::vec4 camPos,
	const aten::GeomParameter* __restrict__ geoms,
	const aten::PrimitiveParamter* __restrict__ prims,
	const aten::mat4* __restrict__ matrices,
	hipTextureObject_t vtxPos)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.isHit = false;

	hitbools[idx] = 0;

	if (path.isTerminate) {
		return;
	}

	// Sample data from texture.
	float4 data;
	surf2Dread(&data, gbuffer, ix * sizeof(float4), iy);

	// NOTE
	// x : objid
	// y : primid
	// zw : bary centroid

	int objid = __float_as_int(data.x);
	int primid = __float_as_int(data.y);

	isects[idx].objid = objid;
	isects[idx].primid = primid;

	// bary centroid.
	isects[idx].a = data.z;
	isects[idx].b = data.w;

	if (objid >= 0) {
		aten::PrimitiveParamter prim;
		prim.v0 = ((aten::vec4*)prims)[primid * aten::PrimitiveParamter_float4_size + 0];
		prim.v1 = ((aten::vec4*)prims)[primid * aten::PrimitiveParamter_float4_size + 1];

		isects[idx].mtrlid = prim.mtrlid;
		isects[idx].meshid = prim.gemoid;

		const auto* obj = &geoms[objid];

		float4 p0 = tex1Dfetch<float4>(vtxPos, prim.idx[0]);
		float4 p1 = tex1Dfetch<float4>(vtxPos, prim.idx[1]);
		float4 p2 = tex1Dfetch<float4>(vtxPos, prim.idx[2]);

		real a = data.z;
		real b = data.w;
		real c = 1 - a - b;

		// �d�S���W�n(barycentric coordinates).
		// v0�.
		// p = (1 - a - b)*v0 + a*v1 + b*v2
		auto p = c * p0 + a * p1 + b * p2;
		aten::vec4 vp(p.x, p.y, p.z, 1.0f);

		if (obj->mtxid >= 0) {
			auto mtxL2W = matrices[obj->mtxid * 2 + 0];
			vp = mtxL2W.apply(vp);
		}

		isects[idx].t = (camPos - vp).length();

		path.isHit = true;
		hitbools[idx] = 1;
	}
	else {
		path.isHit = false;
		hitbools[idx] = 0;
	}
}

namespace idaten
{
	void SVGFPathTracing::onScreenSpaceHitTest(
		int width, int height,
		int bounce,
		hipTextureObject_t texVtxPos)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		aten::vec4 campos = aten::vec4(m_camParam.origin, 1.0f);

		CudaGLResourceMap rscmap(&m_gbuffer);
		auto gbuffer = m_gbuffer.bind();

		hitTestPrimaryRayInScreenSpace << <grid, block >> > (
			gbuffer,
			m_paths[Resolution::Hi].ptr(),
			m_isects.ptr(),
			m_hitbools.ptr(),
			width, height,
			campos,
			m_shapeparam.ptr(),
			m_primparams.ptr(),
			m_mtxparams.ptr(),
			texVtxPos);

		checkCudaKernel(hitTestPrimaryRayInScreenSpace);
	}
}
