#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

//#define ENABLE_DEBUG_1PIXEL

#ifdef ENABLE_DEBUG_1PIXEL
#define DEBUG_IX	(140)
#define DEBUG_IY	(511 - 81)
#endif

template <bool isFillAOV>
__global__ void genPath(
	idaten::SVGFPathTracing::Path* paths,
	aten::ray* rays,
	int width, int height,
	int sample, int maxSamples,
	unsigned int frame,
	const aten::CameraParameter* __restrict__ camera,
	const unsigned int* sobolmatrices,
	unsigned int* random)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isKill) {
		path.isTerminate = true;
		return;
	}

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
	auto scramble = random[idx] * 0x1fe3434f;
	path.sampler.init(frame, 0, scramble, sobolmatrices);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
	auto rnd = random[idx];
	auto scramble = rnd * 0x1fe3434f * ((frame + 133 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
	path.sampler.init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 0, scramble);
#endif

	float r1 = path.sampler.nextSample();
	float r2 = path.sampler.nextSample();

	if (isFillAOV) {
		r1 = r2 = 0.5f;
	}

	float s = (ix + r1) / (float)(camera->width);
	float t = (iy + r2) / (float)(camera->height);

	AT_NAME::CameraSampleResult camsample;
	AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

	rays[idx] = camsample.r;

	path.throughput = aten::vec3(1);
	path.pdfb = 0.0f;
	path.isTerminate = false;
	path.isSingular = false;

	path.samples += 1;

	// Accumulate value, so do not reset.
	//path.contrib = aten::vec3(0);
}

__global__ void hitTest(
	idaten::SVGFPathTracing::Path* paths,
	aten::Intersection* isects,
	aten::ray* rays,
	int* hitbools,
	int width, int height,
	const aten::ShapeParameter* __restrict__ shapes, int geomnum,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	aten::mat4* matrices)
{
#if 0
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= width * height) {
		return;
	}
#else
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);
#endif

	auto& path = paths[idx];
	path.isHit = false;

	hitbools[idx] = 0;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	aten::Intersection isect;

	bool isHit = intersectBVH(&ctxt, rays[idx], &isect);

	isects[idx].t = isect.t;
	isects[idx].objid = isect.objid;
	isects[idx].mtrlid = isect.mtrlid;
	isects[idx].meshid = isect.meshid;
	isects[idx].area = isect.area;
	isects[idx].primid = isect.primid;
	isects[idx].a = isect.a;
	isects[idx].b = isect.b;

	path.isHit = isHit;

	hitbools[idx] = isHit ? 1 : 0;
}

template <bool isFirstBounce>
__global__ void shadeMiss(
	idaten::SVGFPathTracing::AOV* aovs,
	idaten::SVGFPathTracing::Path* paths,
	int width, int height)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];

	if (!path.isTerminate && !path.isHit) {
		// TODO
		auto bg = aten::vec3(0);

		if (isFirstBounce) {
			path.isKill = true;

			// Export bg color to albedo buffer.
			aovs[idx].texclr = make_float4(bg.x, bg.y, bg.z, 1);
			aovs[idx].depth = -1;
			aovs[idx].meshid = -1;
			aovs[idx].mtrlid = -1;

			// For exporting separated albedo.
			bg = aten::vec3(1, 1, 1);
		}

		path.contrib += path.throughput * bg;

		path.isTerminate = true;
	}
}

template <bool isFirstBounce>
__global__ void shadeMissWithEnvmap(
	idaten::SVGFPathTracing::AOV* aovs,
	hipTextureObject_t* textures,
	int envmapIdx,
	real envmapAvgIllum,
	real envmapMultiplyer,
	idaten::SVGFPathTracing::Path* paths,
	const aten::ray* __restrict__ rays,
	int width, int height)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];

	if (!path.isTerminate && !path.isHit) {
		auto r = rays[idx];

		auto uv = AT_NAME::envmap::convertDirectionToUV(r.dir);

		auto bg = tex2D<float4>(textures[envmapIdx], uv.x, uv.y);
		auto emit = aten::vec3(bg.x, bg.y, bg.z);

		float misW = 1.0f;
		if (isFirstBounce) {
			path.isKill = true;

			// Export envmap to albedo buffer.
			aovs[idx].texclr = make_float4(emit.x, emit.y, emit.z, 1);
			aovs[idx].depth = -1;
			aovs[idx].meshid = -1;
			aovs[idx].mtrlid = -1;

			// For exporting separated albedo.
			emit = aten::vec3(1, 1, 1);
		}
		else {
			auto pdfLight = AT_NAME::ImageBasedLight::samplePdf(emit, envmapAvgIllum);
			misW = path.pdfb / (pdfLight + path.pdfb);

			emit *= envmapMultiplyer;
		}

		path.contrib += path.throughput * misW * emit;

		path.isTerminate = true;
	}
}

template <bool isFirstBounce>
__global__ void shade(
	idaten::SVGFPathTracing::AOV* aovs,
	aten::mat4 mtxW2C,
	int width, int height,
	idaten::SVGFPathTracing::Path* paths,
	const int* __restrict__ hitindices,
	int hitnum,
	const aten::Intersection* __restrict__ isects,
	aten::ray* rays,
	int frame,
	int bounce, int rrBounce,
	const aten::ShapeParameter* __restrict__ shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	hipTextureObject_t vtxNml,
	const aten::mat4* __restrict__ matrices,
	hipTextureObject_t* textures,
	unsigned int* random)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= hitnum) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.vtxNml = vtxNml;
		ctxt.matrices = matrices;
		ctxt.textures = textures;
	}

	idx = hitindices[idx];

#ifdef ENABLE_DEBUG_1PIXEL
	int ix = DEBUG_IX;
	int iy = DEBUG_IY;
	idx = getIdx(ix, iy, width);
#endif

	auto& path = paths[idx];
	const auto& ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
	auto scramble = random[idx] * 0x1fe3434f;
	path.sampler.init(frame, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
	auto rnd = random[idx];
	auto scramble = rnd * 0x1fe3434f * ((frame + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
	path.sampler.init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 4 + bounce * 300, scramble);
#endif

	aten::hitrecord rec;

	const auto& isect = isects[idx];

	auto obj = &ctxt.shapes[isect.objid];
	evalHitResult(&ctxt, obj, ray, &rec, &isect);

	aten::MaterialParameter mtrl = ctxt.mtrls[rec.mtrlid];

	bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

	// �����ʒu�̖@��.
	// ���̂���̃��C�̓��o���l��.
	aten::vec3 orienting_normal = rec.normal;

	if (mtrl.type != aten::MaterialType::Layer) {
		mtrl.albedoMap = (int)(mtrl.albedoMap >= 0 ? ctxt.textures[mtrl.albedoMap] : -1);
		mtrl.normalMap = (int)(mtrl.normalMap >= 0 ? ctxt.textures[mtrl.normalMap] : -1);
		mtrl.roughnessMap = (int)(mtrl.roughnessMap >= 0 ? ctxt.textures[mtrl.roughnessMap] : -1);
	}

	// Render AOVs.
	// NOTE
	// �����ɖ@����AOV�ɕێ�����Ȃ�A�@���}�b�v�K�p�シ��ׂ�.
	// �������Atemporal reprojection�Aatrous�Ȃǂ̃t�B���^�K�p���ɖ@�����Q�Ƃ���ۂɁA�@���}�b�v���ׂ������Ă͂�����Ă��܂����Ƃ�����.
	// ����ɂ��A�t�B���^�����������悤�ɂ����炸�t�B���^�̕i�����������Ă��܂���肪��������.
	if (isFirstBounce) {
		int ix = idx % width;
		int iy = idx / width;

		// World coordinate to Clip coordinate.
		aten::vec4 pos = aten::vec4(rec.p, 1);
		pos = mtxW2C.apply(pos);

		// normal
		aovs[idx].normal = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, 0);

		// depth, meshid.
		aovs[idx].depth = pos.w;
		aovs[idx].meshid = isect.meshid;
		aovs[idx].mtrlid = rec.mtrlid;

		// texture color.
		auto texcolor = AT_NAME::material::sampleTexture(mtrl.albedoMap, rec.u, rec.v, 1.0f);
		aovs[idx].texclr = make_float4(texcolor.x, texcolor.y, texcolor.z, 1);

		// For exporting separated albedo.
		mtrl.albedoMap = -1;
	}

	// Implicit conection to light.
	if (mtrl.attrib.isEmissive) {
		if (!isBackfacing) {
			float weight = 1.0f;

			if (bounce > 0 && !path.isSingular) {
				auto cosLight = dot(orienting_normal, -ray.dir);
				auto dist2 = aten::squared_length(rec.p - ray.org);

				if (cosLight >= 0) {
					auto pdfLight = 1 / rec.area;

					// Convert pdf area to sradian.
					// http://www.slideshare.net/h013/edubpt-v100
					// p31 - p35
					pdfLight = pdfLight * dist2 / cosLight;

					weight = path.pdfb / (pdfLight + path.pdfb);
				}
			}

			path.contrib += path.throughput * weight * mtrl.baseColor;
		}

		// When ray hit the light, tracing will finish.
		path.isTerminate = true;
		return;
	}

	if (!mtrl.attrib.isTranslucent && isBackfacing) {
		orienting_normal = -orienting_normal;
	}

	// Apply normal map.
	int normalMap = mtrl.normalMap;
	if (mtrl.type == aten::MaterialType::Layer) {
		// �ŕ\�w�� NormalMap ��K�p.
		auto* topmtrl = &ctxt.mtrls[mtrl.layer[0]];
		normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
	}
	AT_NAME::material::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

#if 1
	// Explicit conection to light.
	if (!mtrl.attrib.isSingular)
	{
		real lightSelectPdf = 1;
		aten::LightSampleResult sampleres;

		// TODO
		// Importance sampling.
		int lightidx = aten::cmpMin<int>(path.sampler.nextSample() * lightnum, lightnum - 1);
		lightSelectPdf = 1.0f / lightnum;

		auto light = ctxt.lights[lightidx];

		sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &path.sampler);

		const auto& posLight = sampleres.pos;
		const auto& nmlLight = sampleres.nml;
		real pdfLight = sampleres.pdf;

		auto lightobj = sampleres.obj;

		auto dirToLight = normalize(sampleres.dir);
		auto distToLight = length(posLight - rec.p);

		real distHitObjToRayOrg = AT_MATH_INF;

		// Ray aim to the area light.
		// So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
		auto hitobj = lightobj;

		aten::Intersection isectTmp;
		
		auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;
		auto tmp = rec.p + dirToLight - shadowRayOrg;
		auto shadowRayDir = normalize(tmp);
		aten::ray shadowRay(shadowRayOrg, shadowRayDir);

		bool isHit = intersectCloserBVH(&ctxt, shadowRay, &isectTmp, distToLight - AT_MATH_EPSILON);

		if (isHit) {
			hitobj = (void*)&ctxt.shapes[isectTmp.objid];
		}

		isHit = AT_NAME::scene::hitLight(
			isHit,
			light.attrib,
			lightobj,
			distToLight,
			distHitObjToRayOrg,
			isectTmp.t,
			hitobj);

		if (isHit) {
			auto cosShadow = dot(orienting_normal, dirToLight);

			real pdfb = samplePDF(&ctxt, &mtrl, orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
			auto bsdf = sampleBSDF(&ctxt, &mtrl, orienting_normal, ray.dir, dirToLight, rec.u, rec.v);

			bsdf *= path.throughput;

			// Get light color.
			auto emit = sampleres.finalColor;

			if (light.attrib.isSingular || light.attrib.isInfinite) {
				if (pdfLight > real(0) && cosShadow >= 0) {
					// TODO
					// �W�I���g���^�[���̈����ɂ���.
					// singular light �̏ꍇ�́AfinalColor �ɋ����̏��Z���܂܂�Ă���.
					// inifinite light �̏ꍇ�́A���������ɂȂ�ApdfLight�Ɋ܂܂�鋗�������Ƒł����������H.
					// �i�ł����������̂ŁApdfLight�ɂ͋��������͊܂�ł��Ȃ��j.
					auto misW = pdfLight / (pdfb + pdfLight);
					path.contrib += (misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf;
				}
			}
			else {
				auto cosLight = dot(nmlLight, -dirToLight);

				if (cosShadow >= 0 && cosLight >= 0) {
					auto dist2 = aten::squared_length(sampleres.dir);
					auto G = cosShadow * cosLight / dist2;

					if (pdfb > real(0) && pdfLight > real(0)) {
						// Convert pdf from steradian to area.
						// http://www.slideshare.net/h013/edubpt-v100
						// p31 - p35
						pdfb = pdfb * cosLight / dist2;

						auto misW = pdfLight / (pdfb + pdfLight);

						path.contrib += (misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf;
					}
				}
			}
		}
	}
#endif

	real russianProb = real(1);

	if (bounce > rrBounce) {
		auto t = normalize(path.throughput);
		auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

		russianProb = path.sampler.nextSample();

		if (russianProb >= p) {
			//path.contrib = aten::vec3(0);
			path.isTerminate = true;
		}
		else {
			russianProb = p;
		}
	}
			
	AT_NAME::MaterialSampling sampling;

	sampleMaterial(
		&sampling,
		&ctxt,
		&mtrl,
		orienting_normal,
		ray.dir,
		rec.normal,
		&path.sampler,
		rec.u, rec.v);

	auto nextDir = normalize(sampling.dir);
	auto pdfb = sampling.pdf;
	auto bsdf = sampling.bsdf;

	real c = 1;
	if (!mtrl.attrib.isSingular) {
		// TODO
		// AMD�̂�abs���Ă��邪....
		//c = aten::abs(dot(orienting_normal, nextDir));
		c = dot(orienting_normal, nextDir);
	}

	if (pdfb > 0 && c > 0) {
		path.throughput *= bsdf * c / pdfb;
		path.throughput /= russianProb;
	}
	else {
		path.isTerminate = true;
	}

	// Make next ray.
	rays[idx] = aten::ray(rec.p, nextDir);

	path.pdfb = pdfb;
	path.isSingular = mtrl.attrib.isSingular;
}

__global__ void gather(
	hipSurfaceObject_t dst,
	idaten::SVGFPathTracing::AOV* aovs,
	const idaten::SVGFPathTracing::Path* __restrict__ paths,
	int width, int height)
{
	auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

#ifdef ENABLE_DEBUG_1PIXEL
	ix = DEBUG_IX;
	iy = DEBUG_IY;
#endif

	const auto idx = getIdx(ix, iy, width);

	const auto& path = paths[idx];

	int sample = path.samples;

	float4 contrib = make_float4(path.contrib.x, path.contrib.y, path.contrib.z, 0) / sample;
	contrib.w = sample;

	float lum = AT_NAME::color::luminance(contrib.x, contrib.y, contrib.z);

	aovs[idx].moments += make_float4(lum * lum, lum, 0, 1);

	aovs[idx].color = contrib;

#if 0
	auto n = aovs[idx].moments.w;

	auto m = aovs[idx].moments / n;

	auto var = m.x - m.y * m.y;

	surf2Dwrite(
		make_float4(var, var, var, 1),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
#else
	surf2Dwrite(
		contrib,
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
#endif
}

namespace idaten
{
	void SVGFPathTracing::update(
		GLuint gltex,
		int width, int height,
		const aten::CameraParameter& camera,
		const std::vector<aten::ShapeParameter>& shapes,
		const std::vector<aten::MaterialParameter>& mtrls,
		const std::vector<aten::LightParameter>& lights,
		const std::vector<std::vector<aten::BVHNode>>& nodes,
		const std::vector<aten::PrimitiveParamter>& prims,
		const std::vector<aten::vertex>& vtxs,
		const std::vector<aten::mat4>& mtxs,
		const std::vector<TextureResource>& texs,
		const EnvmapResource& envmapRsc)
	{
		idaten::Renderer::update(
			gltex,
			width, height,
			camera,
			shapes,
			mtrls,
			lights,
			nodes,
			prims,
			vtxs,
			mtxs,
			texs, envmapRsc);

		m_hitbools.init(width * height);
		m_hitidx.init(width * height);

		m_sobolMatrices.init(AT_COUNTOF(sobol::Matrices::matrices));
		m_sobolMatrices.writeByNum(sobol::Matrices::matrices, m_sobolMatrices.maxNum());

		auto& r = aten::getRandom();
		m_random.init(width * height);
		m_random.writeByNum(&r[0], width * height);

		for (int i = 0; i < 2; i++) {
			m_aovs[i].init(width * height);
		}

		for (int i = 0; i < 2; i++) {
			m_atrousClr[i].init(width * height);
			m_atrousVar[i].init(width * height);
		}

		m_tmpBuf.init(width * height);
	}

	static bool doneSetStackSize = false;

	void SVGFPathTracing::render(
		aten::vec4* image,
		int width, int height,
		int maxSamples,
		int maxBounce)
	{
#ifdef __AT_DEBUG__
		if (!doneSetStackSize) {
			size_t val = 0;
			cudaThreadGetLimit(&val, hipLimitStackSize);
			cudaThreadSetLimit(hipLimitStackSize, val * 4);
			doneSetStackSize = true;
		}
#endif

		int bounce = 0;

		m_paths.init(width * height);
		m_isects.init(width * height);
		m_rays.init(width * height);

		hipMemset(m_paths.ptr(), 0, m_paths.bytes());

		CudaGLResourceMap rscmap(&m_glimg);
		auto outputSurf = m_glimg.bind();

		auto vtxTexPos = m_vtxparamsPos.bind();
		auto vtxTexNml = m_vtxparamsNml.bind();

		{
			std::vector<hipTextureObject_t> tmp;
			for (int i = 0; i < m_nodeparam.size(); i++) {
				auto nodeTex = m_nodeparam[i].bind();
				tmp.push_back(nodeTex);
			}
			m_nodetex.writeByNum(&tmp[0], tmp.size());
		}

		if (!m_texRsc.empty())
		{
			std::vector<hipTextureObject_t> tmp;
			for (int i = 0; i < m_texRsc.size(); i++) {
				auto cudaTex = m_texRsc[i].bind();
				tmp.push_back(cudaTex);
			}
			m_tex.writeByNum(&tmp[0], tmp.size());
		}

		static const int rrBounce = 3;

		// Set bounce count to 1 forcibly, aov render mode.
		maxBounce = (m_mode == Mode::AOVar ? 1 : maxBounce);

		auto time = AT_NAME::timer::getSystemTime();

		for (int i = 0; i < maxSamples; i++) {
			int seed = time.milliSeconds;
			//int seed = 0;

			onGenPath(
				width, height,
				i, maxSamples,
				seed,
				vtxTexPos,
				vtxTexNml);

			bounce = 0;

			while (bounce < maxBounce) {
				onHitTest(
					width, height,
					vtxTexPos);
				
				onShadeMiss(width, height, bounce);

				int hitcount = 0;
				idaten::Compaction::compact(
					m_hitidx,
					m_hitbools,
					&hitcount);

				//AT_PRINTF("%d\n", hitcount);

				if (hitcount == 0) {
					break;
				}

				onShade(
					outputSurf,
					hitcount,
					width, height,
					bounce, rrBounce,
					vtxTexPos, vtxTexNml);

				bounce++;
			}
		}

		onGather(outputSurf, width, height, maxSamples);

		if (m_mode == Mode::SVGF)
		{
			onVarianceEstimation(outputSurf, width, height);

			onAtrousFilter(outputSurf, width, height);

			copyFromTmpBufferToAov(width, height);
		}
		else if (m_mode == Mode::VAR) {
			onVarianceEstimation(outputSurf, width, height);
		}

		pick(
			m_pickedInfo.ix, m_pickedInfo.iy, 
			width, height,
			vtxTexPos);

		checkCudaErrors(hipDeviceSynchronize());

		// Toggle aov buffer pos.
		m_curAOVPos = 1 - m_curAOVPos;

		m_frame++;

		{
			m_vtxparamsPos.unbind();
			m_vtxparamsNml.unbind();

			for (int i = 0; i < m_nodeparam.size(); i++) {
				m_nodeparam[i].unbind();
			}
			m_nodetex.reset();

			for (int i = 0; i < m_texRsc.size(); i++) {
				m_texRsc[i].unbind();
			}
			m_tex.reset();
		}
	}

	void SVGFPathTracing::onGenPath(
		int width, int height,
		int sample, int maxSamples,
		int seed,
		hipTextureObject_t texVtxPos,
		hipTextureObject_t texVtxNml)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		if (m_mode == Mode::AOVar) {
			genPath<true> << <grid, block >> > (
				m_paths.ptr(),
				m_rays.ptr(),
				width, height,
				sample, maxSamples,
				m_frame,
				m_cam.ptr(),
				m_sobolMatrices.ptr(),
				m_random.ptr());
		}
		else {
			genPath<false> << <grid, block >> > (
				m_paths.ptr(),
				m_rays.ptr(),
				width, height,
				sample, maxSamples,
				m_frame,
				m_cam.ptr(),
				m_sobolMatrices.ptr(),
				m_random.ptr());
		}

		checkCudaKernel(genPath);
	}

	void SVGFPathTracing::onHitTest(
		int width, int height,
		hipTextureObject_t texVtxPos)
	{
#if 0
		dim3 blockPerGrid_HitTest((width * height + 128 - 1) / 128);
		dim3 threadPerBlock_HitTest(128);

		hitTest << <blockPerGrid_HitTest, threadPerBlock_HitTest >> > (
#else
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		hitTest << <grid, block >> > (
#endif
		//hitTest << <1, 1 >> > (
			m_paths.ptr(),
			m_isects.ptr(),
			m_rays.ptr(),
			m_hitbools.ptr(),
			width, height,
			m_shapeparam.ptr(), m_shapeparam.num(),
			m_lightparam.ptr(), m_lightparam.num(),
			m_nodetex.ptr(),
			m_primparams.ptr(),
			texVtxPos,
			m_mtxparams.ptr());

		checkCudaKernel(hitTest);
	}

	void SVGFPathTracing::onShadeMiss(
		int width, int height,
		int bounce)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();

		if (m_envmapRsc.idx >= 0) {
			if (bounce == 0) {
				shadeMissWithEnvmap<true> << <grid, block >> > (
					curaov.ptr(),
					m_tex.ptr(),
					m_envmapRsc.idx, m_envmapRsc.avgIllum, m_envmapRsc.multiplyer,
					m_paths.ptr(),
					m_rays.ptr(),
					width, height);
			}
			else {
				shadeMissWithEnvmap<false> << <grid, block >> > (
					curaov.ptr(),
					m_tex.ptr(),
					m_envmapRsc.idx, m_envmapRsc.avgIllum, m_envmapRsc.multiplyer,
					m_paths.ptr(),
					m_rays.ptr(),
					width, height);
			}
		}
		else {
			if (bounce == 0) {
				shadeMiss<true> << <grid, block >> > (
					curaov.ptr(),
					m_paths.ptr(),
					width, height);
			}
			else {
				shadeMiss<false> << <grid, block >> > (
					curaov.ptr(),
					m_paths.ptr(),
					width, height);
			}
		}

		checkCudaKernel(shadeMiss);
	}

	void SVGFPathTracing::onShade(
		hipSurfaceObject_t outputSurf,
		int hitcount,
		int width, int height,
		int bounce, int rrBounce,
		hipTextureObject_t texVtxPos,
		hipTextureObject_t texVtxNml)
	{
		aten::mat4 mtxW2V;
		mtxW2V.lookat(
			m_camParam.origin,
			m_camParam.center,
			m_camParam.up);

		m_mtxV2C.perspective(
			m_camParam.znear,
			m_camParam.zfar,
			m_camParam.vfov,
			m_camParam.aspect);

		m_mtxC2V = m_mtxV2C;
		m_mtxC2V.invert();

		aten::mat4 mtxW2C = m_mtxV2C * mtxW2V;

#ifdef ENABLE_DEBUG_1PIXEL
		int blockPerGrid = 1;
		int threadPerBlock = 1;
#else
		dim3 blockPerGrid((hitcount + 64 - 1) / 64);
		dim3 threadPerBlock(64);
#endif

		auto& curaov = getCurAovs();

		if (bounce == 0) {
			shade<true> << <blockPerGrid, threadPerBlock >> > (
				curaov.ptr(), mtxW2C,
				width, height,
				m_paths.ptr(),
				m_hitidx.ptr(), hitcount,
				m_isects.ptr(),
				m_rays.ptr(),
				m_frame,
				bounce, rrBounce,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				texVtxPos, texVtxNml,
				m_mtxparams.ptr(),
				m_tex.ptr(),
				m_random.ptr());
		}
		else {
			shade<false> << <blockPerGrid, threadPerBlock >> > (
				curaov.ptr(), mtxW2C,
				width, height,
				m_paths.ptr(),
				m_hitidx.ptr(), hitcount,
				m_isects.ptr(),
				m_rays.ptr(),
				m_frame,
				bounce, rrBounce,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				texVtxPos, texVtxNml,
				m_mtxparams.ptr(),
				m_tex.ptr(),
				m_random.ptr());
		}

		checkCudaKernel(shade);
	}

	void SVGFPathTracing::onGather(
		hipSurfaceObject_t outputSurf,
		int width, int height,
		int maxSamples)
	{
#ifdef ENABLE_DEBUG_1PIXEL
		int block = 1;
		int grid = 1;
#else
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);
#endif

		if (m_mode == Mode::PT) {
			auto& curaov = getCurAovs();

			gather << <grid, block >> > (
				outputSurf,
				curaov.ptr(),
				m_paths.ptr(),
				width, height);

			checkCudaKernel(gather);
		}
		else if (m_mode == Mode::AOVar) {
			onFillAOV(outputSurf, width, height);
		}
		else {
			auto& curaov = getCurAovs();
			auto& prevaov = getPrevAovs();

			if (isFirstFrame()) {
				gather << <grid, block >> > (
					outputSurf,
					curaov.ptr(),
					m_paths.ptr(),
					width, height);

				checkCudaKernel(gather);
			}
			else {
				onTemporalReprojection(
					outputSurf,
					width, height);
			}
		}

		m_mtxPrevV2C = m_mtxV2C;
	}
}
