#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

#define ENABLE_PERSISTENT_THREAD
#define SEPARATE_SHADOWRAY_HITTEST

//#define ENABLE_DEBUG_1PIXEL

#ifdef ENABLE_DEBUG_1PIXEL
#define DEBUG_IX	(140)
#define DEBUG_IY	(511 - 81)
#endif

template <bool isFillAOV>
__global__ void genPath(
	idaten::SVGFPathTracing::Path* paths,
	aten::ray* rays,
	int width, int height,
	int sample, int maxSamples,
	unsigned int frame,
	const aten::CameraParameter* __restrict__ camera,
	const unsigned int* sobolmatrices,
	unsigned int* random)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isKill) {
		path.isTerminate = true;
		return;
	}

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
	auto scramble = random[idx] * 0x1fe3434f;
	path.sampler.init(frame, 0, scramble, sobolmatrices);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
	auto rnd = random[idx];
	auto scramble = rnd * 0x1fe3434f * ((frame + 133 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
	path.sampler.init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 0, scramble);
#endif

	float r1 = path.sampler.nextSample();
	float r2 = path.sampler.nextSample();

	if (isFillAOV) {
		r1 = r2 = 0.5f;
	}

	float s = (ix + r1) / (float)(camera->width);
	float t = (iy + r2) / (float)(camera->height);

	AT_NAME::CameraSampleResult camsample;
	AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

	rays[idx] = camsample.r;

	path.throughput = aten::vec3(1);
	path.pdfb = 0.0f;
	path.isTerminate = false;
	path.isSingular = false;

	path.samples += 1;

	// Accumulate value, so do not reset.
	//path.contrib = aten::vec3(0);
}

// NOTE
// persistent thread.
// https://gist.github.com/guozhou/b972bb42bbc5cba1f062#file-persistent-cpp-L15

// NOTE
// compute capability 6.0
// http://homepages.math.uic.edu/~jan/mcs572/performance_considerations.pdf
// p3

#define NUM_SM				64	// no. of streaming multiprocessors
#define NUM_WARP_PER_SM		64	// maximum no. of resident warps per SM
#define NUM_BLOCK_PER_SM	32	// maximum no. of resident blocks per SM
#define NUM_BLOCK			(NUM_SM * NUM_BLOCK_PER_SM)
#define NUM_WARP_PER_BLOCK	(NUM_WARP_PER_SM / NUM_BLOCK_PER_SM)
#define WARP_SIZE			32

__device__ unsigned int g_headDev = 0;

__global__ void hitTest(
	idaten::SVGFPathTracing::Path* paths,
	aten::Intersection* isects,
	aten::ray* rays,
	int* hitbools,
	int width, int height,
	const aten::GeomParameter* __restrict__ shapes, int geomnum,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	aten::mat4* matrices,
	int bounce,
	float hitDistLimit)
{
#ifdef ENABLE_PERSISTENT_THREAD
	// warp-wise head index of tasks in a block
	__shared__ volatile unsigned int headBlock[NUM_WARP_PER_BLOCK];

	volatile unsigned int& headWarp = headBlock[threadIdx.y];

	if (blockIdx.x == 0 && threadIdx.x == 0) {
		g_headDev = 0;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	do
	{
		// let lane 0 fetch [wh, wh + WARP_SIZE - 1] for a warp
		if (threadIdx.x == 0) {
			headWarp = atomicAdd(&g_headDev, WARP_SIZE);
		}
		// task index per thread in a warp
		unsigned int idx = headWarp + threadIdx.x;

		if (idx >= width * height) {
			return;
		}

		auto& path = paths[idx];
		path.isHit = false;

		hitbools[idx] = 0;

		if (path.isTerminate) {
			continue;
		}

		aten::Intersection isect;

		float t_max = AT_MATH_INF;

		if (bounce >= 1
			&& !path.isSingular)
		{
			t_max = hitDistLimit;
		}

		bool isHit = intersectClosest(&ctxt, rays[idx], &isect, t_max);

		isects[idx].t = isect.t;
		isects[idx].objid = isect.objid;
		isects[idx].mtrlid = isect.mtrlid;
		isects[idx].meshid = isect.meshid;
		isects[idx].primid = isect.primid;
		isects[idx].a = isect.a;
		isects[idx].b = isect.b;

		if (bounce >= 1
			&& !path.isSingular
			&& isect.t > hitDistLimit)
		{
			isHit = false;
		}

		path.isHit = isHit;

		hitbools[idx] = isHit ? 1 : 0;
	} while (true);
#else
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.isHit = false;

	hitbools[idx] = 0;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	aten::Intersection isect;

	float t_max = AT_MATH_INF;

	if (bounce >= 1
		&& !path.isSingular)
	{
		t_max = hitDistLimit;
	}

	bool isHit = intersectClosest(&ctxt, rays[idx], &isect, t_max);

	isects[idx].t = isect.t;
	isects[idx].objid = isect.objid;
	isects[idx].mtrlid = isect.mtrlid;
	isects[idx].meshid = isect.meshid;
	isects[idx].area = isect.area;
	isects[idx].primid = isect.primid;
	isects[idx].a = isect.a;
	isects[idx].b = isect.b;

	if (bounce >= 1
		&& !path.isSingular
		&& isect.t > hitDistLimit)
	{
		isHit = false;
	}

	path.isHit = isHit;

	hitbools[idx] = isHit ? 1 : 0;
#endif
}

template <bool isFirstBounce>
__global__ void shadeMiss(
	hipSurfaceObject_t aovExportBuffer,
	float4* aovNormalDepth,
	float4* aovTexclrTemporalWeight,
	float4* aovMomentMeshid,
	idaten::SVGFPathTracing::Path* paths,
	int width, int height)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];

	if (!path.isTerminate && !path.isHit) {
		// TODO
		auto bg = aten::vec3(0);

		if (isFirstBounce) {
			path.isKill = true;

			// Export bg color to albedo buffer.
			aovTexclrTemporalWeight[idx] = make_float4(bg.x, bg.y, bg.z, aovTexclrTemporalWeight[idx].w);
			aovNormalDepth[idx].w = -1;
			aovMomentMeshid[idx].w = -1;

			// For exporting separated albedo.
			bg = aten::vec3(1, 1, 1);

			if (aovExportBuffer > 0) {
				surf2Dwrite(
					aovNormalDepth[idx],
					aovExportBuffer,
					ix * sizeof(float4), iy,
					hipBoundaryModeTrap);
			}
		}

		path.contrib += path.throughput * bg;

		path.isTerminate = true;
	}
}

template <bool isFirstBounce>
__global__ void shadeMissWithEnvmap(
	hipSurfaceObject_t aovExportBuffer,
	float4* aovNormalDepth,
	float4* aovTexclrTemporalWeight,
	float4* aovMomentMeshid,
	hipTextureObject_t* textures,
	int envmapIdx,
	real envmapAvgIllum,
	real envmapMultiplyer,
	idaten::SVGFPathTracing::Path* paths,
	const aten::ray* __restrict__ rays,
	int width, int height)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];

	if (!path.isTerminate && !path.isHit) {
		auto r = rays[idx];

		auto uv = AT_NAME::envmap::convertDirectionToUV(r.dir);

		auto bg = tex2D<float4>(textures[envmapIdx], uv.x, uv.y);
		auto emit = aten::vec3(bg.x, bg.y, bg.z);

		float misW = 1.0f;
		if (isFirstBounce) {
			path.isKill = true;

			// Export envmap to albedo buffer.
			aovTexclrTemporalWeight[idx] = make_float4(emit.x, emit.y, emit.z, aovTexclrTemporalWeight[idx].w);
			aovNormalDepth[idx].w = -1;
			aovMomentMeshid[idx].w = -1;

			// For exporting separated albedo.
			emit = aten::vec3(1, 1, 1);

			if (aovExportBuffer > 0) {
				surf2Dwrite(
					aovNormalDepth[idx],
					aovExportBuffer,
					ix * sizeof(float4), iy,
					hipBoundaryModeTrap);
			}
		}
		else {
			auto pdfLight = AT_NAME::ImageBasedLight::samplePdf(emit, envmapAvgIllum);
			misW = path.pdfb / (pdfLight + path.pdfb);

			emit *= envmapMultiplyer;
		}

		path.contrib += path.throughput * misW * emit;

		path.isTerminate = true;
	}
}

template <bool isFirstBounce, int ShadowRayNum>
__global__ void shade(
	float4* aovNormalDepth,
	float4* aovTexclrTemporalWeight,
	float4* aovMomentMeshid,
	hipSurfaceObject_t aovExportBuffer,
	aten::mat4 mtxW2C,
	int width, int height,
	idaten::SVGFPathTracing::Path* paths,
	const int* __restrict__ hitindices,
	int hitnum,
	const aten::Intersection* __restrict__ isects,
	aten::ray* rays,
	int frame,
	int bounce, int rrBounce,
	const aten::GeomParameter* __restrict__ shapes, int geomnum,
	const aten::MaterialParameter* __restrict__ mtrls,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	hipTextureObject_t vtxNml,
	const aten::mat4* __restrict__ matrices,
	hipTextureObject_t* textures,
	unsigned int* random,
	idaten::SVGFPathTracing::ShadowRay* shadowRays)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= hitnum) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.vtxNml = vtxNml;
		ctxt.matrices = matrices;
		ctxt.textures = textures;
	}

	idx = hitindices[idx];

#ifdef ENABLE_DEBUG_1PIXEL
	int ix = DEBUG_IX;
	int iy = DEBUG_IY;
	idx = getIdx(ix, iy, width);
#endif

	__shared__ idaten::SVGFPathTracing::Path shPaths[64];
	__shared__ idaten::SVGFPathTracing::ShadowRay shShadowRays[64];
	__shared__ aten::MaterialParameter shMtrls[64];

	shPaths[threadIdx.x] = paths[idx];

	const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
	auto scramble = random[idx] * 0x1fe3434f;
	shPaths[threadIdx.x].sampler.init(frame, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
	auto rnd = random[idx];
	auto scramble = rnd * 0x1fe3434f * ((frame + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
	shPaths[threadIdx.x].sampler.init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 4 + bounce * 300, scramble);
#endif

	aten::hitrecord rec;

	const auto& isect = isects[idx];

	auto obj = &ctxt.shapes[isect.objid];
	evalHitResult(&ctxt, obj, ray, &rec, &isect);

	shMtrls[threadIdx.x] = ctxt.mtrls[rec.mtrlid];

	bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

	// �����ʒu�̖@��.
	// ���̂���̃��C�̓��o���l��.
	aten::vec3 orienting_normal = rec.normal;

	if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
		shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
		shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
		shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
	}

	// Render AOVs.
	// NOTE
	// �����ɖ@����AOV�ɕێ�����Ȃ�A�@���}�b�v�K�p�シ��ׂ�.
	// �������Atemporal reprojection�Aatrous�Ȃǂ̃t�B���^�K�p���ɖ@�����Q�Ƃ���ۂɁA�@���}�b�v���ׂ������Ă͂�����Ă��܂����Ƃ�����.
	// ����ɂ��A�t�B���^�����������悤�ɂ����炸�t�B���^�̕i�����������Ă��܂���肪��������.
	if (isFirstBounce) {
		int ix = idx % width;
		int iy = idx / width;

		// World coordinate to Clip coordinate.
		aten::vec4 pos = aten::vec4(rec.p, 1);
		pos = mtxW2C.apply(pos);

		// normal, depth
		aovNormalDepth[idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);

		// meshid.
		aovMomentMeshid[idx].w = isect.meshid;

		// texture color.
		auto texcolor = AT_NAME::material::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, 1.0f);
		aovTexclrTemporalWeight[idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, aovTexclrTemporalWeight[idx].w);

		// For exporting separated albedo.
		shMtrls[threadIdx.x].albedoMap = -1;

		if (aovExportBuffer > 0) {
			surf2Dwrite(
				aovNormalDepth[idx],
				aovExportBuffer,
				ix * sizeof(float4), iy,
				hipBoundaryModeTrap);
		}
	}

	// Implicit conection to light.
	if (shMtrls[threadIdx.x].attrib.isEmissive) {
		if (!isBackfacing) {
			float weight = 1.0f;

			if (bounce > 0 && !shPaths[threadIdx.x].isSingular) {
				auto cosLight = dot(orienting_normal, -ray.dir);
				auto dist2 = aten::squared_length(rec.p - ray.org);

				if (cosLight >= 0) {
					auto pdfLight = 1 / rec.area;

					// Convert pdf area to sradian.
					// http://www.slideshare.net/h013/edubpt-v100
					// p31 - p35
					pdfLight = pdfLight * dist2 / cosLight;

					weight = shPaths[threadIdx.x].pdfb / (pdfLight + shPaths[threadIdx.x].pdfb);
				}
			}

			shPaths[threadIdx.x].contrib += shPaths[threadIdx.x].throughput * weight * shMtrls[threadIdx.x].baseColor;
		}

		// When ray hit the light, tracing will finish.
		shPaths[threadIdx.x].isTerminate = true;
		paths[idx] = shPaths[threadIdx.x];
		return;
	}

	if (!shMtrls[threadIdx.x].attrib.isTranslucent && isBackfacing) {
		orienting_normal = -orienting_normal;
	}

	// Apply normal map.
	int normalMap = shMtrls[threadIdx.x].normalMap;
	if (shMtrls[threadIdx.x].type == aten::MaterialType::Layer) {
		// �ŕ\�w�� NormalMap ��K�p.
		auto* topmtrl = &ctxt.mtrls[shMtrls[threadIdx.x].layer[0]];
		normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
	}
	AT_NAME::material::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

#ifdef SEPARATE_SHADOWRAY_HITTEST
	shShadowRays[threadIdx.x].isActive = false;
#endif

	auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1));

#if 1
	// Explicit conection to light.
	if (!shMtrls[threadIdx.x].attrib.isSingular)
	{
		for (int i = 0; i < ShadowRayNum; i++) {
			real lightSelectPdf = 1;
			aten::LightSampleResult sampleres;

			// TODO
			// Importance sampling.
			int lightidx = aten::cmpMin<int>(shPaths[threadIdx.x].sampler.nextSample() * lightnum, lightnum - 1);
			lightSelectPdf = 1.0f / lightnum;

			aten::LightParameter light;
			light.pos = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 0];
			light.dir = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 1];
			light.le = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 2];
			light.v0 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 3];
			light.v1 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 4];
			light.v2 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 5];
			//auto light = ctxt.lights[lightidx];

			sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &shPaths[threadIdx.x].sampler);

			const auto& posLight = sampleres.pos;
			const auto& nmlLight = sampleres.nml;
			real pdfLight = sampleres.pdf;

			auto dirToLight = normalize(sampleres.dir);
			auto distToLight = length(posLight - rec.p);

			auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;
			auto tmp = rec.p + dirToLight - shadowRayOrg;
			auto shadowRayDir = normalize(tmp);

#ifdef SEPARATE_SHADOWRAY_HITTEST
			shShadowRays[threadIdx.x].isActive = true;
			shShadowRays[threadIdx.x].ray[i] = aten::ray(shadowRayOrg, shadowRayDir);
			shShadowRays[threadIdx.x].targetLightId[i] = lightidx;
			shShadowRays[threadIdx.x].distToLight[i] = distToLight;
			shShadowRays[threadIdx.x].lightcontrib[i] = aten::vec3(0);
#else
			auto lightobj = sampleres.obj;

			real distHitObjToRayOrg = AT_MATH_INF;

			// Ray aim to the area light.
			// So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
			auto hitobj = lightobj;

			aten::Intersection isectTmp;

			aten::ray shadowRay(shadowRayOrg, shadowRayDir);

			bool isHit = intersectCloser(&ctxt, shadowRay, &isectTmp, distToLight - AT_MATH_EPSILON);

			if (isHit) {
				hitobj = (void*)&ctxt.shapes[isectTmp.objid];
			}

			isHit = AT_NAME::scene::hitLight(
				isHit,
				light.attrib,
				lightobj,
				distToLight,
				distHitObjToRayOrg,
				isectTmp.t,
				hitobj);

			if (isHit)
#endif
			{
				auto cosShadow = dot(orienting_normal, dirToLight);

				real pdfb = samplePDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
				auto bsdf = sampleBSDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v, albedo);

				bsdf *= shPaths[threadIdx.x].throughput;

				// Get light color.
				auto emit = sampleres.finalColor;

				if (light.attrib.isSingular || light.attrib.isInfinite) {
					if (pdfLight > real(0) && cosShadow >= 0) {
						// TODO
						// �W�I���g���^�[���̈����ɂ���.
						// singular light �̏ꍇ�́AfinalColor �ɋ����̏��Z���܂܂�Ă���.
						// inifinite light �̏ꍇ�́A���������ɂȂ�ApdfLight�Ɋ܂܂�鋗�������Ƒł����������H.
						// �i�ł����������̂ŁApdfLight�ɂ͋��������͊܂�ł��Ȃ��j.
						auto misW = pdfLight / (pdfb + pdfLight);
#ifdef SEPARATE_SHADOWRAY_HITTEST
						shShadowRays[threadIdx.x].lightcontrib[i] =
#else
						shPaths[threadIdx.x].contrib +=
#endif
							(misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf / (float)ShadowRayNum;
					}
				}
				else {
					auto cosLight = dot(nmlLight, -dirToLight);

					if (cosShadow >= 0 && cosLight >= 0) {
						auto dist2 = aten::squared_length(sampleres.dir);
						auto G = cosShadow * cosLight / dist2;

						if (pdfb > real(0) && pdfLight > real(0)) {
							// Convert pdf from steradian to area.
							// http://www.slideshare.net/h013/edubpt-v100
							// p31 - p35
							pdfb = pdfb * cosLight / dist2;

							auto misW = pdfLight / (pdfb + pdfLight);
#ifdef SEPARATE_SHADOWRAY_HITTEST
							shShadowRays[threadIdx.x].lightcontrib[i] =
#else
							shPaths[threadIdx.x].contrib +=
#endif
								(misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf / (float)ShadowRayNum;
						}
					}
				}
			}
		}
	}
#endif

	real russianProb = real(1);

	if (bounce > rrBounce) {
		auto t = normalize(shPaths[threadIdx.x].throughput);
		auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

		russianProb = shPaths[threadIdx.x].sampler.nextSample();

		if (russianProb >= p) {
			//shPaths[threadIdx.x].contrib = aten::vec3(0);
			shPaths[threadIdx.x].isTerminate = true;
		}
		else {
			russianProb = p;
		}
	}
			
	AT_NAME::MaterialSampling sampling;

	sampleMaterial(
		&sampling,
		&ctxt,
		&shMtrls[threadIdx.x],
		orienting_normal,
		ray.dir,
		rec.normal,
		&shPaths[threadIdx.x].sampler,
		rec.u, rec.v,
		albedo);

	auto nextDir = normalize(sampling.dir);
	auto pdfb = sampling.pdf;
	auto bsdf = sampling.bsdf;

	real c = 1;
	if (!shMtrls[threadIdx.x].attrib.isSingular) {
		// TODO
		// AMD�̂�abs���Ă��邪....
		//c = aten::abs(dot(orienting_normal, nextDir));
		c = dot(orienting_normal, nextDir);
	}

	if (pdfb > 0 && c > 0) {
		shPaths[threadIdx.x].throughput *= bsdf * c / pdfb;
		shPaths[threadIdx.x].throughput /= russianProb;
	}
	else {
		shPaths[threadIdx.x].isTerminate = true;
	}

	// Make next ray.
	rays[idx] = aten::ray(rec.p, nextDir);

	shPaths[threadIdx.x].pdfb = pdfb;
	shPaths[threadIdx.x].isSingular = shMtrls[threadIdx.x].attrib.isSingular;

	paths[idx] = shPaths[threadIdx.x];
	shadowRays[idx] = shShadowRays[threadIdx.x];
}

template <int ShadowRayNum>
__global__ void hitShadowRay(
	idaten::SVGFPathTracing::Path* paths,
	int* hitindices,
	int hitnum,
	const idaten::SVGFPathTracing::ShadowRay* __restrict__ shadowRays,
	const aten::GeomParameter* __restrict__ shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	const aten::mat4* __restrict__ matrices)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= hitnum) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	idx = hitindices[idx];

	auto& shadowRay = shadowRays[idx];

	if (shadowRay.isActive) {
#pragma unroll
		for (int i = 0; i < ShadowRayNum; i++) {
			auto targetLightId = shadowRay.targetLightId[i];
			auto distToLight = shadowRay.distToLight[i];

			auto light = &ctxt.lights[targetLightId];
			auto lightobj = (light->objid >= 0 ? &ctxt.shapes[light->objid] : nullptr);

			real distHitObjToRayOrg = AT_MATH_INF;

			// Ray aim to the area light.
			// So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
			const aten::GeomParameter* hitobj = lightobj;

			aten::Intersection isectTmp;

			bool isHit = false;
			isHit = intersectCloser(&ctxt, shadowRay.ray[i], &isectTmp, distToLight - AT_MATH_EPSILON);

			if (isHit) {
				hitobj = &ctxt.shapes[isectTmp.objid];
			}

			isHit = AT_NAME::scene::hitLight(
				isHit,
				light->attrib,
				lightobj,
				distToLight,
				distHitObjToRayOrg,
				isectTmp.t,
				hitobj);

			if (isHit) {
				paths[idx].contrib += shadowRay.lightcontrib[i];
			}
		}
	}
}

__global__ void gather(
	hipSurfaceObject_t dst,
	float4* aovColorVariance,
	float4* aovMomentMeshid,
	const idaten::SVGFPathTracing::Path* __restrict__ paths,
	int width, int height)
{
	auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

#ifdef ENABLE_DEBUG_1PIXEL
	ix = DEBUG_IX;
	iy = DEBUG_IY;
#endif

	const auto idx = getIdx(ix, iy, width);

	const auto& path = paths[idx];

	int sample = path.samples;

	float3 contrib = make_float3(path.contrib.x, path.contrib.y, path.contrib.z) / sample;
	//contrib.w = sample;

	float lum = AT_NAME::color::luminance(contrib.x, contrib.y, contrib.z);

	aovMomentMeshid[idx].x += lum * lum;
	aovMomentMeshid[idx].y += lum;
	aovMomentMeshid[idx].z += 1;

	aovColorVariance[idx] = make_float4(contrib.x, contrib.y, contrib.z, aovColorVariance[idx].w);

#if 0
	auto n = aovs[idx].moments.w;

	auto m = aovs[idx].moments / n;

	auto var = m.x - m.y * m.y;

	surf2Dwrite(
		make_float4(var, var, var, 1),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
#else
	surf2Dwrite(
		make_float4(contrib, 0),
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
#endif
}

namespace idaten
{
	void SVGFPathTracing::onGenPath(
		int width, int height,
		int sample, int maxSamples,
		int seed,
		hipTextureObject_t texVtxPos,
		hipTextureObject_t texVtxNml)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		if (m_mode == Mode::AOVar) {
			genPath<true> << <grid, block >> > (
				m_paths.ptr(),
				m_rays.ptr(),
				width, height,
				sample, maxSamples,
				m_frame,
				m_cam.ptr(),
				m_sobolMatrices.ptr(),
				m_random.ptr());
		}
		else {
			genPath<false> << <grid, block >> > (
				m_paths.ptr(),
				m_rays.ptr(),
				width, height,
				sample, maxSamples,
				m_frame,
				m_cam.ptr(),
				m_sobolMatrices.ptr(),
				m_random.ptr());
		}

		checkCudaKernel(genPath);
	}

	void SVGFPathTracing::onHitTest(
		int width, int height,
		int bounce,
		hipTextureObject_t texVtxPos)
	{
		if (bounce == 0) {
			onScreenSpaceHitTest(width, height, bounce, texVtxPos);
		}
		else {
#ifdef ENABLE_PERSISTENT_THREAD
			hitTest << <NUM_BLOCK, dim3(WARP_SIZE, NUM_WARP_PER_BLOCK) >> > (
#else
			dim3 block(BLOCK_SIZE, BLOCK_SIZE);
			dim3 grid(
				(width + block.x - 1) / block.x,
				(height + block.y - 1) / block.y);

			hitTest << <grid, block >> > (
#endif
				//hitTest << <1, 1 >> > (
				m_paths.ptr(),
				m_isects.ptr(),
				m_rays.ptr(),
				m_hitbools.ptr(),
				width, height,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				texVtxPos,
				m_mtxparams.ptr(),
				bounce,
				m_hitDistLimit);

			checkCudaKernel(hitTest);
		}
	}

	void SVGFPathTracing::onShadeMiss(
		int width, int height,
		int bounce,
		hipSurfaceObject_t aovExportBuffer)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int curaov = getCurAovs();

		if (m_envmapRsc.idx >= 0) {
			if (bounce == 0) {
				shadeMissWithEnvmap<true> << <grid, block >> > (
					aovExportBuffer,
					m_aovNormalDepth[curaov].ptr(),
					m_aovTexclrTemporalWeight[curaov].ptr(),
					m_aovMomentMeshid[curaov].ptr(),
					m_tex.ptr(),
					m_envmapRsc.idx, m_envmapRsc.avgIllum, m_envmapRsc.multiplyer,
					m_paths.ptr(),
					m_rays.ptr(),
					width, height);
			}
			else {
				shadeMissWithEnvmap<false> << <grid, block >> > (
					aovExportBuffer,
					m_aovNormalDepth[curaov].ptr(),
					m_aovTexclrTemporalWeight[curaov].ptr(),
					m_aovMomentMeshid[curaov].ptr(),
					m_tex.ptr(),
					m_envmapRsc.idx, m_envmapRsc.avgIllum, m_envmapRsc.multiplyer,
					m_paths.ptr(),
					m_rays.ptr(),
					width, height);
			}
		}
		else {
			if (bounce == 0) {
				shadeMiss<true> << <grid, block >> > (
					aovExportBuffer,
					m_aovNormalDepth[curaov].ptr(),
					m_aovTexclrTemporalWeight[curaov].ptr(),
					m_aovMomentMeshid[curaov].ptr(),
					m_paths.ptr(),
					width, height);
			}
			else {
				shadeMiss<false> << <grid, block >> > (
					aovExportBuffer,
					m_aovNormalDepth[curaov].ptr(),
					m_aovTexclrTemporalWeight[curaov].ptr(),
					m_aovMomentMeshid[curaov].ptr(),
					m_paths.ptr(),
					width, height);
			}
		}

		checkCudaKernel(shadeMiss);
	}

	void SVGFPathTracing::onShade(
		hipSurfaceObject_t outputSurf,
		hipSurfaceObject_t aovExportBuffer,
		int hitcount,
		int width, int height,
		int bounce, int rrBounce,
		hipTextureObject_t texVtxPos,
		hipTextureObject_t texVtxNml)
	{
		m_mtxW2V.lookat(
			m_camParam.origin,
			m_camParam.center,
			m_camParam.up);

		m_mtxV2C.perspective(
			m_camParam.znear,
			m_camParam.zfar,
			m_camParam.vfov,
			m_camParam.aspect);

		m_mtxC2V = m_mtxV2C;
		m_mtxC2V.invert();

		m_mtxV2W = m_mtxW2V;
		m_mtxV2W.invert();

		aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

#ifdef ENABLE_DEBUG_1PIXEL
		int blockPerGrid = 1;
		int threadPerBlock = 1;
#else
		dim3 blockPerGrid((hitcount + 64 - 1) / 64);
		dim3 threadPerBlock(64);
#endif

		int curaov = getCurAovs();

		if (bounce == 0) {
			shade<true, ShdowRayNum> << <blockPerGrid, threadPerBlock >> > (
				m_aovNormalDepth[curaov].ptr(),
				m_aovTexclrTemporalWeight[curaov].ptr(),
				m_aovMomentMeshid[curaov].ptr(),
				aovExportBuffer,
				mtxW2C,
				width, height,
				m_paths.ptr(),
				m_hitidx.ptr(), hitcount,
				m_isects.ptr(),
				m_rays.ptr(),
				m_frame,
				bounce, rrBounce,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				texVtxPos, texVtxNml,
				m_mtxparams.ptr(),
				m_tex.ptr(),
				m_random.ptr(),
				m_shadowRays.ptr());
		}
		else {
			shade<false, ShdowRayNum> << <blockPerGrid, threadPerBlock >> > (
				m_aovNormalDepth[curaov].ptr(),
				m_aovTexclrTemporalWeight[curaov].ptr(),
				m_aovMomentMeshid[curaov].ptr(),
				aovExportBuffer,
				mtxW2C,
				width, height,
				m_paths.ptr(),
				m_hitidx.ptr(), hitcount,
				m_isects.ptr(),
				m_rays.ptr(),
				m_frame,
				bounce, rrBounce,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				texVtxPos, texVtxNml,
				m_mtxparams.ptr(),
				m_tex.ptr(),
				m_random.ptr(),
				m_shadowRays.ptr());
		}

		checkCudaKernel(shade);

#ifdef SEPARATE_SHADOWRAY_HITTEST
		hitShadowRay<ShdowRayNum> << <blockPerGrid, threadPerBlock >> > (
			m_paths.ptr(),
			m_hitidx.ptr(), hitcount,
			m_shadowRays.ptr(),
			m_shapeparam.ptr(), m_shapeparam.num(),
			m_mtrlparam.ptr(),
			m_lightparam.ptr(), m_lightparam.num(),
			m_nodetex.ptr(),
			m_primparams.ptr(),
			texVtxPos,
			m_mtxparams.ptr());

		checkCudaKernel(hitShadowRay);
#endif
	}

	void SVGFPathTracing::onGather(
		hipSurfaceObject_t outputSurf,
		int width, int height,
		int maxSamples)
	{
#ifdef ENABLE_DEBUG_1PIXEL
		int block = 1;
		int grid = 1;
#else
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);
#endif
		int curaov = getCurAovs();

		if (m_mode == Mode::PT) {
			gather << <grid, block >> > (
				outputSurf,
				m_aovColorVariance[curaov].ptr(),
				m_aovMomentMeshid[curaov].ptr(),
				m_paths.ptr(),
				width, height);

			checkCudaKernel(gather);
		}
		else if (m_mode == Mode::AOVar) {
			onFillAOV(outputSurf, width, height);
		}
		else {
			if (isFirstFrame()) {
				gather << <grid, block >> > (
					outputSurf,
					m_aovColorVariance[curaov].ptr(),
					m_aovMomentMeshid[curaov].ptr(),
					m_paths.ptr(),
					width, height);

				checkCudaKernel(gather);
			}
			else {
				onTemporalReprojection(
					outputSurf,
					width, height);
			}
		}

		m_mtxPrevW2V = m_mtxW2V;
	}
}
