#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void checkIfSingular(
	idaten::SVGFPathTracing::Path* paths,
	int width, int height)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width || iy >= height) {
		return;
	}

	int idx = getIdx(ix, iy, width);

	if (!paths[idx].isSingular) {
		paths[idx].isKill = true;
		paths[idx].isTerminate = true;
	}
}

__global__ void coarseBuffers(
	const idaten::SVGFPathTracing::Path* __restrict__ srcPaths,
	const aten::ray* __restrict__ srcRays,
	const float4* __restrict__ srcAovNormalDepth,
	const float4* __restrict__ srcAovMomentMeshid,
	idaten::SVGFPathTracing::Path* dstPaths,
	aten::ray* dstRays,
	float4* dstAovNormalDepth,
	float4* dstAovMomentMeshid,
	int width, int height,
	int lowResWidth, int lowResHeight)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= lowResWidth || iy >= lowResHeight) {
		return;
	}

	int hiResX = ix * 2;
	int hiResY = iy * 2;

	int idx_0 = getIdx(hiResX + 0, hiResY + 0, width);
	int idx_1 = getIdx(hiResX + 1, hiResY + 0, width);
	int idx_2 = getIdx(hiResX + 0, hiResY + 1, width);
	int idx_3 = getIdx(hiResX + 1, hiResY + 1, width);

	const idaten::SVGFPathTracing::Path paths[4] = {
		srcPaths[idx_0],
		srcPaths[idx_1],
		srcPaths[idx_2],
		srcPaths[idx_3],
	};
	float4 nmlDepth[4] = {
		srcAovNormalDepth[idx_0],
		srcAovNormalDepth[idx_1],
		srcAovNormalDepth[idx_2],
		srcAovNormalDepth[idx_3],
	};
	int indices[4] = {
		idx_0, idx_1, idx_2, idx_3,
	};

	// Depth����ԑ傫�����̂�I��.
	float maxDepth = -1.0f;
	int pos = -1;

#pragma unroll
	for (int i = 0; i < 4; i++) {
		if (nmlDepth[i].w > maxDepth
			&& !paths[i].isSingular
			&& !paths[i].isKill
			&& !paths[i].isTerminate)
		{
			maxDepth = nmlDepth[i].w;
			pos = i;
		}
	}

	int idx = getIdx(ix, iy, lowResWidth);

	if (pos >= 0) {
		dstPaths[idx] = paths[pos];
		dstAovNormalDepth[idx] = nmlDepth[pos];

		int srcIdx = indices[pos];

		dstRays[idx] = srcRays[srcIdx];
		dstAovMomentMeshid[idx] = srcAovMomentMeshid[srcIdx];
	}
	else {
		int srcIdx = getIdx(hiResX, hiResY, width);

		dstPaths[idx] = srcPaths[srcIdx];
		dstAovNormalDepth[idx] = srcAovNormalDepth[srcIdx];
		dstRays[idx] = srcRays[srcIdx];
		dstAovMomentMeshid[idx] = srcAovMomentMeshid[srcIdx];

		dstPaths[idx].isKill = true;
		dstPaths[idx].isTerminate = true;
	}

	// Reset contribution.
	dstPaths[idx].contrib = aten::vec3(0.0f);
}

inline __device__ float4 samplePoint(
	const float4* __restrict__ buffer,
	int w, int h,
	int x, int y,
	int offsetx, int offsety)
{
	x = clamp(x + offsetx, 0, w - 1);
	y = clamp(y + offsety, 0, h - 1);

	int idx = getIdx(x, y, w);

	return buffer[idx];
}

inline __device__ float4 sampleBilinear(
	const float4* __restrict__ buffer,
	int w, int h,
	int x, int y,
	int offsetx, int offsety)
{
	x = clamp(x + offsetx, 0, w);
	y = clamp(y + offsety, 0, h);

	float uvx = x / (float)w;
	float uvy = y / (float)h;

	return sampleBilinear(buffer, uvx, uvy, w, h);
}

__global__ void onUpsamplingAndMerge(
	hipSurfaceObject_t dst,
	const float4* __restrict__ inLowResColor,
	const float4* __restrict__ inLowResNmlDepth,
	const float4* __restrict__ inHiResColor,
	const float4* __restrict__ inHiResNmlDepth,
	const float4* __restrict__ aovTexclrTemporalWeight,
	int lowResWidth, int lowResHeight,
	int hiResWidth, int hiResHeight)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= hiResWidth || iy >= hiResHeight) {
		return;
	}

	int hiResIdx = getIdx(ix, iy, hiResWidth);

	int pos = (iy & 0x01) * 2 + (ix & 0x01);

	float4 lowResNmlDepth[4];
	float4 lowResClr[4];

	int lx = ix / 2;
	int ly = iy / 2;

	int w = lowResWidth;
	int h = lowResHeight;

	switch (pos) {
	case 0:
		lowResNmlDepth[0] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 0);
		lowResNmlDepth[1] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 1, 0);
		lowResNmlDepth[2] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 1);
		lowResNmlDepth[3] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 1, 1);
		lowResClr[0] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, 0);
		lowResClr[1] = sampleBilinear(inLowResColor, w, h, lx, ly, 1, 0);
		lowResClr[2] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, 1);
		lowResClr[3] = sampleBilinear(inLowResColor, w, h, lx, ly, 1, 1);
		break;
	case 1:
		lowResNmlDepth[0] = samplePoint(inLowResNmlDepth, w, h, lx, ly, -1, 0);
		lowResNmlDepth[1] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 0);
		lowResNmlDepth[2] = samplePoint(inLowResNmlDepth, w, h, lx, ly, -1, 1);
		lowResNmlDepth[3] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 1);
		lowResClr[0] = sampleBilinear(inLowResColor, w, h, lx, ly, -1, 0);
		lowResClr[1] = sampleBilinear(inLowResColor, w, h, lx, ly, 1, 0);
		lowResClr[2] = sampleBilinear(inLowResColor, w, h, lx, ly, -1, 1);
		lowResClr[3] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, 1);
		break;
	case 2:
		lowResNmlDepth[0] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, -1);
		lowResNmlDepth[1] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 1, -1);
		lowResNmlDepth[2] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 0);
		lowResNmlDepth[3] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 1, 0);
		lowResClr[0] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, -1);
		lowResClr[1] = sampleBilinear(inLowResColor, w, h, lx, ly, 1, -1);
		lowResClr[2] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, 0);
		lowResClr[3] = sampleBilinear(inLowResColor, w, h, lx, ly, 1, 0);
		break;
	case 3:
		lowResNmlDepth[0] = samplePoint(inLowResNmlDepth, w, h, lx, ly, -1, -1);
		lowResNmlDepth[1] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, -1);
		lowResNmlDepth[2] = samplePoint(inLowResNmlDepth, w, h, lx, ly, -1, 0);
		lowResNmlDepth[3] = samplePoint(inLowResNmlDepth, w, h, lx, ly, 0, 0);
		lowResClr[0] = sampleBilinear(inLowResColor, w, h, lx, ly, -1, -1);
		lowResClr[1] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, -1);
		lowResClr[2] = sampleBilinear(inLowResColor, w, h, lx, ly, -1, 0);
		lowResClr[3] = sampleBilinear(inLowResColor, w, h, lx, ly, 0, 0);
		break;
	}

	static const float bilateralWeight[] = {
		9.0 / 16.0, 3.0 / 16.0, 3.0 / 16.0, 1.0 / 16.0,
		3.0 / 16.0, 9.0 / 16.0, 1.0 / 16.0, 3.0 / 16.0,
		3.0 / 16.0, 1.0 / 16.0, 9.0 / 16.0, 3.0 / 16.0,
		1.0 / 16.0, 3.0 / 16.0, 3.0 / 16.0, 9.0 / 16.0,
	};

	float4 hiResNmlDepth = inHiResNmlDepth[hiResIdx];

	float4 sum = make_float4(0.0f);
	float sumWeight = 0.0001f;

	for (int i = 0; i < 4; i++) {
		float depthWeight = clamp(1.0 / (0.0001f + abs(hiResNmlDepth.w - lowResNmlDepth[i].w)), 0.0f, 1.0f);

		// Disable depth.
		hiResNmlDepth.w = 0.0f;
		lowResNmlDepth[i].w = 0.0f;

		float nmlWeight = clamp(powf(dot(hiResNmlDepth, lowResNmlDepth[i]), 32), 0.0f, 1.0f);

		float weight = nmlWeight * depthWeight * bilateralWeight[pos * 4 + i];
		sum += lowResClr[i] * weight;
		sumWeight += weight;
	}

	sum /= sumWeight;

	// Merge.
	float4 hiResColor = inHiResColor[hiResIdx];
	//hiResColor = sum;

	hiResColor += inLowResColor[getIdx(lx, ly, lowResWidth)];

	// Multiply Albedo.
	hiResColor *= aovTexclrTemporalWeight[hiResIdx];

	surf2Dwrite(
		hiResColor,
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::coarseBuffer(int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int curaov = getCurAovs();

		int lowResWidth = width / 2;
		int lowResHeight = height / 2;

		coarseBuffers << <grid, block >> > (
			m_paths[Resolution::Hi].ptr(),
			m_rays[Resolution::Hi].ptr(),
			m_aovNormalDepth[Resolution::Hi][curaov].ptr(),
			m_aovMomentMeshid[Resolution::Hi][curaov].ptr(),
			m_paths[Resolution::Low].ptr(),
			m_rays[Resolution::Low].ptr(),
			m_aovNormalDepth[Resolution::Low][curaov].ptr(),
			m_aovMomentMeshid[Resolution::Low][curaov].ptr(),
			width, height,
			lowResWidth, lowResHeight);
		checkCudaKernel(coarseBuffers);

		// Terminate path which is not singular.
		checkIfSingular << <grid, block >> > (
			m_paths[Resolution::Hi].ptr(),
			width, height);
		checkCudaKernel(checkIfSingular);
	}

	void SVGFPathTracing::upsamplingAndMerge(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int curaov = getCurAovs();

		int lowResWidth = width / 2;
		int lowResHeight = height / 2;

		onUpsamplingAndMerge << <grid, block >> > (
			outputSurf,
			m_aovColorVariance[Resolution::Low][curaov].ptr(),
			m_aovNormalDepth[Resolution::Low][curaov].ptr(),
			m_aovColorVariance[Resolution::Hi][curaov].ptr(),
			m_aovNormalDepth[Resolution::Hi][curaov].ptr(),
			m_aovTexclrTemporalWeight[Resolution::Hi][curaov].ptr(),
			lowResWidth, lowResHeight,
			width, height);
		checkCudaKernel(onUpsamplingAndMerge);
	}
}