#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void coarseBuffers(
	hipSurfaceObject_t dstNmlDepth,
	const idaten::SVGFPathTracing::Path* __restrict__ srcPaths,
	const aten::ray* __restrict__ srcRays,
	const float4* __restrict__ srcAovNormalDepth,
	const float4* __restrict__ srcAovTexclrTemporalWeight,
	const float4* __restrict__ srcAovMomentMeshid,
	idaten::SVGFPathTracing::Path* dstPaths,
	aten::ray* dstRays,
	float4* dstAovNormalDepth,
	float4* dstAovTexclrTemporalWeight,
	float4* dstAovMomentMeshid,
	int width, int height,
	int lowResWidth, int lowResHeight)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= lowResWidth || iy >= lowResHeight) {
		return;
	}

	int hiResX = ix * 2;
	int hiResY = iy * 2;

	int idx_0 = getIdx(hiResX + 0, hiResY + 0, width);
	int idx_1 = getIdx(hiResX + 1, hiResY + 0, width);
	int idx_2 = getIdx(hiResX + 0, hiResY + 1, width);
	int idx_3 = getIdx(hiResX + 1, hiResY + 1, width);

	idx_0 = min(idx_0, width * height - 1);
	idx_1 = min(idx_1, width * height - 1);
	idx_2 = min(idx_2, width * height - 1);
	idx_3 = min(idx_3, width * height - 1);

	const idaten::SVGFPathTracing::Path paths[4] = {
		srcPaths[idx_0],
		srcPaths[idx_1],
		srcPaths[idx_2],
		srcPaths[idx_3],
	};
	float4 nmlDepth[4] = {
		srcAovNormalDepth[idx_0],
		srcAovNormalDepth[idx_1],
		srcAovNormalDepth[idx_2],
		srcAovNormalDepth[idx_3],
	};
	int indices[4] = {
		idx_0, idx_1, idx_2, idx_3,
	};

	// Depth����ԑ傫�����̂�I��.
	float maxDepth = -1.0f;
	int pos = -1;

#pragma unroll
	for (int i = 0; i < 4; i++) {
		if (nmlDepth[i].w > maxDepth
			&& !paths[i].isKill
			&& !paths[i].isTerminate)
		{
			maxDepth = nmlDepth[i].w;
			pos = i;
		}
	}

	int idx = getIdx(ix, iy, lowResWidth);

	if (pos >= 0) {
		dstPaths[idx] = paths[pos];
		dstAovNormalDepth[idx] = nmlDepth[pos];

		int srcIdx = indices[pos];

		dstRays[idx] = srcRays[srcIdx];
		dstAovTexclrTemporalWeight[idx] = srcAovTexclrTemporalWeight[srcIdx];
		dstAovMomentMeshid[idx] = srcAovMomentMeshid[srcIdx];
	}
	else {
		int srcIdx = getIdx(hiResX, hiResY, width);

		dstPaths[idx] = srcPaths[srcIdx];
		dstRays[idx] = srcRays[srcIdx];
		dstAovNormalDepth[idx] = srcAovNormalDepth[srcIdx];
		dstAovTexclrTemporalWeight[idx] = srcAovTexclrTemporalWeight[srcIdx];
		dstAovMomentMeshid[idx] = srcAovMomentMeshid[srcIdx];
	}

	surf2Dwrite(
		dstAovNormalDepth[idx],
		dstNmlDepth,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

__global__ void copyBackToOriginalBuffer(
	const idaten::SVGFPathTracing::Path* __restrict__ srcPaths,
	const aten::ray* __restrict__ srcRays,
	const float4* __restrict__ srcAovNormalDepth,
	const float4* __restrict__ srcAovTexclrTemporalWeight,
	const float4* __restrict__ srcAovMomentMeshid,
	idaten::SVGFPathTracing::Path* dstPaths,
	aten::ray* dstRays,
	float4* dstAovNormalDepth,
	float4* dstAovTexclrTemporalWeight,
	float4* dstAovMomentMeshid,
	int lowResWidth, int lowResHeight)
{
	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= lowResWidth || iy >= lowResHeight) {
		return;
	}

	int idx = getIdx(ix, iy, lowResWidth);

	dstPaths[idx] = srcPaths[idx];
	dstRays[idx] = srcRays[idx];
	dstAovNormalDepth[idx] = srcAovNormalDepth[idx];
	dstAovTexclrTemporalWeight[idx] = srcAovTexclrTemporalWeight[idx];
	dstAovMomentMeshid[idx] = srcAovMomentMeshid[idx];
}

namespace idaten
{
	void SVGFPathTracing::onCoarseBuffer(int width, int height)
	{
		int lowResWidth = width / 2;
		int lowResHeight = height / 2;

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(lowResWidth + block.x - 1) / block.x,
			(lowResHeight + block.y - 1) / block.y);

		int curaov = getCurAovs();

		m_tmpPaths.init(lowResWidth * lowResHeight);
		m_tmpRays.init(lowResWidth * lowResHeight);
		m_tmpAovBuffer[0].init(lowResWidth * lowResHeight);
		m_tmpAovBuffer[1].init(lowResWidth * lowResHeight);
		m_tmpAovBuffer[2].init(width * lowResHeight);

		m_aovLowResNmlDepth.map();
		auto aovLowResNmlDepthExportBuffer = m_aovLowResNmlDepth.bind();

		// Coarse buffers.
		coarseBuffers << <grid, block >> > (
			aovLowResNmlDepthExportBuffer,
			m_paths.ptr(),
			m_rays.ptr(),
			m_aovNormalDepth[curaov].ptr(),
			m_aovTexclrTemporalWeight[curaov].ptr(),
			m_aovMomentMeshid[curaov].ptr(),
			m_tmpPaths.ptr(),
			m_tmpRays.ptr(),
			m_tmpAovBuffer[0].ptr(),
			m_tmpAovBuffer[1].ptr(),
			m_tmpAovBuffer[2].ptr(),
			width, height,
			lowResWidth, lowResHeight);
		checkCudaKernel(coarseBuffers);

		m_aovLowResNmlDepth.unbind();
		m_aovLowResNmlDepth.unmap();

		// TODO
		// Copy back to orignal buffers.
		// �{���Ȃ痘�p����o�b�t�@�̐؂�ւ�������΁A�R�s�[�͕K�v�Ȃ��A���x���オ��Ǝv����.
		// ����ł�����̕��@���ƃ������ߖ�ɂȂ�.
		copyBackToOriginalBuffer << <grid, block >> > (
			m_tmpPaths.ptr(),
			m_tmpRays.ptr(),
			m_tmpAovBuffer[0].ptr(),
			m_tmpAovBuffer[1].ptr(),
			m_tmpAovBuffer[2].ptr(),
			m_paths.ptr(),
			m_rays.ptr(),
			m_aovNormalDepth[curaov].ptr(),
			m_aovTexclrTemporalWeight[curaov].ptr(),
			m_aovMomentMeshid[curaov].ptr(),
			lowResWidth, lowResHeight);
		checkCudaKernel(copyBackToOriginalBuffer);
	}
}