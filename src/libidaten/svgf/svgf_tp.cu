#include "hip/hip_runtime.h"
#include "svgf/svgf_pt.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

inline __device__ void computePrevScreenPos(
	int ix, int iy,
	float centerDepth,
	int width, int height,
	aten::vec4* prevPos,
	const aten::mat4* __restrict__ mtxs)
{
	// NOTE
	// Pview = (Xview, Yview, Zview, 1)
	// mtxV2C = W 0 0  0
	//          0 H 0  0
	//          0 0 A  B
	//          0 0 -1 0
	// mtxV2C * Pview = (Xclip, Yclip, Zclip, Wclip) = (Xclip, Yclip, Zclip, Zview)
	//  Wclip = Zview = depth
	// Xscr = Xclip / Wclip = Xclip / Zview = Xclip / depth
	// Yscr = Yclip / Wclip = Yclip / Zview = Yclip / depth
	//
	// Xscr * depth = Xclip
	// Xview = mtxC2V * Xclip

	const aten::mat4 mtxC2V = mtxs[0];
	const aten::mat4 mtxPrevV2C = mtxs[1];

	float2 uv = make_float2(ix + 0.5, iy + 0.5);
	uv /= make_float2(width - 1, height - 1);	// [0, 1]
	uv = uv * 2.0f - 1.0f;	// [0, 1] -> [-1, 1]

	aten::vec4 pos(uv.x, uv.y, 0, 0);

	// Screen-space -> Clip-space.
	pos.x *= centerDepth;
	pos.y *= centerDepth;

	// Clip-space -> View-space
	pos = mtxC2V.apply(pos);
	pos.z = -centerDepth;
	pos.w = 1.0;

	// Reproject previous screen position.
	*prevPos = mtxPrevV2C.apply(pos);
	*prevPos /= prevPos->w;

	*prevPos = *prevPos * 0.5 + 0.5;	// [-1, 1] -> [0, 1]
}

__global__ void temporalReprojection(
	const idaten::SVGFPathTracing::Path* __restrict__ paths,
	const aten::CameraParameter* __restrict__ camera,
	hipSurfaceObject_t* curAovs,
	hipSurfaceObject_t* prevAovs,
	const aten::mat4* __restrict__ mtxs,
	hipSurfaceObject_t dst,
	int width, int height)
{
	const int ix = blockIdx.x * blockDim.x + threadIdx.x;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	const auto path = paths[idx];

	float4 depth_meshid;
	surf2Dread(
		&depth_meshid,
		curAovs[idaten::SVGFPathTracing::AOVType::depth_meshid],
		ix * sizeof(float4), iy);

	const float centerDepth = aten::clamp(depth_meshid.x, camera->znear, camera->zfar);
	const int centerMeshId = (int)depth_meshid.y;

	// ����̃t���[���̃s�N�Z���J���[.
	float4 curColor = make_float4(path.contrib.x, path.contrib.y, path.contrib.z, 0) / path.samples;
	curColor.w = 1;

	if (centerMeshId < 0) {
		// �w�i�Ȃ̂ŁA���̂܂܏o�͂��ďI���.
		surf2Dwrite(
			curColor,
			dst,
			ix * sizeof(float4), iy,
			hipBoundaryModeTrap);

		return;
	}

	float4 centerNormal;
	surf2Dread(
		&centerNormal,
		curAovs[idaten::SVGFPathTracing::AOVType::normal],
		ix * sizeof(float4), iy);

	// [0, 1] -> [-1, 1]
	centerNormal = 2 * centerNormal - 1;
	centerNormal.w = 0;

	float4 sum = make_float4(0, 0, 0, 0);
	float weight = 0.0f;

	float4 prevDepthMeshId;
	float4 prevNormal;

	for (int y = -1; y <= 1; y++) {
		for (int x = -1; x <= 1; x++) {
			int xx = clamp(ix + x, 0, width - 1);
			int yy = clamp(iy + y, 0, height - 1);

			// �O�̃t���[���̃N���b�v��ԍ��W���v�Z.
			aten::vec4 prevPos;
			computePrevScreenPos(
				xx, yy,
				centerDepth,
				width, height,
				&prevPos,
				mtxs);

			// [0, 1]�͈͓̔��ɓ����Ă��邩.
			bool isInsideX = (0.0 <= prevPos.x) && (prevPos.x <= 1.0);
			bool isInsideY = (0.0 <= prevPos.y) && (prevPos.y <= 1.0);

			if (isInsideX && isInsideY) {
				// �O�̃t���[���̃X�N���[�����W.
				int px = (int)(prevPos.x * width - 0.5f);
				int py = (int)(prevPos.y * height - 0.5f);

				px = clamp(px, 0, width - 1);
				py = clamp(py, 0, height - 1);

				surf2Dread(
					&prevDepthMeshId,
					prevAovs[idaten::SVGFPathTracing::AOVType::depth_meshid],
					px * sizeof(float4), py);

				const float prevDepth = aten::clamp(depth_meshid.x, camera->znear, camera->zfar);
				const int prevMeshId = (int)depth_meshid.y;

				surf2Dread(
					&prevNormal,
					prevAovs[idaten::SVGFPathTracing::AOVType::normal],
					px * sizeof(float4), py);

				// [0, 1] -> [-1, 1]
				prevNormal = 2 * prevNormal - 1;
				prevNormal.w = 0;

				// TODO
				// �������b�V����ł����C�g�̂��΂̖��邭�Ȃ����s�N�Z�����E���Ă��܂��ꍇ�̑΍􂪕K�v.

				static const float zThreshold = 0.05f;
				static const float nThreshold = 0.98f;

				float Wz = clamp((zThreshold - abs(1 - centerDepth / prevDepth)) / zThreshold, 0.0f, 1.0f);
				float Wn = clamp((dot(centerNormal, prevNormal) - nThreshold) / (1.0f - nThreshold), 0.0f, 1.0f);
				float Wm = centerMeshId == prevMeshId ? 1.0f : 0.0f;

				// �O�̃t���[���̃s�N�Z���J���[���擾.
				float4 prev;
				surf2Dread(
					&prev, 
					prevAovs[idaten::SVGFPathTracing::AOVType::clr_history],
					px * sizeof(float4), py);

				float W = Wz * Wn * Wm;
				sum += prev * W;
				weight += W;
			}
		}
	}

	if (weight > 0.0f) {
		sum /= weight;
		curColor = 0.2 * curColor + 0.8 * sum;
	}

	surf2Dwrite(
		curColor,
		curAovs[idaten::SVGFPathTracing::AOVType::clr_history],
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);

	surf2Dwrite(
		curColor,
		dst,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void SVGFPathTracing::onTemporalReprojection(
		hipSurfaceObject_t outputSurf,
		int width, int height)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& curaov = getCurAovs();
		auto& prevaov = getPrevAovs();

		aten::mat4 mtxs[2] = {
			m_mtxC2V,
			m_mtxPrevV2C,
		};

		m_mtxs.init(sizeof(aten::mat4) * AT_COUNTOF(mtxs));
		m_mtxs.writeByNum(mtxs, AT_COUNTOF(mtxs));

		temporalReprojection << <grid, block >> > (
			m_paths.ptr(),
			m_cam.ptr(),
			curaov.ptr(),
			prevaov.ptr(),
			m_mtxs.ptr(),
			outputSurf,
			width, height);

		m_mtxs.reset();
	}
}
