#include "kernel/bvh.cuh"
#include "kernel/intersect.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"

#define STACK_SIZE	(64)

AT_DEVICE_API bool intersectBVH(
	const Context* ctxt,
	const aten::ray& r,
	float t_min, float t_max,
	aten::hitrecord* rec)
{
	int stackbuf[STACK_SIZE];

	stackbuf[0] = 0;

	int stackpos = 1;
	int nestedStackPos = -1;

	aten::ray transformedRay = r;
	bool isNested = false;
	aten::hitrecord recTmp;
	bool isHit = false;

	int nodeid = -1;
	aten::BVHNode* node = nullptr;

	while (stackpos > 0) {
		if (stackpos == nestedStackPos) {
			nestedStackPos = -1;
			isNested = false;
			transformedRay = r;
		}

		nodeid = stackbuf[stackpos - 1];
		stackpos--;

		if (nodeid >= 0) {
			node = &ctxt->nodes[nodeid];

			if (node->isLeaf()) {
				if (node->nestid >= 0) {
					if (aten::aabb::hit(transformedRay, node->boxmin, node->boxmax, t_min, t_max)) {
						nestedStackPos = isNested ? nestedStackPos : stackpos;
						stackbuf[stackpos++] = (int)node->nestid;

						if (!isNested) {
							const auto& param = ctxt->shapes[(int)node->shapeid];
							transformedRay.org = param.mtxW2L.apply(r.org);
							transformedRay.dir = param.mtxW2L.applyXYZ(r.dir);
							transformedRay.dir = normalize(transformedRay.dir);
							isNested = true;
						}
					}
				}
				else {
					isHit = false;

					const auto* s = &ctxt->shapes[(int)node->shapeid];

					if (node->primid >= 0) {
						const auto& prim = ctxt->prims[(int)node->primid];
						isHit = intersectShape(s, &prim, ctxt, transformedRay, t_min, t_max, &recTmp);
						recTmp.mtrlid = prim.mtrlid;
					}
					else {
						isHit = intersectShape(s, nullptr, ctxt, transformedRay, t_min, t_max, &recTmp);
						recTmp.mtrlid = s->mtrl.idx;
					}

					if (isHit) {
						if (recTmp.t < rec->t) {
							*rec = recTmp;
							rec->obj = (void*)s;
						}
					}
				}
			}
			else {
				if (aten::aabb::hit(transformedRay, node->boxmin, node->boxmax, t_min, t_max)) {
					stackbuf[stackpos++] = (int)node->left;
					stackbuf[stackpos++] = (int)node->right;

					if (stackpos > STACK_SIZE) {
						//AT_ASSERT(false);
						return false;
					}
				}
			}
		}
	}

	isHit = (rec->obj != nullptr);

	if (isHit) {
		evalHitResult(ctxt, (aten::ShapeParameter*)rec->obj, r, rec);
	}

	return isHit;
}