#include "hip/hip_runtime.h"
#include "kernel/bvh.cuh"
#include "kernel/intersect.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"

#define STACK_SIZE	(64)

struct BVHRay : public aten::ray {
	aten::vec3 inv;
	int sign[3];

	__device__ BVHRay(const aten::ray& r)
	{
		org = r.org;
		dir = r.dir;

		inv = real(1) / dir;

		sign[0] = (inv.x < real(0) ? 1 : 0);
		sign[1] = (inv.y < real(0) ? 1 : 0);
		sign[2] = (inv.z < real(0) ? 1 : 0);
	}
};

__device__ bool intersectAABB(
	const BVHRay* ray,
	const float4* aabb,
	real& t_result)
{
	// NOTE
	// https://github.com/hpicgs/cgsee/wiki/Ray-Box-Intersection-on-the-GPU

	float tmin, tmax, tymin, tymax, tzmin, tzmax;

	tmin = (aabb[ray->sign[0]].x - ray->org.x) * ray->inv.x;
	tmax = (aabb[1 - ray->sign[0]].x - ray->org.x) * ray->inv.x;
	
	tymin = (aabb[ray->sign[1]].y - ray->org.y) * ray->inv.y;
	tymax = (aabb[1 - ray->sign[1]].y - ray->org.y) * ray->inv.y;
	
	tzmin = (aabb[ray->sign[2]].z - ray->org.z) * ray->inv.z;
	tzmax = (aabb[1 - ray->sign[2]].z - ray->org.z) * ray->inv.z;
	
	tmin = max(max(tmin, tymin), tzmin);
	tmax = min(min(tmax, tymax), tzmax);

	if (tmin > tmax) {
		return false;
	}

	t_result = tmin;

	return true;
}

enum IntersectType {
	Closest,
	Closer,
	Any,
};

struct BVHCandidate {
	int exid;
	int shapeid;
};

__device__ bool intersectBVHTriangles(
	hipTextureObject_t nodes,
	const Context* ctxt,
	const aten::ray r,
	float t_min, float t_max,
	aten::Intersection* isect,
	IntersectType type = IntersectType::Closest)
{
	aten::Intersection isectTmp;

	int nodeid = 0;
	float4 node;	// x:left, y:right
	float4 attrib;	// x:shapeid, y:primid, z:nestid

	float4 _boxmin;
	float4 _boxmax;

	aten::vec3 boxmin;
	aten::vec3 boxmax;

	isect->t = t_max;

	for (;;) {
		if (nodeid < 0) {
			break;
		}

		node = tex1Dfetch<float4>(nodes, 4 * nodeid + 0);	// x : hit, y: miss
		attrib = tex1Dfetch<float4>(nodes, 4 * nodeid + 1);	// x : shapeid, y : primgid, z : exid
		_boxmin = tex1Dfetch<float4>(nodes, 4 * nodeid + 2);
		_boxmax = tex1Dfetch<float4>(nodes, 4 * nodeid + 3);

		boxmin = aten::vec3(_boxmin.x, _boxmin.y, _boxmin.z);
		boxmax = aten::vec3(_boxmax.x, _boxmax.y, _boxmax.z);

		bool isHit = false;

		if (attrib.x >= 0 || attrib.y >= 0) {
			const auto& prim = ctxt->prims[(int)attrib.y];

			isectTmp.t = AT_MATH_INF;
			isHit = hitTriangle(&prim, ctxt, r, &isectTmp);
			isectTmp.mtrlid = prim.mtrlid;

			if (isectTmp.t < isect->t) {
				*isect = isectTmp;
				isect->objid = (int)attrib.x;
				isect->primid = (int)attrib.y;

				if (type == IntersectType::Closer) {
					return true;
				}
			}
		}
		else {
			isHit = aten::aabb::hit(r, boxmin, boxmax, t_min, t_max);
		}

		if (isHit) {
			nodeid = (int)node.x;
		}
		else {
			nodeid = (int)node.y;
		}
	}

	return (isect->objid >= 0);
}

__device__ bool intersectBVH(
	hipTextureObject_t nodes,
	const Context* ctxt,
	const aten::ray r,
	float t_min, float t_max,
	aten::Intersection* isect,
	IntersectType type = IntersectType::Closest)
{
	aten::Intersection isectTmp;

	real hitt = t_max;
	isect->t = t_max;

	int tmpexid = -1;

	int nodeid = 0;
	float4 node;	// x:left, y:right
	float4 attrib;	// x:shapeid, y:primid, z:nestid

	float4 aabb[2];

	BVHRay bvhray(r);
	real t = AT_MATH_INF;

	for (;;) {
		if (nodeid < 0) {
			break;
		}

		node = tex1Dfetch<float4>(nodes, 4 * nodeid + 0);	// x : hit, y: miss
		attrib = tex1Dfetch<float4>(nodes, 4 * nodeid + 1);	// x : shapeid, y : primgid, z : exid
		aabb[0] = tex1Dfetch<float4>(nodes, 4 * nodeid + 2);
		aabb[1] = tex1Dfetch<float4>(nodes, 4 * nodeid + 3);

		auto boxmin = aten::vec3(aabb[0].x, aabb[0].y, aabb[0].z);
		auto boxmax = aten::vec3(aabb[1].x, aabb[1].y, aabb[1].z);

		bool isHit = false;

		if (attrib.x >= 0 || attrib.y >= 0) {
			// Leaf.
			tmpexid = -1;

			const auto* s = &ctxt->shapes[(int)attrib.x];

			if (attrib.z >= 0) {	// exid
				aten::ray transformedRay;

				if (s->mtxid >= 0) {
					auto mtxW2L = ctxt->matrices[s->mtxid * 2 + 1];
					transformedRay.dir = mtxW2L.applyXYZ(r.dir);
					transformedRay.dir = normalize(transformedRay.dir);
					transformedRay.org = mtxW2L.apply(r.org) + AT_MATH_EPSILON * transformedRay.dir;
				}
				else {
					transformedRay = r;
				}

				isHit = intersectBVHTriangles(ctxt->nodes[(int)attrib.z], ctxt, transformedRay, t_min, t_max, &isectTmp);
			}
			else {
				// TODO
				// Only sphere...
				//isHit = intersectShape(s, nullptr, ctxt, r, t_min, t_max, &recTmp, &recOptTmp);
				isectTmp.t = AT_MATH_INF;
				isHit = hitSphere(s, r, t_min, t_max, &isectTmp);
				isectTmp.mtrlid = s->mtrl.idx;
			}

			if (isHit) {
				if (isectTmp.t < isect->t) {
					*isect = isectTmp;
					isect->objid = (int)attrib.x;

					if (type == IntersectType::Closer) {
						return true;
					}
				}
			}
		}
		else {
			isHit = aten::aabb::hit(r, boxmin, boxmax, t_min, t_max, &t);
		}

		if (isHit) {
			nodeid = (int)node.x;
		}
		else {
			nodeid = (int)node.y;
		}
	}

	return (isect->objid >= 0);
}

__device__ bool intersectBVH(
	const Context* ctxt,
	const aten::ray& r,
	aten::Intersection* isect)
{
	float t_min = AT_MATH_EPSILON;
	float t_max = AT_MATH_INF;

	bool isHit = intersectBVH(
		ctxt->nodes[0],
		ctxt,
		r,
		t_min, t_max,
		isect);

	return isHit;
}

__device__ bool intersectCloserBVH(
	const Context* ctxt,
	const aten::ray& r,
	aten::Intersection* isect,
	const float t_max)
{
	float t_min = AT_MATH_EPSILON;

	bool isHit = intersectBVH(
		ctxt->nodes[0],
		ctxt,
		r,
		t_min, t_max,
		isect,
		IntersectType::Closer);

	return isHit;
}