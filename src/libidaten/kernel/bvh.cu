#include "hip/hip_runtime.h"
#include "kernel/bvh.cuh"
#include "kernel/intersect.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"

#define STACK_SIZE	(64)

__device__ bool intersectBVH(
	const Context* ctxt,
	const aten::ray& r,
	float t_min, float t_max,
	aten::hitrecord* rec)
{
	int stackbuf[STACK_SIZE];

	stackbuf[0] = 0;

	int stackpos = 1;
	int nestedStackPos = -1;

	aten::ray transformedRay = r;
	bool isNested = false;
	aten::hitrecord recTmp;
	bool isHit = false;

	int nodeid = -1;
	float4 node;	// x:left, y:right
	float4 attrib;	// x:shapeid, y:primid, z:nestid

	float4 _boxmin;
	float4 _boxmax;

	aten::vec3 boxmin;
	aten::vec3 boxmax;

	while (stackpos > 0) {
		if (stackpos == nestedStackPos) {
			nestedStackPos = -1;
			isNested = false;
			transformedRay = r;
		}

		nodeid = stackbuf[stackpos - 1];
		stackpos--;

		if (nodeid >= 0) {
			node = tex1Dfetch<float4>(ctxt->nodes, 4 * nodeid + 0);
			attrib = tex1Dfetch<float4>(ctxt->nodes, 4 * nodeid + 1);
			_boxmin = tex1Dfetch<float4>(ctxt->nodes, 4 * nodeid + 2);
			_boxmax = tex1Dfetch<float4>(ctxt->nodes, 4 * nodeid + 3);

			boxmin = aten::make_float3(_boxmin.x, _boxmin.y, _boxmin.z);
			boxmax = aten::make_float3(_boxmax.x, _boxmax.y, _boxmax.z);

			if (node.x < 0 && node.y < 0) {
				if (attrib.z >= 0) {
					if (aten::aabb::hit(transformedRay, boxmin, boxmax, t_min, t_max)) {
						nestedStackPos = isNested ? nestedStackPos : stackpos;
						stackbuf[stackpos++] = (int)attrib.z;

						if (!isNested) {
							const auto& param = ctxt->shapes[(int)attrib.x];
							transformedRay.org = param.mtxW2L.apply(r.org);
							transformedRay.dir = param.mtxW2L.applyXYZ(r.dir);
							transformedRay.dir = normalize(transformedRay.dir);
							isNested = true;
						}
					}
				}
				else {
					isHit = false;

					const auto* s = &ctxt->shapes[(int)attrib.x];

					if (attrib.y >= 0) {
						const auto& prim = ctxt->prims[(int)attrib.y];
						isHit = intersectShape(s, &prim, ctxt, transformedRay, t_min, t_max, &recTmp);
						recTmp.mtrlid = prim.mtrlid;
					}
					else {
						isHit = intersectShape(s, nullptr, ctxt, transformedRay, t_min, t_max, &recTmp);
						recTmp.mtrlid = s->mtrl.idx;
					}

					if (isHit) {
						if (recTmp.t < rec->t) {
							*rec = recTmp;
							rec->obj = (void*)s;
						}
					}
				}
			}
			else {
				if (aten::aabb::hit(transformedRay, boxmin, boxmax, t_min, t_max)) {
					stackbuf[stackpos++] = (int)node.x;
					stackbuf[stackpos++] = (int)node.y;

					if (stackpos > STACK_SIZE) {
						//AT_ASSERT(false);
						return false;
					}
				}
			}
		}
	}

	isHit = (rec->obj != nullptr);

	if (isHit) {
		evalHitResult(ctxt, (aten::ShapeParameter*)rec->obj, r, rec);
	}

	return isHit;
}