#include "hip/hip_runtime.h"
#include "kernel/bvh.cuh"
#include "kernel/intersect.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"

#define STACK_SIZE	(64)

__device__ bool intersectBVH(
	hipTextureObject_t nodes,
	int* exid,
	int* shapeid,
	const Context* ctxt,
	const aten::ray r,
	float t_min, float t_max,
	aten::hitrecord* rec)
{
	int stackbuf[STACK_SIZE];

	stackbuf[0] = 0;

	int stackpos = 1;

	aten::hitrecord recTmp;
	bool isHit = false;

	real hitt = AT_MATH_INF;

	*exid = -1;
	int tmpexid = -1;

	*shapeid = -1;
	int tmpshapeid = -1;

	int nodeid = -1;
	float4 node;	// x:left, y:right
	float4 attrib;	// x:shapeid, y:primid, z:nestid

	float4 _boxmin;
	float4 _boxmax;

	aten::vec3 boxmin;
	aten::vec3 boxmax;

	while (stackpos > 0) {
		nodeid = stackbuf[stackpos - 1];
		stackpos--;

		if (nodeid >= 0) {
			node = tex1Dfetch<float4>(nodes, 4 * nodeid + 0);	// x : left, y: right
			attrib = tex1Dfetch<float4>(nodes, 4 * nodeid + 1);	// x : shapeid, y : primgid, z : nestid, w : exid
			_boxmin = tex1Dfetch<float4>(nodes, 4 * nodeid + 2);
			_boxmax = tex1Dfetch<float4>(nodes, 4 * nodeid + 3);

			boxmin = aten::make_float3(_boxmin.x, _boxmin.y, _boxmin.z);
			boxmax = aten::make_float3(_boxmax.x, _boxmax.y, _boxmax.z);

			if (node.x < 0 && node.y < 0) {
				if (attrib.z >= 0) {
					if (aten::aabb::hit(r, boxmin, boxmax, t_min, t_max)) {
						stackbuf[stackpos++] = (int)attrib.z;
						tmpshapeid = (int)attrib.x;
					}
				}
				else {
					isHit = false;
					tmpexid = -1;

					const auto* s = &ctxt->shapes[(int)attrib.x];

					if (attrib.w >= 0) {	// exid
						real t = AT_MATH_INF;
						isHit = aten::aabb::hit(r, boxmin, boxmax, t_min, t_max, &t);
						recTmp.t = t;
						tmpexid = attrib.w;
					}
					else if (attrib.y >= 0) {	// primid
						const auto& prim = ctxt->prims[(int)attrib.y];
						isHit = intersectShape(s, &prim, ctxt, r, t_min, t_max, &recTmp);
						recTmp.mtrlid = prim.mtrlid;
					}
					else {
						isHit = intersectShape(s, nullptr, ctxt, r, t_min, t_max, &recTmp);
						recTmp.mtrlid = s->mtrl.idx;
						tmpexid = -1;
					}

					if (isHit) {
						if (recTmp.t < hitt) {
							hitt = recTmp.t;
							*exid = tmpexid;
							*shapeid = tmpshapeid;
						}
						if (tmpexid < 0) {
							if (recTmp.t < rec->t) {
								*rec = recTmp;
								rec->obj = (void*)s;
							}
						}
					}
				}
			}
			else {
				if (aten::aabb::hit(r, boxmin, boxmax, t_min, t_max)) {
					stackbuf[stackpos++] = (int)node.x;
					stackbuf[stackpos++] = (int)node.y;

					if (stackpos > STACK_SIZE) {
						//AT_ASSERT(false);
						return false;
					}
				}
			}
		}
	}

	return (rec->obj != nullptr);
}

__device__ bool intersectBVH(
	hipTextureObject_t nodes,
	const Context* ctxt,
	const aten::ray r,
	float t_min, float t_max,
	aten::hitrecord* rec)
{
	int stackbuf[STACK_SIZE];

	stackbuf[0] = 0;

	int stackpos = 1;

	aten::hitrecord recTmp;
	bool isHit = false;

	int nodeid = -1;
	float4 node;	// x:left, y:right
	float4 attrib;	// x:shapeid, y:primid, z:nestid

	float4 _boxmin;
	float4 _boxmax;

	aten::vec3 boxmin;
	aten::vec3 boxmax;

	while (stackpos > 0) {
		nodeid = stackbuf[stackpos - 1];
		stackpos--;

		if (nodeid >= 0) {
			node = tex1Dfetch<float4>(nodes, 4 * nodeid + 0);	// x : left, y: right
			attrib = tex1Dfetch<float4>(nodes, 4 * nodeid + 1);	// x : shapeid, y : primgid, z : nestid, w : exid
			_boxmin = tex1Dfetch<float4>(nodes, 4 * nodeid + 2);
			_boxmax = tex1Dfetch<float4>(nodes, 4 * nodeid + 3);

			boxmin = aten::make_float3(_boxmin.x, _boxmin.y, _boxmin.z);
			boxmax = aten::make_float3(_boxmax.x, _boxmax.y, _boxmax.z);

			if (node.x < 0 && node.y < 0) {
				isHit = false;

				const auto* s = &ctxt->shapes[(int)attrib.x];

				const auto& prim = ctxt->prims[(int)attrib.y];
				isHit = intersectShape(s, &prim, ctxt, r, t_min, t_max, &recTmp);
				recTmp.mtrlid = prim.mtrlid;

				if (isHit) {
					if (recTmp.t < rec->t) {
						*rec = recTmp;
						rec->obj = (void*)s;
					}
				}
			}
			else {
				if (aten::aabb::hit(r, boxmin, boxmax, t_min, t_max)) {
					stackbuf[stackpos++] = (int)node.x;
					stackbuf[stackpos++] = (int)node.y;

					if (stackpos > STACK_SIZE) {
						//AT_ASSERT(false);
						return false;
					}
				}
			}
		}
	}

	return (rec->obj != nullptr);
}


__device__ bool intersectBVH(
	const Context* ctxt,
	const aten::ray& r,
	float t_min, float t_max,
	aten::hitrecord* rec)
{
	int exid = -1;
	int shapeid = -1;

	bool isHit = intersectBVH(
		ctxt->nodes[0],
		&exid,
		&shapeid,
		ctxt,
		r,
		t_min, t_max,
		rec);

	if (exid >= 0) {
		aten::hitrecord recTmp;

		const auto& param = ctxt->shapes[shapeid];

		aten::ray transformedRay;
		transformedRay.org = param.mtxW2L.apply(r.org);
		transformedRay.dir = param.mtxW2L.applyXYZ(r.dir);
		transformedRay.dir = normalize(transformedRay.dir);

		if (intersectBVH(ctxt->nodes[exid], ctxt, transformedRay, t_min, t_max, &recTmp)) {
			if (recTmp.t < rec->t) {
				*rec = recTmp;
				isHit = true;
			}
		}
	}

	if (isHit) {
		evalHitResult(ctxt, (aten::ShapeParameter*)rec->obj, r, rec);
	}

	return isHit;
}