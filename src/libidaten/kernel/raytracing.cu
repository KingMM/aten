#include "hip/hip_runtime.h"
#include "kernel/raytracing.h"

#include "hip/hip_runtime.h"
#include ""

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten.h"

// ���s�x�N�g�����v�Z.
__host__ __device__ aten::vec3 getOrthoVector(const aten::vec3& n)
{
	aten::vec3 p;

	// NOTE
	// dot���v�Z�����Ƃ��Ƀ[���ɂȂ�悤�ȃx�N�g��.
	// k �� normalize �v�Z�p.

	if (abs(n.z) > 0.0f) {
		float k = sqrtf(n.y * n.y + n.z * n.z);
		p.x = 0;
		p.y = -n.z / k;
		p.z = n.y / k;
	}
	else {
		float k = sqrtf(n.x * n.x + n.y * n.y);
		p.x = n.y / k;
		p.y = -n.x / k;
		p.z = 0;
	}

	return std::move(p);
}

struct Context {
	int geomnum;
	aten::ShapeParameter* shapes;

	aten::MaterialParameter* mtrls;
};

struct CameraSampleResult {
	aten::ray r;
	aten::vec3 posOnLens;
	aten::vec3 nmlOnLens;

	__host__ __device__ CameraSampleResult() {}
};

struct Camera {
	aten::vec3 origin;

	float aspect;
	aten::vec3 center;

	aten::vec3 u;
	aten::vec3 v;

	aten::vec3 dir;
	aten::vec3 right;
	aten::vec3 up;

	float dist;
	int width;
	int height;
};

__host__ void initCamera(
	Camera& camera,
	const aten::vec3& origin,
	const aten::vec3& lookat,
	const aten::vec3& up,
	float vfov,	// vertical fov.
	uint32_t width, uint32_t height)
{
	float theta = Deg2Rad(vfov);

	camera.aspect = width / (float)height;

	float half_height = tanf(theta / 2);
	float half_width = camera.aspect * half_height;

	camera.origin = origin;

	// �J�������W�x�N�g��.
	camera.dir = normalize(lookat - origin);
	camera.right = normalize(cross(camera.dir, up));
	camera.up = cross(camera.right, camera.dir);

	camera.center = origin + camera.dir;

	// �X�N���[����UV�x�N�g��.
	camera.u = half_width * camera.right;
	camera.v = half_height * camera.up;

	camera.dist = height / (2.0f * tanf(theta / 2));

	camera.width = width;
	camera.height = height;
}

__host__ __device__ void sampleCamera(
	CameraSampleResult* sample,
	Camera* camera,
	float s, float t)
{
	// [0, 1] -> [-1, 1]
	s = 2 * s - 1;
	t = 2 * t - 1;

	auto screenPos = s * camera->u + t * camera->v;

	screenPos = screenPos + camera->center;

	auto dirToScr = screenPos - camera->origin;

	sample->posOnLens = screenPos;
	sample->nmlOnLens = camera->dir;
	sample->r = aten::ray(camera->origin, dirToScr);
}

__host__ __device__ bool intersect(
	const aten::ray* r,
	aten::hitrecord* rec,
	const Context* ctx)
{
	bool isHit = false;

	aten::hitrecord tmp;

	for (int i = 0; i < ctx->geomnum; i++) {
		const auto& s = ctx->shapes[i];
		if (aten::sphere::hit(s, *r, AT_MATH_EPSILON, AT_MATH_INF, tmp)) {
			if (tmp.t < rec->t) {
				*rec = tmp;
				rec->obj = (void*)&ctx->shapes[i];
				rec->mtrlid = ctx->shapes[i].mtrlid;

				isHit = true;
			}
		}
	}

	return isHit;
}

__global__ void raytracing(
	float4* p,
	int width, int height,
	Camera* camera,
	aten::ShapeParameter* shapes, int num,
	aten::MaterialParameter* mtrls)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctx;
	{
		ctx.geomnum = num;
		ctx.shapes = shapes;
		ctx.mtrls = mtrls;
	}

	const auto idx = iy * camera->width + ix;

	float s = ix / (float)camera->width;
	float t = iy / (float)camera->height;

	CameraSampleResult camsample;
	sampleCamera(&camsample, camera, s, t);

	aten::vec3 contrib(0);
	aten::vec3 throughput(1);

	int depth = 0;

	aten::ray ray = camsample.r;

	while (depth < 5) {
		aten::hitrecord rec;

		if (intersect(&ray, &rec, &ctx)) {
			const aten::MaterialParameter& m = ctx.mtrls[rec.mtrlid];

			if (m.attrib.isEmissive) {
				auto emit = m.baseColor;
				contrib = throughput * emit;
				break;
			}

			// �����ʒu�̖@��.
			// ���̂���̃��C�̓��o���l��.
			const aten::vec3 orienting_normal = dot(rec.normal, ray.dir) < 0.0 ? rec.normal : -rec.normal;

			if (m.attrib.isSingular || m.attrib.isTranslucent) {
			}
			else {
				// TODO
				auto* sphere = &ctx.shapes[0];;
				aten::LightParameter light(aten::LightType::Area, aten::LightAttributeArea);
				light.object.ptr = sphere;
				light.le = ctx.mtrls[sphere->mtrlid].baseColor;

				auto funcHitTestSphere = [] AT_DEVICE_API(const aten::vec3& o, const aten::UnionIdxPtr& object, aten::vec3& pos, aten::sampler* smpl, aten::hitrecord& _rec)
				{
					aten::ShapeParameter* s = (aten::ShapeParameter*)object.ptr;

					pos = s->center;

					auto dir = pos - o;
					auto dist = dir.length();

					aten::ray r(o, normalize(dir));
					bool isHit = aten::sphere::hit(*s, r, AT_MATH_EPSILON, AT_MATH_INF, _rec);

					return isHit;
				};

				aten::LightSampleResult sampleres = aten::AreaLight::sample(
					funcHitTestSphere,
					light,
					rec.p,
					nullptr);

				aten::vec3 dirToLight = sampleres.dir;
				auto len = dirToLight.length();

				dirToLight.normalize();

				auto albedo = m.baseColor;

				aten::ray shadowRay(rec.p, dirToLight);

				aten::hitrecord tmpRec;

				auto funcHitTest = [&] AT_DEVICE_API (const aten::ray& _r, float t_min, float t_max, aten::hitrecord& _rec)
				{
					return intersect(&_r, &_rec, &ctx);
				};

				if (aten::scene::hitLight(funcHitTest, light, sampleres.pos, shadowRay, AT_MATH_EPSILON, AT_MATH_INF, tmpRec)) {
					auto lightColor = sampleres.finalColor;

					if (light.attrib.isInfinite) {
						len = 1.0f;
					}

					const auto c0 = max(0.0f, dot(orienting_normal, dirToLight));
					float c1 = 1.0f;

					if (!light.attrib.isSingular) {
						c1 = max(0.0f, dot(sampleres.nml, -dirToLight));
					}

					auto G = c0 * c1 / (len * len);

					contrib += throughput * (albedo * lightColor) * G;
				}
			}
		}
		else {
			break;
		}

		depth++;
	}

	p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
}

aten::ShapeParameter g_spheres[] = {
	aten::ShapeParameter(aten::vec3(0, 0, -10), 1.0f),
	aten::ShapeParameter(aten::vec3(3, 0, -10), 1.0f),
};

void renderRayTracing(
	aten::vec4* image,
	int width, int height)
{
	Camera camera;
	initCamera(
		camera,
		aten::vec3(0, 0, 0),
		aten::vec3(0, 0, -1),
		aten::vec3(0, 1, 0),
		30,
		width, height);

	const std::vector<aten::material*>& mtrls = aten::material::getMaterials();

#if 1
	aten::CudaMemory dst(sizeof(float4) * width * height);

	aten::TypedCudaMemory<Camera> cam(&camera, 1);

	// Bind material.
	g_spheres[0].mtrlid = 0;
	g_spheres[1].mtrlid = 1;
	
	aten::TypedCudaMemory<aten::ShapeParameter> spheres(AT_COUNTOF(g_spheres));
	spheres.writeByNum(g_spheres, AT_COUNTOF(g_spheres));

	std::vector<aten::MaterialParameter> mtrlparams;
	for (auto m : mtrls) {
		mtrlparams.push_back(m->param());
	}

	aten::TypedCudaMemory<aten::MaterialParameter> materials(mtrlparams.size());
	materials.writeByNum(&mtrlparams[0], mtrlparams.size());

	dim3 block(32, 32);
	dim3 grid(
		(width + block.x - 1) / block.x,
		(height + block.y - 1) / block.y);

	raytracing << <grid, block >> > (
	//raytracing << <dim3(1, 1), block >> > (
		(float4*)dst.ptr(), 
		width, height, 
		cam.ptr(),
		spheres.ptr(), AT_COUNTOF(g_spheres),
		materials.ptr());

	checkCudaErrors(hipDeviceSynchronize());

	dst.read(image, sizeof(aten::vec4) * width * height);
#else
	for (int y = 0; y < height; y++) {
		for (int x = 0; x < width; x++) {
			raytracing(
				x, y,
				(float4*)image,
				width, height,
				&camera,
				g_spheres, AT_COUNTOF(g_spheres));
		}
	}
#endif
}
