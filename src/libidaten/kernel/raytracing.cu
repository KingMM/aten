#include "hip/hip_runtime.h"
#include "kernel/raytracing.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

struct Path {
	aten::ray ray;
	aten::vec3 throughput;
	aten::hitrecord rec;
	bool isHit;
	bool isTerminate;
};

__global__ void genPath(
	Path* paths,
	int width, int height,
	aten::CameraParameter* camera)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * camera->width + ix;

	float s = (ix + 0.5f) / (float)(camera->width - 1);
	float t = (iy + 0.5f) / (float)(camera->height - 1);

	auto camsample = AT_NAME::PinholeCamera::sample(*camera, s, t, nullptr);

	auto& path = paths[idx];

	path.ray = camsample.r;
	path.throughput = aten::vec3(1);
	path.isHit = false;
	path.isTerminate = false;
}

__global__ void hitTest(
	Path* paths,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	aten::BVHNode* nodes,
	aten::PrimitiveParamter* prims,
	aten::vertex* vertices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
	}
	
	aten::hitrecord rec;
	bool isHit = intersectBVH(&ctxt, path.ray, AT_MATH_EPSILON, AT_MATH_INF, &rec);

	path.isHit = isHit;
	path.rec = rec;
}

__global__ void raytracing(
	//float4* p,
	hipSurfaceObject_t outSurface,
	Path* paths,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	aten::BVHNode* nodes,
	aten::PrimitiveParamter* prims,
	aten::vertex* vertices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];

	if (!path.isHit) {
		return;
	}
	if (path.isTerminate) {
		return;
	}

	aten::vec3 contrib(0);

	const aten::MaterialParameter* mtrl = &ctxt.mtrls[path.rec.mtrlid];

	if (mtrl->attrib.isEmissive) {
		contrib = path.throughput * mtrl->baseColor;

		path.isTerminate = true;
		//p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
		surf2Dwrite(make_float4(contrib.x, contrib.y, contrib.z, 1), outSurface, ix * sizeof(float4), iy, hipBoundaryModeTrap);
		
		return;
	}

	// �����ʒu�̖@��.
	// ���̂���̃��C�̓��o���l��.
	const aten::vec3 orienting_normal = dot(path.rec.normal, path.ray.dir) < 0.0 ? path.rec.normal : -path.rec.normal;

	if (mtrl->attrib.isSingular || mtrl->attrib.isTranslucent) {
		AT_NAME::MaterialSampling sampling;
			
		sampleMaterial(
			&sampling,
			mtrl,
			orienting_normal, 
			path.ray.dir,
			path.rec.normal,
			nullptr,
			path.rec.u, path.rec.v);

		auto nextDir = normalize(sampling.dir);

		path.throughput *= sampling.bsdf;

		// Make next ray.
		path.ray = aten::ray(path.rec.p, nextDir);
	}
	else {
		// TODO
		auto light = lights[0];
		auto* sphere = &ctxt.shapes[light.object.idx];
		light.object.ptr = sphere;

		aten::LightSampleResult sampleres;
		sampleLight(&sampleres, &light, path.rec.p, nullptr);

		aten::vec3 dirToLight = sampleres.dir;
		auto len = dirToLight.length();

		dirToLight.normalize();

		aten::ray shadowRay(path.rec.p, dirToLight);

		aten::hitrecord tmpRec;

		auto funcHitTest = [&] AT_DEVICE_API(const aten::ray& _r, float t_min, float t_max, aten::hitrecord* _rec)
		{
			return intersectBVH(&ctxt, _r, t_min, t_max, _rec);
		};

		if (AT_NAME::scene::hitLight(funcHitTest, light, sampleres.pos, shadowRay, AT_MATH_EPSILON, AT_MATH_INF, &tmpRec)) {
			if (light.attrib.isInfinite) {
				len = 1.0f;
			}

			const auto c0 = max(0.0f, dot(orienting_normal, dirToLight));
			float c1 = 1.0f;

			if (!light.attrib.isSingular) {
				c1 = max(0.0f, dot(sampleres.nml, -dirToLight));
			}

			auto G = c0 * c1 / (len * len);

			contrib += path.throughput * (mtrl->baseColor * sampleres.finalColor) * G;
		}

		path.isTerminate = true;
		//p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
		surf2Dwrite(make_float4(contrib.x, contrib.y, contrib.z, 1), outSurface, ix * sizeof(float4), iy, hipBoundaryModeTrap);
	}
}

__global__ void addFuncs()
{
	addLighFuncs();
	addMaterialFuncs();
	addIntersectFuncs();
}

namespace idaten {
	void RayTracing::prepare()
	{
		addFuncs << <1, 1 >> > ();
	}

	void RayTracing::update(
		GLuint gltex,
		int width, int height,
		const aten::CameraParameter& camera,
		const std::vector<aten::ShapeParameter>& shapes,
		const std::vector<aten::MaterialParameter>& mtrls,
		const std::vector<aten::LightParameter>& lights,
		const std::vector<aten::BVHNode>& nodes,
		const std::vector<aten::PrimitiveParamter>& prims,
		const std::vector<aten::vertex>& vtxs)
	{
#if 0
		size_t size_stack = 0;
		checkCudaErrors(cudaThreadGetLimit(&size_stack, hipLimitStackSize));
		checkCudaErrors(cudaThreadSetLimit(hipLimitStackSize, 12928));
		checkCudaErrors(cudaThreadGetLimit(&size_stack, hipLimitStackSize));

		AT_PRINTF("Stack size %d\n", size_stack);
#endif

#if 0
		dst.init(sizeof(float4) * width * height);
#else
		glimg.init(gltex, CudaGLRscRegisterType::WriteOnly);
#endif

		cam.init(sizeof(camera));
		cam.writeByNum(&camera, 1);

		shapeparam.init(shapes.size());
		shapeparam.writeByNum(&shapes[0], shapes.size());

		mtrlparam.init(mtrls.size());
		mtrlparam.writeByNum(&mtrls[0], mtrls.size());

		lightparam.init(lights.size());
		lightparam.writeByNum(&lights[0], lights.size());

		nodeparam.init(nodes.size());
		nodeparam.writeByNum(&nodes[0], nodes.size());

		if (!prims.empty()) {
			primparams.init(prims.size());
			primparams.writeByNum(&prims[0], prims.size());
		}

		if (!vtxs.empty()) {
			vtxparams.init(vtxs.size());
			vtxparams.writeByNum(&vtxs[0], vtxs.size());
		}
	}

	void RayTracing::render(
		aten::vec4* image,
		int width, int height)
	{
		dim3 block(16, 16);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int depth = 0;

		idaten::TypedCudaMemory<Path> paths;
		paths.init(width * height);

		CudaGLResourceMap rscmap(&glimg);
		auto outputSurf = glimg.bindToWrite();

		genPath << <grid, block >> > (
			paths.ptr(),
			width, height,
			cam.ptr());

		//checkCudaErrors(hipDeviceSynchronize());

		while (depth < 5) {
			hitTest << <grid, block >> > (
				paths.ptr(),
				width, height,
				shapeparam.ptr(), shapeparam.num(),
				mtrlparam.ptr(),
				lightparam.ptr(), lightparam.num(),
				nodeparam.ptr(),
				primparams.ptr(),
				vtxparams.ptr());

			auto err = hipGetLastError();
			if (err != hipSuccess) {
				AT_PRINTF("Cuda Kernel Err(hitTest) [%s]\n", hipGetErrorString(err));
			}

			//checkCudaErrors(hipDeviceSynchronize());

			raytracing << <grid, block >> > (
				//(float4*)dst.ptr(),
				outputSurf,
				paths.ptr(),
				width, height,
				shapeparam.ptr(), shapeparam.num(),
				mtrlparam.ptr(),
				lightparam.ptr(), lightparam.num(),
				nodeparam.ptr(),
				primparams.ptr(),
				vtxparams.ptr());

			err = hipGetLastError();
			if (err != hipSuccess) {
				AT_PRINTF("Cuda Kernel Err(raytracing) [%s]\n", hipGetErrorString(err));
			}

			//checkCudaErrors(hipDeviceSynchronize());

			depth++;
		}

		checkCudaErrors(hipDeviceSynchronize());

		//dst.read(image, sizeof(aten::vec4) * width * height);
	}
}