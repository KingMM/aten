#include "kernel/RadixSort.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"

namespace idaten
{
	RadixSort::~RadixSort()
	{
		if (m_deviceKeys) {
			thrust::device_vector<uint32_t>& deviceKeys = *(thrust::device_vector<uint32_t>*)(m_deviceKeys);
			deviceKeys.clear();
			delete m_deviceKeys;
		}
		if (m_deviceValues) {
			thrust::device_vector<uint32_t>& deviceValues = *(thrust::device_vector<uint32_t>*)(m_deviceValues);
			deviceValues.clear();
			delete m_deviceValues;
		}
	}

	void RadixSort::init(uint32_t num)
	{
		if (!m_deviceKeys) {
			m_deviceKeys = new thrust::device_vector<uint32_t>(num);
		}

		if (!m_deviceValues) {
			m_deviceValues = new thrust::device_vector<uint32_t>(num);
		}
	}

	static void radixSort(
		uint32_t num,
		thrust::device_vector<uint32_t> deviceKeys,
		thrust::device_vector<uint32_t> deviceValues,
		TypedCudaMemory<uint32_t>& refSortedKeys,
		TypedCudaMemory<uint32_t>& refSortedValues,
		std::vector<uint32_t>* resultHostKeys/*= nullptr*/,
		std::vector<uint32_t>* resultHostValues/*= nullptr*/)
	{
		thrust::sort_by_key(deviceKeys.begin(), deviceKeys.begin() + num, deviceValues.begin());

		auto sortedKeys = thrust::raw_pointer_cast(deviceKeys.data());
		auto sortedValues = thrust::raw_pointer_cast(deviceValues.data());

		refSortedKeys.init(deviceKeys.size() * sizeof(uint32_t));
		refSortedKeys.writeByNum(sortedKeys, num);

		refSortedValues.init(deviceValues.size() * sizeof(uint32_t));
		refSortedValues.writeByNum(sortedValues, num);

		if (resultHostKeys) {
			thrust::host_vector<uint32_t> hostKeys = deviceKeys;
			for (int i = 0; i < num; i++) {
				resultHostKeys->push_back(hostKeys[i]);
			}
		}

		if (resultHostValues) {
			thrust::host_vector<uint32_t> hostValues = deviceValues;
			for (int i = 0; i < num; i++) {
				resultHostValues->push_back(hostValues[i]);
			}
		}
	}

	void RadixSort::sort(
		const std::vector<uint32_t>& keys,
		const std::vector<uint32_t>& values,
		TypedCudaMemory<uint32_t>& refSortedKeys,
		TypedCudaMemory<uint32_t>& refSortedValues,
		std::vector<uint32_t>* resultHostKeys/*= nullptr*/,
		std::vector<uint32_t>* resultHostValues/*= nullptr*/)
	{
		AT_ASSERT(keys.size() == values.size());

		uint32_t num = (uint32_t)keys.size();

		thrust::host_vector<uint32_t> hostKeys(num);
		thrust::host_vector<uint32_t> hostValues(num);

		for (uint32_t i = 0; i < num; i++) {
			hostKeys[i] = keys[i];
			hostValues[i] = values[i];
		}

		// copy unsorted data from host to device
		thrust::device_vector<uint32_t> deviceKeys = hostKeys;
		thrust::device_vector<uint32_t> deviceValues = hostValues;

		radixSort(
			num,
			deviceKeys,
			deviceValues,
			refSortedKeys,
			refSortedValues,
			resultHostKeys,
			resultHostValues);
	}

	void RadixSort::sort(
		uint32_t num,
		TypedCudaMemory<uint32_t>& keys,
		TypedCudaMemory<uint32_t>& values,
		TypedCudaMemory<uint32_t>& refSortedKeys,
		TypedCudaMemory<uint32_t>& refSortedValues,
		std::vector<uint32_t>* resultHostKeys/*= nullptr*/,
		std::vector<uint32_t>* resultHostValues/*= nullptr*/)
	{
		AT_ASSERT(m_deviceKeys);
		AT_ASSERT(m_deviceValues);

		AT_ASSERT(keys.num() == values.num());
		AT_ASSERT(keys.num() <= num);

		// copy unsorted data from host to device
		thrust::device_vector<uint32_t>& deviceKeys = *(thrust::device_vector<uint32_t>*)(m_deviceKeys);
		thrust::device_vector<uint32_t>& deviceValues = *(thrust::device_vector<uint32_t>*)(m_deviceValues);

		auto dkeys = thrust::raw_pointer_cast(deviceKeys.data());
		checkCudaErrors(hipMemcpyAsync(dkeys, keys.ptr(), keys.bytes(), hipMemcpyDeviceToDevice));

		auto dvalues = thrust::raw_pointer_cast(deviceValues.data());
		checkCudaErrors(hipMemcpyAsync(dvalues, values.ptr(), values.bytes(), hipMemcpyDeviceToDevice));

		radixSort(
			num,
			deviceKeys,
			deviceValues,
			refSortedKeys,
			refSortedValues,
			resultHostKeys,
			resultHostValues);
	}
}