#include "hip/hip_runtime.h"
#include "kernel/compaction.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include <vector>
#include <numeric>

// NOTE
// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
// https://github.com/bcrusco/CUDA-Path-Tracer/blob/master/stream_compaction/efficient.cu

// �u���b�N�P�ʂŌv�Z���� exclusiveScan �̑��a�l�𑫂������̂��v�Z����.
__global__ void computeBlockCount(
	int* dst,
	int num,	// block count per grid used in exclusiveScan.
	int stride,	// thread count per block used in exclusiveScan.
	const int* src0, 
	const int* src1)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	if (index == 0) {
		dst[index] = 0;
	}
	else {
		dst[index] = src0[index * stride - 1] + src1[index * stride - 1];
	}
}

// �u���b�N���ƂɑO�̃u���b�N�܂ł� exclusiveScan �̑��a�l�𑫂������̂����Z����.
__global__ void incrementBlocks(
	int* data,
	int num,
	const int* incr)	// value to increment for each blocks.
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	data[index] += incr[blockIdx.x];
}

__global__ void exclusiveScan(int* dst, int num, int stride, const int* src)
{
	extern __shared__ int temp[];

	int index = threadIdx.x;
	int offset = 1;

	auto n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n * 2 >= num) {
		return;
	}

	// Copy input data to shared memory
	temp[2 * index] = src[2 * index + (blockIdx.x * blockDim.x * 2)];
	temp[2 * index + 1] = src[2 * index + 1 + (blockIdx.x * blockDim.x * 2)];

	// Up sweep
	for (int d = stride >> 1; d > 0; d >>= 1) {
		__syncthreads();

		if (index < d) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	// Clear the root
	if (index == 0) {
		temp[stride - 1] = 0;
	}

	// Down sweep
	for (int d = 1; d < stride; d *= 2) {
		offset >>= 1;
		__syncthreads();

		if (index < d && offset > 0) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	// Write to output array
	dst[2 * index + (blockIdx.x * blockDim.x * 2)] = temp[2 * index];
	dst[2 * index + 1 + (blockIdx.x * blockDim.x * 2)] = temp[2 * index + 1];
}

__global__ void scatter(
	int* dst,
	int* count,
	int num,
	const int* bools,
	const int* indices,
	const int* src)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= num) {
		return;
	}

	if (bools[idx] > 0) {
		int pos = indices[idx];
		dst[pos] = src[idx];
	}

	if (idx == 0) {
		*count = bools[num - 1] + indices[num - 1];
	}
}

namespace idaten
{
	static int g_maxInputNum = 0;
	static int g_blockSize = 0;

	idaten::TypedCudaMemory<int> g_increments;
	idaten::TypedCudaMemory<int> g_tmp;
	idaten::TypedCudaMemory<int> g_work;

	idaten::TypedCudaMemory<int> g_indices;
	idaten::TypedCudaMemory<int> g_iota;
	idaten::TypedCudaMemory<int> g_counts;

	void Compaction::init(
		int maxInputNum,
		int blockSize)
	{
		AT_ASSERT(g_maxInputNum == 0);

		if (g_maxInputNum == 0) {
			g_maxInputNum = maxInputNum;
			g_blockSize = blockSize;

			int blockPerGrid = (maxInputNum - 1) / blockSize + 1;

			g_increments.init(blockPerGrid);
			g_tmp.init(blockPerGrid);
			g_work.init(blockPerGrid);

			g_indices.init(g_maxInputNum);

			std::vector<int> iota(g_maxInputNum);
			std::iota(iota.begin(), iota.end(), 0);

			g_iota.init(iota.size());
			g_iota.writeByNum(&iota[0], iota.size());

			g_counts.init(1);
		}
	}

	void Compaction::clear()
	{
		g_maxInputNum = 0;
		g_blockSize = 0;

		g_increments.free();
		g_tmp.free();
		g_work.free();

		g_indices.free();
		g_iota.free();
		g_counts.free();
	}

	void scan(
		const int blocksize,
		idaten::TypedCudaMemory<int>& src,
		idaten::TypedCudaMemory<int>& dst)
	{
		AT_ASSERT(dst.maxNum() <= g_maxInputNum);

		int blockPerGrid = (dst.maxNum() - 1) / blocksize + 1;

		exclusiveScan << <blockPerGrid, blocksize / 2, blocksize * sizeof(int) >> > (
			dst.ptr(),
			dst.maxNum(),
			blocksize,
			src.ptr());

		checkCudaKernel(exclusiveScan);

		if (blockPerGrid <= 1) {
			// If number of block is 1, finish.
			return;
		}

		int tmpBlockPerGrid = (blockPerGrid - 1) / blocksize + 1;
		int tmpBlockSize = blockPerGrid;

		computeBlockCount << <tmpBlockPerGrid, tmpBlockSize >> > (
			g_increments.ptr(),
			g_increments.maxNum(),
			blocksize,
			src.ptr(),
			dst.ptr());

		checkCudaKernel(computeBlockCount);

		idaten::TypedCudaMemory<int>* input = &g_increments;
		idaten::TypedCudaMemory<int>* output = &g_tmp;

		idaten::TypedCudaMemory<int>* tmpptr = &g_tmp;

		int elementNum = blockPerGrid;

		int count = 1;
		int innerBlockPerGrid = 0;

		std::vector<int> stackBlockPerGrid;

		// Scan blocks.
		for (;;) {
			innerBlockPerGrid = (elementNum - 1) / blocksize + 1;
			stackBlockPerGrid.push_back(elementNum);

			exclusiveScan << <innerBlockPerGrid, blocksize / 2, blocksize * sizeof(int) >> >(
				g_work.ptr(),
				g_work.maxNum(),
				blocksize,
				input->ptr());

			checkCudaKernel(iterate_exclusiveScan);

			if (innerBlockPerGrid <= 1) {
				//hipMemcpy(tmp.ptr(), work.ptr(), work.bytes(), hipMemcpyDeviceToDevice);
				tmpptr = &g_work;
				break;
			}

			int innerTmpBlockPerGrid = (innerBlockPerGrid - 1) / blocksize + 1;
			int innerTmpBlockSize = innerBlockPerGrid;

			computeBlockCount << <innerTmpBlockPerGrid, innerTmpBlockSize >> > (
				output->ptr(),
				output->maxNum(),
				blocksize,
				input->ptr(),
				g_work.ptr());

			checkCudaKernel(iterate_computeBlockCount);

			// swap.
			auto p = input;
			input = output;
			output = p;

			elementNum = innerBlockPerGrid;
			count++;
		}

#if 1
		input = tmpptr;
		output = &g_increments;

		for (int i = count - 1; i >= 0; i--) {
			// blocks per grid.
			auto bpg = stackBlockPerGrid[i];

			auto threadPerBlock = (output->maxNum() + bpg - 1) / bpg;

			incrementBlocks << <bpg, threadPerBlock >> > (
				output->ptr(),
				output->maxNum(),
				input->ptr());

			checkCudaKernel(iterate_incrementBlocks);

			// swap.
			auto p = input;
			input = output;
			output = p;
		}

		idaten::TypedCudaMemory<int>* incrResult = (count & 0x1 == 0 ? tmpptr : &g_increments);
#endif

		incrementBlocks << <blockPerGrid, blocksize >> > (
			dst.ptr(),
			dst.maxNum(),
			incrResult->ptr());

		checkCudaKernel(incrementBlocks);
	}

	void Compaction::compact(
		idaten::TypedCudaMemory<int>& dst,
		idaten::TypedCudaMemory<int>& bools,
		int* result/*= nullptr*/)
	{
		scan(g_blockSize, bools, g_indices);

		int num = dst.maxNum();
		int blockPerGrid = (num - 1) / g_blockSize + 1;

		scatter << <blockPerGrid, g_blockSize >> > (
			dst.ptr(),
			g_counts.ptr(),
			dst.maxNum(),
			bools.ptr(),
			g_indices.ptr(),
			g_iota.ptr());

		if (result) {
			g_counts.readByNum(result);
		}
	}

#if 0
	// test implementation.
	void Compaction::compact()
	{
#if 1
		const int blocksize = g_blockSize;

		int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 0, 25, 25, 25 };

		//int c = aten::nextPow2(AT_COUNTOF(f));
		int c = AT_COUNTOF(f);

		std::vector<int> x(c);
		memcpy(&x[0], f, sizeof(int) * AT_COUNTOF(f));

		idaten::TypedCudaMemory<int> src;
		src.init(x.size());
		src.writeByNum(&x[0], x.size());

		idaten::TypedCudaMemory<int> dst;
		dst.init(x.size());

		scan(blocksize, src, dst);

		std::vector<int> buffer(x.size());
		dst.readByNum(&buffer[0]);

		int xxx = 0;
#else
		const int blocksize = g_blockSize;

		int b[] = { 1, 0, 1, 0, 1, 0, 1, 0 };
		int v[] = { 0, 1, 2, 3, 4, 5, 6, 7 };

		AT_ASSERT(AT_COUNTOF(b) == AT_COUNTOF(v));

		int num = AT_COUNTOF(b);

		std::vector<int> buffer(num);

		idaten::TypedCudaMemory<int> bools;
		bools.init(num);
		bools.writeByNum(b, num);

		idaten::TypedCudaMemory<int> indices;
		indices.init(num);

		scan(blocksize, bools, indices);

		indices.readByNum(&buffer[0]);

		idaten::TypedCudaMemory<int> values;
		values.init(num);
		values.writeByNum(v, num);

		idaten::TypedCudaMemory<int> dst;
		dst.init(num);

		idaten::TypedCudaMemory<int> count;
		count.init(1);

		int blockPerGrid = (num - 1) / blocksize + 1;

		scatter << <blockPerGrid, blocksize >> > (
			dst.ptr(),
			count.ptr(),
			dst.maxNum(),
			bools.ptr(),
			indices.ptr(),
			values.ptr());

		dst.readByNum(&buffer[0]);

		int _count = -1;
		count.readByNum(&_count);

		int xxx = 0;
#endif
	}
#endif
}
