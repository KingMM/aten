#include "hip/hip_runtime.h"
#include "kernel/compaction.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

// NOTE
// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
// https://github.com/bcrusco/CUDA-Path-Tracer/blob/master/stream_compaction/efficient.cu

// �u���b�N�P�ʂŌv�Z���� exclusiveScan �̑��a�l�𑫂������̂��v�Z����.
__global__ void computeBlockCount(
	int* dst,
	int num,	// block count per grid used in exclusiveScan.
	int stride,	// thread count per block used in exclusiveScan.
	const int* src0, 
	const int* src1)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	if (index == 0) {
		dst[index] = 0;
	}
	else {
		dst[index] = src0[index * stride - 1] + src1[index * stride - 1];
	}
}

// �u���b�N���ƂɑO�̃u���b�N�܂ł� exclusiveScan �̑��a�l�𑫂������̂����Z����.
__global__ void incrementBlocks(
	int* data,
	int num,
	const int* incr)	// value to increment for each blocks.
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	data[index] += incr[blockIdx.x];
}

__global__ void exclusiveScan(int* dst, int num, int stride, const int* src)
{
	extern __shared__ int temp[];

	int index = threadIdx.x;
	int offset = 1;

	auto n = blockIdx.x * blockDim.x + threadIdx.x;
	if (n * 2 >= num) {
		return;
	}

	// Copy input data to shared memory
	temp[2 * index] = src[2 * index + (blockIdx.x * blockDim.x * 2)];
	temp[2 * index + 1] = src[2 * index + 1 + (blockIdx.x * blockDim.x * 2)];

	// Up sweep
	for (int d = stride >> 1; d > 0; d >>= 1) {
		__syncthreads();

		if (index < d) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	// Clear the root
	if (index == 0) {
		temp[stride - 1] = 0;
	}

	// Down sweep
	for (int d = 1; d < stride; d *= 2) {
		offset >>= 1;
		__syncthreads();

		if (index < d && offset > 0) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	// Write to output array
	dst[2 * index + (blockIdx.x * blockDim.x * 2)] = temp[2 * index];
	dst[2 * index + 1 + (blockIdx.x * blockDim.x * 2)] = temp[2 * index + 1];
}

__global__ void scatter(
	int* dst,
	int* count,
	int num,
	const int* bools,
	const int* indices,
	const int* src)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (idx >= num) {
		return;
	}

	if (bools[idx] > 0) {
		int pos = indices[idx];
		dst[pos] = src[idx];
	}

	if (idx == 0) {
		*count = bools[num - 1] + indices[num - 1];
	}
}

namespace idaten {

	// TODO
	static const int blocksize = 8;

	void scan(
		idaten::TypedCudaMemory<int>& src,
		idaten::TypedCudaMemory<int>& dst)
	{
		int blockPerGrid = (dst.maxNum() - 1) / blocksize + 1;

		exclusiveScan << <blockPerGrid, blocksize / 2, blocksize * sizeof(int) >> > (
			dst.ptr(),
			dst.maxNum(),
			blocksize,
			src.ptr());

		if (blockPerGrid <= 1) {
			return;
		}

		idaten::TypedCudaMemory<int> incr;
		incr.init(blockPerGrid);

		int tmpBlockPerGrid = (blockPerGrid - 1) / blocksize + 1;
		int tmpBlockSize = blockPerGrid;

		computeBlockCount << <tmpBlockPerGrid, tmpBlockSize >> > (
			incr.ptr(),
			incr.maxNum(),
			blocksize,
			src.ptr(),
			dst.ptr());

		idaten::TypedCudaMemory<int> tmp;
		tmp.init(blockPerGrid);

		idaten::TypedCudaMemory<int> work;
		work.init(blockPerGrid);

		idaten::TypedCudaMemory<int>* input = &incr;
		idaten::TypedCudaMemory<int>* output = &tmp;

		int elementNum = blockPerGrid;

		int count = 1;
		int innerBlockPerGrid = 0;

		std::vector<int> stackBlockPerGrid;

		// Scan blocks.
		for (;;) {
			innerBlockPerGrid = (elementNum - 1) / blocksize + 1;
			stackBlockPerGrid.push_back(elementNum);

			exclusiveScan << <innerBlockPerGrid, blocksize / 2, blocksize * sizeof(int) >> >(
				work.ptr(),
				work.maxNum(),
				blocksize,
				input->ptr());

			if (innerBlockPerGrid <= 1) {
				hipMemcpy(tmp.ptr(), work.ptr(), work.bytes(), hipMemcpyDeviceToDevice);
				break;
			}

			int innerTmpBlockPerGrid = (innerBlockPerGrid - 1) / blocksize + 1;
			int innerTmpBlockSize = innerBlockPerGrid;

			computeBlockCount << <innerTmpBlockPerGrid, innerTmpBlockSize >> > (
				output->ptr(),
				output->maxNum(),
				blocksize,
				input->ptr(),
				work.ptr());

			// swap.
			auto p = input;
			input = output;
			output = p;

			elementNum = innerBlockPerGrid;
			count++;
		}

#if 1
		input = &tmp;
		output = &incr;

		for (int i = count - 1; i >= 0; i--) {
			// blocks per grid.
			auto bpg = stackBlockPerGrid[i];

			auto threadPerBlock = (output->maxNum() + bpg - 1) / bpg;

			incrementBlocks << <bpg, threadPerBlock >> > (
				output->ptr(),
				output->maxNum(),
				input->ptr());

			// swap.
			auto p = input;
			input = output;
			output = p;
		}

		idaten::TypedCudaMemory<int>* incrResult = (count & 0x1 == 0 ? &tmp : &incr);
#endif

		incrementBlocks << <blockPerGrid, blocksize >> > (
			dst.ptr(),
			dst.maxNum(),
			incrResult->ptr());
	}

	void compact()
	{
#if 0
		int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 0, 25, 25, 25 };

		//int c = aten::nextPow2(AT_COUNTOF(f));
		int c = AT_COUNTOF(f);

		std::vector<int> x(c);
		memcpy(&x[0], f, sizeof(int) * AT_COUNTOF(f));

		idaten::TypedCudaMemory<int> src;
		src.init(x.size());
		src.writeByNum(&x[0], x.size());

		idaten::TypedCudaMemory<int> dst;
		dst.init(x.size());

		scan(src, dst);

		std::vector<int> buffer(x.size());
		dst.readByNum(&buffer[0]);

		int xxx = 0;
#else
		int b[] = { 1, 0, 1, 0, 1, 0, 1, 0 };
		int v[] = { 0, 1, 2, 3, 4, 5, 6, 7 };

		AT_ASSERT(AT_COUNTOF(b) == AT_COUNTOF(v));

		int num = AT_COUNTOF(b);

		std::vector<int> buffer(num);

		idaten::TypedCudaMemory<int> bools;
		bools.init(num);
		bools.writeByNum(b, num);

		idaten::TypedCudaMemory<int> indices;
		indices.init(num);

		scan(bools, indices);

		indices.readByNum(&buffer[0]);

		idaten::TypedCudaMemory<int> values;
		values.init(num);
		values.writeByNum(v, num);

		idaten::TypedCudaMemory<int> dst;
		dst.init(num);

		idaten::TypedCudaMemory<int> count;
		count.init(1);

		int blockPerGrid = (num - 1) / blocksize + 1;

		scatter << <blockPerGrid, blocksize >> > (
			dst.ptr(),
			count.ptr(),
			dst.maxNum(),
			bools.ptr(),
			indices.ptr(),
			values.ptr());

		dst.readByNum(&buffer[0]);

		int _count = -1;
		count.readByNum(&_count);

		int xxx = 0;
#endif
	}
}
