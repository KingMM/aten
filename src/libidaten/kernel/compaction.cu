#include "hip/hip_runtime.h"
#include "kernel/compaction.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

// NOTE
// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html

// �u���b�N�P�ʂŌv�Z���� exclusiveScan �̑��a�l�𑫂������̂��v�Z����.
__global__ void computeBlockCount(
	int* dst,
	int num,	// block count per grid used in exclusiveScan.
	int stride,	// thread count per block used in exclusiveScan.
	const int* src0, 
	const int* src1)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	if (index == 0) {
		dst[index] = 0;
	}
	else {
		dst[index] = src0[index * stride - 1] + src1[index * stride - 1];
	}
}

// �u���b�N���ƂɑO�̃u���b�N�܂ł� exclusiveScan �̑��a�l�𑫂������̂����Z����.
__global__ void incrementBlocks(
	int* data,
	int num,
	const int* incr)	// value to increment for each blocks.
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index >= num) {
		return;
	}

	data[index] += incr[blockIdx.x];
}

__global__ void exclusiveScan(int* dst, int num, int stride, const int* src)
{
	extern __shared__ int temp[];

	int index = threadIdx.x;
	int offset = 1;

	// Copy input data to shared memory
	temp[2 * index] = src[2 * index + (blockIdx.x * blockDim.x * 2)];
	temp[2 * index + 1] = src[2 * index + 1 + (blockIdx.x * blockDim.x * 2)];

	// Up sweep
	for (int d = stride >> 1; d > 0; d >>= 1) {
		__syncthreads();

		if (index < d) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	// Clear the root
	if (index == 0) {
		temp[stride - 1] = 0;
	}

	// Down sweep
	for (int d = 1; d < stride; d *= 2) {
		offset >>= 1;
		__syncthreads();

		if (index < d && offset > 0) {
			int ai = offset * (2 * index + 1) - 1;
			int bi = offset * (2 * index + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	// Write to output array
	dst[2 * index + (blockIdx.x * blockDim.x * 2)] = temp[2 * index];
	dst[2 * index + 1 + (blockIdx.x * blockDim.x * 2)] = temp[2 * index + 1];
}

namespace idaten {

	void scan(
		idaten::TypedCudaMemory<int>& src,
		idaten::TypedCudaMemory<int>& dst)
	{
		int blocksize = 8;
		int blockPerGrid = (dst.maxNum() - 1) / blocksize + 1;

		exclusiveScan << <blockPerGrid, blocksize / 2, blocksize * sizeof(int) >> > (
			dst.ptr(),
			dst.maxNum(),
			blocksize,
			src.ptr());

		if (blockPerGrid <= 1) {
			return;
		}

		idaten::TypedCudaMemory<int> incr;
		incr.init(blockPerGrid);

		int tmpBlockPerGrid = (blockPerGrid - 1) / blocksize + 1;
		int tmpBlockSize = blockPerGrid;

		computeBlockCount << <tmpBlockPerGrid, tmpBlockSize >> > (
			incr.ptr(),
			incr.maxNum(),
			blocksize,
			src.ptr(),
			dst.ptr());

		idaten::TypedCudaMemory<int> tmp;
		tmp.init(blockPerGrid);

		idaten::TypedCudaMemory<int> work;
		work.init(blockPerGrid);

		idaten::TypedCudaMemory<int>* input = &incr;
		idaten::TypedCudaMemory<int>* output = &tmp;

		int elementNum = blockPerGrid;

		int count = 1;
		int innerBlockPerGrid = 0;

		std::vector<int> stackBlockPerGrid;

		// Scan blocks.
		for (;;) {
			innerBlockPerGrid = (elementNum - 1) / blocksize + 1;
			stackBlockPerGrid.push_back(elementNum);

			exclusiveScan << <innerBlockPerGrid, blocksize / 2, blocksize * sizeof(int) >> >(
				work.ptr(),
				work.maxNum(),
				blocksize,
				input->ptr());

			if (innerBlockPerGrid <= 1) {
				hipMemcpy(tmp.ptr(), work.ptr(), work.bytes(), hipMemcpyDeviceToDevice);
				break;
			}

			int innerTmpBlockPerGrid = (innerBlockPerGrid - 1) / blocksize + 1;
			int innerTmpBlockSize = innerBlockPerGrid;

			computeBlockCount << <innerTmpBlockPerGrid, innerTmpBlockSize >> > (
				output->ptr(),
				output->maxNum(),
				blocksize,
				input->ptr(),
				work.ptr());

			// swap.
			auto p = input;
			input = output;
			output = p;

			elementNum = innerBlockPerGrid;
			count++;
		}

#if 1
		input = &tmp;
		output = &incr;

		for (int i = count - 1; i >= 0; i--) {
			// blocks per grid.
			auto bpg = stackBlockPerGrid[i];

			auto threadPerBlock = (output->maxNum() + bpg - 1) / bpg;

			incrementBlocks << <bpg, threadPerBlock >> > (
				output->ptr(),
				output->maxNum(),
				input->ptr());

			// swap.
			auto p = input;
			input = output;
			output = p;
		}

		idaten::TypedCudaMemory<int>* incrResult = (count & 0x1 == 0 ? &tmp : &incr);
#endif

		incrementBlocks << <blockPerGrid, blocksize >> > (
			dst.ptr(),
			dst.maxNum(),
			incrResult->ptr());
	}

	void compact()
	{
		int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3, 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3, 3, 1 };
		//int f[] = { 3, 1, 7, 0, 4, 1, 6, 3 };
		//int f[] = { 0, 25, 25, 25 };

		//int c = aten::nextPow2(AT_COUNTOF(f));
		int c = AT_COUNTOF(f);

		std::vector<int> x(c);
		memcpy(&x[0], f, sizeof(int) * AT_COUNTOF(f));

		idaten::TypedCudaMemory<int> src;
		src.init(x.size());
		src.writeByNum(&x[0], x.size());

		idaten::TypedCudaMemory<int> dst;
		dst.init(x.size());

		scan(src, dst);
	}
}
