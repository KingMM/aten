#include "hip/hip_runtime.h"
#include "kernel/LBVHBuilder.h"
#include "kernel/RadixSort.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudaTextureResource.h"

#include "accelerator/GpuPayloadDefs.h"

//#pragma optimize( "", off)

// NOTE
// https://github.com/leonardo-domingues/atrbvh

__device__  int computeLongestCommonPrefix(
	const uint32_t* sortedKeys,
	uint32_t numOfElems,
	int index1, int index2,
	uint32_t key1)
{
	// No need to check the upper bound, since i+1 will be at most numberOfElements - 1 (one 
	// thread per internal node)
	if (index2 < 0 || index2 >= numOfElems)
	{
		return 0;
	}

	auto key2 = sortedKeys[index2];

	if (key1 == key2)
	{

		return 32 + __clz(index1 ^ index2);
	}

	auto ret = __clz(key1 ^ key2);

	return ret;
}

__global__ void buildTree(
	uint32_t numOfElems,
	const uint32_t* __restrict__ sortedKeys,
	idaten::LBVHBuilder::LBVHNode* nodes)
{
	const auto i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= numOfElems - 1) {
		return;
	}

	const auto key1 = sortedKeys[i];

	const auto lcp1 = computeLongestCommonPrefix(sortedKeys, numOfElems, i, i + 1, key1);
	const auto lcp2 = computeLongestCommonPrefix(sortedKeys, numOfElems, i, i - 1, key1);

	auto d = (lcp1 - lcp2) < 0 ? -1 : 1;

	// Compute upper bound for the length of the range
	const auto minLcp = computeLongestCommonPrefix(sortedKeys, numOfElems, i, i - d, key1);
	int lMax = 2;
	while (computeLongestCommonPrefix(sortedKeys, numOfElems, i, i + lMax * d, key1) > minLcp)
	{
		lMax *= 2;
	}

	// Find other end using binary search
	int l = 0;
	int t = lMax;
	while (t > 1)
	{
		t = t / 2;
		auto lcp = computeLongestCommonPrefix(sortedKeys, numOfElems, i, i + (l + t) * d, key1);
		if (lcp > minLcp)
		{
			l += t;
		}
	}
	const auto j = i + l * d;

	// Find the split position using binary search
	const auto nodeLcp = computeLongestCommonPrefix(sortedKeys, numOfElems, i, j, key1);
	int s = 0;
	int divisor = 2;
	t = l;
	while (t > 1)
	{
		t = (l + divisor - 1) / divisor;
		auto lcp = computeLongestCommonPrefix(sortedKeys, numOfElems, i, i + (s + t) * d, key1);
		if (lcp > nodeLcp)
		{
			s += t;
		}
		divisor *= 2;
	}

	const auto splitPosition = i + s * d + min(d, 0);

	auto* node = nodes + i;
	if (i == 0) {
		node->parent = -1;
	}
	node->order = i;
	node->isLeaf = false;

	uint32_t leafBaseIdx = numOfElems - 1;

	if (min(i, j) == splitPosition) {
		node->left = leafBaseIdx + splitPosition;

		auto* leaf = nodes + node->left;
		leaf->order = node->left;
		leaf->parent = i;
		leaf->left = -1;
		leaf->right = -1;
		leaf->rangeMin = 0;
		leaf->rangeMax = 0;
		leaf->isLeaf = true;
	}
	else {
		node->left = splitPosition;

		auto* child = nodes + node->left;
		child->order = node->left;
		child->parent = i;
		child->isLeaf = false;
	}

	if (max(i, j) == splitPosition + 1) {
		node->right = leafBaseIdx + splitPosition + 1;

		auto* leaf = nodes + node->right;
		leaf->order = node->right;
		leaf->parent = i;
		leaf->left = -1;
		leaf->right = -1;
		leaf->rangeMin = 0;
		leaf->rangeMax = 0;
		leaf->isLeaf = true;
	}
	else {
		node->right = splitPosition + 1;

		auto* child = nodes + node->right;
		child->order = node->right;
		child->parent = i;
		child->isLeaf = false;
	}

	node->rangeMin = min(i, j);
	node->rangeMax = max(i, j);
}

__device__ __host__ inline void onApplyTraverseOrder(
	int idx,
	int numberOfTris,
	int triIdOffset,
	const idaten::LBVHBuilder::LBVHNode* src,
	aten::ThreadedBvhNode* dst)
{
	const auto* node = &src[idx];

	const idaten::LBVHBuilder::LBVHNode* next = node->left >= 0 ? &src[node->left] : nullptr;

	auto gpunode = &dst[idx];

	gpunode->shapeid = -1;
	gpunode->exid = -1;
	gpunode->meshid = -1;

	if (node->isLeaf) {
		// Base index to convert node index to triangle index.
		int leafBaseIdx = numberOfTris - 1;

		int leafId = node->order - leafBaseIdx;
		int triId = triIdOffset + leafId;

		gpunode->primid = (float)triId;

#if defined(GPGPU_TRAVERSE_SBVH)
		// For ThreadedSbvhNode, this is "isleaf".
		gpunode->shapeid = 1;
#endif
	}
	else {
		gpunode->primid = -1;
	}

	gpunode->hit = -1;
	gpunode->miss = -1;

	bool isOrdered = false;

	if (node->isLeaf) {
		// Hit/Miss.
		// Always sibling.

		// The leaf has parent surely.
		auto parent = &src[node->parent];

		auto left = parent->left >= 0 ? &src[parent->left] : nullptr;
		auto right = parent->right >= 0 ? &src[parent->right] : nullptr;

		if (left == node) {
			// Sibling.
			gpunode->hit = (float)right->order;
			gpunode->miss = (float)right->order;

			isOrdered = true;
		}
	}
	else {
		// Hit.
		// Always the next node in the array.
		if (next) {
			gpunode->hit = (float)next->order;
		}
		else {
			gpunode->hit = -1;
		}
	}

	if (!isOrdered)
	{
		// Miss.

		// Search the parent.
		auto parentId = node->parent;
		const auto parent = (parentId >= 0
			? &src[parentId]
			: nullptr);

		if (parent) {
			const auto left = parent->left >= 0 ? &src[parent->left] : nullptr;
			const auto right = parent->right >= 0 ? &src[parent->right] : nullptr;

			if (left == node && right) {
				// Traverse to sibling (= parent's right)
				auto sibling = right;
				gpunode->miss = (float)sibling->order;
			}
			else {
				auto curParent = parent;

				// Traverse to ancester's right.
				for (;;) {
					// Search the grand parent.
					auto grandParentId = curParent->parent;
					const auto grandParent = (grandParentId >= 0
						? &src[grandParentId]
						: nullptr);

					if (grandParent) {
						const auto _left = grandParent->left >= 0 ? &src[grandParent->left] : nullptr;
						const auto _right = grandParent->right >= 0 ? &src[grandParent->right] : nullptr;

						auto sibling = _right;

						if (sibling) {
							if (sibling != curParent) {
								gpunode->miss = (float)sibling->order;

								if (node->isLeaf && gpunode->hit < 0) {
									gpunode->hit = (float)sibling->order;
								}

								break;
							}
						}
					}
					else {
						gpunode->miss = -1;
						break;
					}

					curParent = grandParent;
				}
			}
		}
		else {
			gpunode->miss = -1;
		}
	}
}

__global__ void applyTraverseOrder(
	uint32_t numberOfNodes,
	int numberOfTris,
	int triIdOffset,
	const idaten::LBVHBuilder::LBVHNode* __restrict__ src,
	aten::ThreadedBvhNode* dst)
{
	const auto idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= numberOfNodes) {
		return;
	}

	onApplyTraverseOrder(idx, numberOfTris, triIdOffset, src, dst);
}

__device__ inline void computeBoundingBox(
	float4 v0,
	float4 v1,
	float4 v2,
	float4* aabbMin,
	float4* aabbMax)
{
	aabbMin->x = min(min(v0.x, v1.x), v2.x);
	aabbMin->y = min(min(v0.y, v1.y), v2.y);
	aabbMin->z = min(min(v0.z, v1.z), v2.z);

	aabbMax->x = max(max(v0.x, v1.x), v2.x);
	aabbMax->y = max(max(v0.y, v1.y), v2.y);
	aabbMax->z = max(max(v0.z, v1.z), v2.z);
}

__device__ inline void computeBoundingBox(
	float4 bboxMin_0,
	float4 bboxMax_0,
	float4 bboxMin_1,
	float4 bboxMax_1,
	float4* aabbMin,
	float4* aabbMax)
{
	aabbMin->x = min(bboxMin_0.x, bboxMin_1.x);
	aabbMin->y = min(bboxMin_0.y, bboxMin_1.y);
	aabbMin->z = min(bboxMin_0.z, bboxMin_1.z);

	aabbMax->x = max(bboxMax_0.x, bboxMax_1.x);
	aabbMax->y = max(bboxMax_0.y, bboxMax_1.y);
	aabbMax->z = max(bboxMax_0.z, bboxMax_1.z);
}

__forceinline__ __device__ float4 getFloat4(hipTextureObject_t tex, int idx)
{
	return tex1Dfetch<float4>(tex, idx);
}

__forceinline__ __device__ float4 getFloat4(float4* data, int idx)
{
	return data[idx];
}

template <typename T>
__global__ void computeBoudingBox(
	int numberOfTris,
	const idaten::LBVHBuilder::LBVHNode* __restrict__ src,
	aten::ThreadedBvhNode* dst,
	const aten::PrimitiveParamter* __restrict__ tris,
	T vtxPos,
	uint32_t* executedIdxArray)
{
	const int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	const int firstThreadIdxInBlock = blockIdx.x * blockDim.x;
	const int lastThreadIdxInBlock = firstThreadIdxInBlock + blockDim.x - 1;

	// Initialize cache of bounding boxes in shared memory
	extern __shared__ float4 sharedBboxMin[];
	__shared__ float4* sharedBboxMax;
	if (threadIdx.x == 0)
	{
		sharedBboxMax = sharedBboxMin + blockDim.x;
	}
	__syncthreads();

	// Check for valid threads
	if (idx >= numberOfTris)
	{
		return;
	}

	// NOTE
	// Number of Internal Nodes = Number of Triangles - 1.
	int leafNodeIdx = idx + numberOfTris - 1;

	// Base index to convert node index to triangle index.
	int leafBaseIdx = numberOfTris - 1;

	const auto* node = &src[leafNodeIdx];
	auto* gpunode = &dst[leafNodeIdx];

	// Calculate leaves bounding box.
	int leafId = node->order - leafBaseIdx;
	int triId = leafId;

	aten::PrimitiveParamter prim;
	prim.v0 = ((aten::vec4*)tris)[triId * aten::PrimitiveParamter_float4_size + 0];

	float4 v0 = getFloat4(vtxPos, prim.idx[0]);
	float4 v1 = getFloat4(vtxPos, prim.idx[1]);
	float4 v2 = getFloat4(vtxPos, prim.idx[2]);

	float4 aabbMin, aabbMax;
	computeBoundingBox(v0, v1, v2, &aabbMin, &aabbMax);

	// Keep bouding box to shared memory.
	sharedBboxMin[threadIdx.x] = aabbMin;
	sharedBboxMax[threadIdx.x] = aabbMax;

	gpunode->boxmin = aten::vec3(aabbMin.x, aabbMin.y, aabbMin.z);
	gpunode->boxmax = aten::vec3(aabbMax.x, aabbMax.y, aabbMax.z);

#if 0
	printf("Vtx(%d : %d %d %d) [%f, %f, %f] [%f, %f, %f] [%f, %f, %f]\n", 
		triId,
		prim.idx[0], prim.idx[1], prim.idx[2],
		v0.x, v0.y, v0.z, v1.x, v1.y, v1.z, v2.x, v2.y, v2.z);
	printf("Target[%d] [%f, %f, %f] [%f, %f, %f]\n", leafNodeIdx, aabbMin.x, aabbMin.y, aabbMin.z, aabbMax.x, aabbMax.y, aabbMax.z);
#endif

	__syncthreads();

	// ���[�t����e�ւ��ǂ��Ă���.

	int lastNode = idx;
	int targetId = node->parent;

	while (targetId >= 0)
	{
		// �^�[�Q�b�g�͐e�m�[�h�ŁA�����ł͎q�m�[�h���������Ă���ł��낤�X���b�h�̃C���f�b�N�X���擾����.
		// �C���f�b�N�X�̔z��� 0xffffffff �ŏ���������Ă��āA�������ꂽ��X���b�h�̃C���f�b�N�X�Œu�������.
		// �܂�A�z����̒l�� 0xffffffff �ł�������A�������Ƃ������ƂɂȂ�.
		const auto childNodeThreadIdx = atomicExch(&executedIdxArray[targetId], idx);

		if (childNodeThreadIdx == 0xffffffff) {
			// �������Ȃ̂ŁA����ȏ�͉������Ȃ�.
			return;
		}

		auto* targetSrc = &src[targetId];
		auto* targetDst = &dst[targetId];

		float4 childAABBMin, childAABBMax;

		// �q�m�[�h���������Ă���ł��낤�X���b�h������u���b�N�ŏ������Ă��邩�ǂ���.
		if (firstThreadIdxInBlock <= childNodeThreadIdx
			&& childNodeThreadIdx <= lastThreadIdxInBlock)
		{
			// ����u���b�N�ŏ�������Ă���̂ŁAshared memory �ɃL���b�V������Ă���f�[�^���擾����.

			// �u���b�N���ł̃X���b�hID�ɕϊ�.
			int threadIdxInBlock = childNodeThreadIdx - firstThreadIdxInBlock;

			childAABBMin = sharedBboxMin[threadIdxInBlock];
			childAABBMax = sharedBboxMax[threadIdxInBlock];
		}
		else {
			// ����u���b�N�ŏ�������Ă��Ȃ��̂ŁA�z��Ɋi�[����Ă���f�[�^���擾����.

			int childIdx = targetSrc->left;

			if (childIdx == lastNode) {
				childIdx = targetSrc->right;
			}

			auto* tmp = &dst[childIdx];

			childAABBMin = make_float4(tmp->boxmin.x, tmp->boxmin.y, tmp->boxmin.z, 0);
			childAABBMax = make_float4(tmp->boxmax.x, tmp->boxmax.y, tmp->boxmax.z, 0);
		}

		__syncthreads();

		computeBoundingBox(
			aabbMin, aabbMax,
			childAABBMin, childAABBMax,
			&aabbMin, &aabbMax);

		// Keep bouding box to shared memory.
		sharedBboxMin[threadIdx.x] = aabbMin;
		sharedBboxMax[threadIdx.x] = aabbMax;

		targetDst->boxmin = aten::vec3(aabbMin.x, aabbMin.y, aabbMin.z);
		targetDst->boxmax = aten::vec3(aabbMax.x, aabbMax.y, aabbMax.z);

		//printf("Target[%d] [%f, %f, %f] [%f, %f, %f]\n", targetId, aabbMin.x, aabbMin.y, aabbMin.z, aabbMax.x, aabbMax.y, aabbMax.z);

		__syncthreads();

		// Update last processed node
		lastNode = targetId;

		// Update target node pointer
		targetId = targetSrc->parent;
	}
}

__forceinline__ __device__ unsigned int expandBits(unsigned int value)
{
	value = (value * 0x00010001u) & 0xFF0000FFu;
	value = (value * 0x00000101u) & 0x0F00F00Fu;
	value = (value * 0x00000011u) & 0xC30C30C3u;
	value = (value * 0x00000005u) & 0x49249249u;
	return value;
}

__forceinline__ __device__ unsigned int computeMortonCode(aten::vec3 point)
{
	// Discretize the unit cube into a 10 bit integer
	uint3 discretized;
	discretized.x = (unsigned int)min(max(point.x * 1024.0f, 0.0f), 1023.0f);
	discretized.y = (unsigned int)min(max(point.y * 1024.0f, 0.0f), 1023.0f);
	discretized.z = (unsigned int)min(max(point.z * 1024.0f, 0.0f), 1023.0f);

	discretized.x = expandBits(discretized.x);
	discretized.y = expandBits(discretized.y);
	discretized.z = expandBits(discretized.z);

	return discretized.x * 4 + discretized.y * 2 + discretized.z;
}

template <typename T>
__global__ void genMortonCode(
	int numberOfTris,
	const aten::aabb sceneBbox,
	const aten::PrimitiveParamter* __restrict__ tris,
	T vtxPos,
	uint32_t* mortonCodes,
	uint32_t* indices)
{
	const auto idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx >= numberOfTris) {
		return;
	}

	aten::PrimitiveParamter prim;
	prim.v0 = ((aten::vec4*)tris)[idx * aten::PrimitiveParamter_float4_size + 0];

	float4 v0 = getFloat4(vtxPos, prim.idx[0]);
	float4 v1 = getFloat4(vtxPos, prim.idx[1]);
	float4 v2 = getFloat4(vtxPos, prim.idx[2]);

	aten::vec3 vmin = aten::vec3(
		min(min(v0.x, v1.x), v2.x),
		min(min(v0.y, v1.y), v2.y),
		min(min(v0.z, v1.z), v2.z));

	aten::vec3 vmax = aten::vec3(
		max(max(v0.x, v1.x), v2.x),
		max(max(v0.y, v1.y), v2.y),
		max(max(v0.z, v1.z), v2.z));

	aten::vec3 center = (vmin + vmax) * 0.5f;

	// Normalize [0, 1].
	const auto size = sceneBbox.size();
	const auto bboxMin = sceneBbox.minPos();
	center = (center - bboxMin) / size;

	auto code = computeMortonCode(center);

	mortonCodes[idx] = code;
	indices[idx] = idx;
}

namespace idaten
{
	template <typename T>
	void onBuild(
		idaten::CudaTextureResource& dst,
		std::vector<aten::PrimitiveParamter>& tris,
		int triIdOffset,
		const aten::aabb& sceneBbox,
		T vtxPos,
		std::vector<aten::ThreadedBvhNode>* threadedBvhNodes)
	{
		TypedCudaMemory<aten::PrimitiveParamter> triangles;
		TypedCudaMemory<uint32_t> mortonCodes;
		TypedCudaMemory<uint32_t> indices;

		uint32_t numOfElems = (uint32_t)tris.size();

		triangles.init(numOfElems);
		mortonCodes.init(numOfElems);
		indices.init(numOfElems);

		// Compute morton code.
		{
			triangles.writeByNum(&tris[0], (uint32_t)tris.size());

			uint32_t numberOfTris = triangles.maxNum();

			dim3 block(256, 1, 1);
			dim3 grid((numberOfTris + block.x - 1) / block.x, 1, 1);

			genMortonCode << <grid, block >> > (
				numberOfTris,
				sceneBbox,
				triangles.ptr(),
				vtxPos,
				mortonCodes.ptr(),
				indices.ptr());

			checkCudaKernel(genMortonCode);
		}

		// Radix sort.
		TypedCudaMemory<uint32_t> sortedKeys;
		std::vector<uint32_t> v;
		RadixSort::sort(mortonCodes, indices, sortedKeys, &v);

		uint32_t numInternalNode = numOfElems - 1;
		uint32_t numLeaves = numOfElems;

		TypedCudaMemory<LBVHBuilder::LBVHNode> nodesLbvh;
		nodesLbvh.init(numInternalNode + numLeaves);

		// Build tree.
		{
			dim3 block(256, 1, 1);
			dim3 grid((numOfElems + block.x - 1) / block.x, 1, 1);

			buildTree << <grid, block >> > (
				numOfElems,
				sortedKeys.ptr(),
				nodesLbvh.ptr());

			checkCudaKernel(buildTree);
		}

		TypedCudaMemory<aten::ThreadedBvhNode> nodes;
		nodes.init(numInternalNode + numLeaves);

		// Convert to gpu bvh tree nodes.
		{
			numOfElems = numInternalNode + numLeaves;

			dim3 block(128, 1, 1);
			dim3 grid((numOfElems + block.x - 1) / block.x, 1, 1);

			applyTraverseOrder << <grid, block >> > (
				numOfElems,
				numLeaves,
				triIdOffset,
				nodesLbvh.ptr(),
				nodes.ptr());

			checkCudaKernel(applyTraverseOrder);
		}

		// Compute bouding box.
		{
			uint32_t numberOfTris = triangles.maxNum();

			uint32_t* executedIdxArray;
			checkCudaErrors(hipMalloc(&executedIdxArray, (numberOfTris - 1) * sizeof(uint32_t)));
			checkCudaErrors(hipMemset(executedIdxArray, 0xFF, (numberOfTris - 1) * sizeof(uint32_t)));

			dim3 block(128, 1, 1);
			dim3 grid((numberOfTris + block.x - 1) / block.x, 1, 1);

			size_t sharedMemorySize = block.x * sizeof(float4) * 2;

			computeBoudingBox << <grid, block, sharedMemorySize >> > (
				numberOfTris,
				nodesLbvh.ptr(),
				nodes.ptr(),
				triangles.ptr(),
				vtxPos,
				executedIdxArray);

			checkCudaKernel(computeBoudingBox);

			checkCudaErrors(hipFree(executedIdxArray));
		}

		if (threadedBvhNodes) {
			threadedBvhNodes->resize(nodes.maxNum());
			nodes.read(&(*threadedBvhNodes)[0], 0);
		}

		dst.initFromDeviceMemory(
			(aten::vec4*)nodes.ptr(),
			sizeof(aten::ThreadedBvhNode) / sizeof(float4),
			nodes.maxNum());

		
	}

	void LBVHBuilder::build(
		idaten::CudaTextureResource& dst,
		std::vector<aten::PrimitiveParamter>& tris,
		int triIdOffset,
		const aten::aabb& sceneBbox,
		idaten::CudaTextureResource& texRscVtxPos,
		std::vector<aten::ThreadedBvhNode>* threadedBvhNodes/*= nullptr*/)
	{		
		auto vtxPos = texRscVtxPos.bind();

		onBuild(dst, tris, triIdOffset, sceneBbox, vtxPos, threadedBvhNodes);

		texRscVtxPos.unbind();
	}

	void LBVHBuilder::build(
		idaten::CudaTextureResource& dst,
		std::vector<aten::PrimitiveParamter>& tris,
		int triIdOffset,
		const aten::aabb& sceneBbox,
		CudaGLBuffer& vboVtxPos,
		std::vector<aten::ThreadedBvhNode>* threadedBvhNodes/*= nullptr*/)
	{
		vboVtxPos.map();

		float4* vtxPos = nullptr;
		size_t bytes = 0;
		vboVtxPos.bind((void**)&vtxPos, bytes);

		onBuild(dst, tris, triIdOffset, sceneBbox, vtxPos, threadedBvhNodes);

		vboVtxPos.unbind();
		vboVtxPos.unmap();
	}

	void LBVHBuilder::build()
	{
		static const uint32_t keys[] = {
			1, 2, 4, 5, 19, 24, 25, 30,
		};

		std::vector<uint32_t> values;
		for (auto k : keys) {
			values.push_back(k);
		}

		TypedCudaMemory<uint32_t> sortedKeys;
		std::vector<uint32_t> v;
		RadixSort::sort(values, sortedKeys, &v);

		uint32_t numOfElems = values.size();

		uint32_t numInternalNode = numOfElems - 1;
		uint32_t numLeaves = numOfElems;

		TypedCudaMemory<LBVHNode> nodesLbvh;
		nodesLbvh.init(numInternalNode + numLeaves);

		{
			dim3 block(256, 1, 1);
			dim3 grid((numOfElems + block.x - 1) / block.x, 1, 1);

			buildTree << <grid, block >> > (
				numOfElems,
				sortedKeys.ptr(),
				nodesLbvh.ptr());
		}

		TypedCudaMemory<aten::ThreadedBvhNode> nodes;
		nodes.init(numInternalNode + numLeaves);

#if 1
		{
			numOfElems = numInternalNode + numLeaves;

			dim3 block(256, 1, 1);
			dim3 grid((numOfElems + block.x - 1) / block.x, 1, 1);

			applyTraverseOrder << <grid, block >> > (
				numOfElems,
				numLeaves,
				0,
				nodesLbvh.ptr(),
				nodes.ptr());
		}
		std::vector<aten::ThreadedBvhNode> tmp1(nodes.maxNum());
		nodes.read(&tmp1[0], 0);
#else
		std::vector<LBVHNode> tmp(m_nodesLbvh.maxNum());
		m_nodesLbvh.read(&tmp[0], 0);

		std::vector<aten::ThreadedBvhNode> tmp1(m_nodes.maxNum());

		for (int n = 0; n < numInternalNode + numLeaves; n++) {
			onApplyTraverseOrder(n, &tmp[0], &tmp1[0]);
		}
#endif
	}
}