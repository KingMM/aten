#include "hip/hip_runtime.h"
#include "kernel/pathtracing.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void renderAOV(
	float4* aovs,
	int width, int height,
	int sample, int maxSamples,
	int seed,
	const aten::CameraParameter* __restrict__ camera,
	const aten::ShapeParameter* __restrict__ shapes, int geomnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	const aten::mat4* __restrict__ matrices,
	const unsigned int* sobolmatrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	aten::sampler sampler;
	sampler.init((iy * height * 4 + ix * 4) * maxSamples + sample + 1 + seed, sobolmatrices);

	float s = (ix + sampler.nextSample()) / (float)(camera->width);
	float t = (iy + sampler.nextSample()) / (float)(camera->height);

	AT_NAME::CameraSampleResult camsample;
	AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	aten::Intersection isect;

	bool isHit = intersectBVH(&ctxt, camsample.r, &isect);

	aovs[idx].x = isHit ? isect.mtrlid : -1;		// material id.
	aovs[idx].y = isHit ? isect.t : AT_MATH_INF;	// depth.
}

enum ReferPos {
	UpperLeft,
	LowerLeft,
	UpperRight,
	LowerRight,
};

__global__ void geometryRender(
	const idaten::PathTracing::Path* __restrict__ paths,
	const float4* __restrict__ aovs,
	hipSurfaceObject_t outSurface,
	int width, int height,
	int mwidth, int mheight)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	static const int ratio = 2;

	if (ix >= width && iy >= height) {
		return;
	}

	int mx = ix / (float)ratio;
	int my = iy / (float)ratio;

	// NOTE
	// +y
	// |
	// |
	// 0 ---> +x

	// NOTE
	// ul
	// +y ------- ur
	// |          |
	// |          |
	// ll ---- +x lr

	int2 pos[4] = {
		make_int2(mx, min(my + 1, mheight - 1)),						// upper-left.
		make_int2(mx, my),												// lower-left.
		make_int2(min(mx + 1, mwidth - 1), min(my + 1, mheight - 1)),	// upper-right.
		make_int2(min(mx + 1, mwidth - 1), my),							// lower-right.
	};

	// ��_�i�����j����̔䗦���v�Z.
	real u = aten::abs<int>(ix - pos[ReferPos::LowerLeft].x * ratio) / (real)ratio;
	real v = aten::abs<int>(iy - pos[ReferPos::LowerLeft].y * ratio) / (real)ratio;

	u = aten::clamp(u, AT_MATH_EPSILON, real(1));
	v = aten::clamp(v, AT_MATH_EPSILON, real(1));

	int refmidx = getIdx(ix, iy, width);
	const int mtrlIdx = (int)aovs[refmidx].x;

	real norms[4] = {
		1 / (u * (1 - v)),
		1 / (u * v),
		1 / ((1 - u) * (1 - v)),
		1 / ((1 - u) * v),
	};

	real sumWeight = 0;

	aten::vec3 denom;
	
	for (int i = 0; i < 4; i++) {
		auto midx = getIdx(pos[i].x * ratio, pos[i].y * ratio, width);
		int refMtrlIdx = (int)aovs[midx].x;

		int coeff = (mtrlIdx == refMtrlIdx ? 1 : 0);
		auto weight = norms[i] * coeff;;

		auto cidx = getIdx(pos[i].x, pos[i].y, mwidth);

		sumWeight += weight;
		denom += paths[cidx].contrib / (real)paths[cidx].samples * weight;
	}

	denom = denom / (sumWeight + AT_MATH_EPSILON);

	float4 data;
#if 1
	surf2Dread(&data, outSurface, ix * sizeof(float4), iy);

	// First data.w value is 0.
	int n = data.w;
	data = n * data + make_float4(denom.x, denom.y, denom.z, 0);
	data /= (n + 1);
	data.w = n + 1;
#else
	data = make_float4(denom.x, denom.y, denom.z, 1);
#endif

	surf2Dwrite(
		data,
		outSurface,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten
{
	void PathTracingGeometryRendering::update(
		GLuint gltex,
		int width, int height,
		const aten::CameraParameter& camera,
		const std::vector<aten::ShapeParameter>& shapes,
		const std::vector<aten::MaterialParameter>& mtrls,
		const std::vector<aten::LightParameter>& lights,
		const std::vector<std::vector<aten::BVHNode>>& nodes,
		const std::vector<aten::PrimitiveParamter>& prims,
		const std::vector<aten::vertex>& vtxs,
		const std::vector<aten::mat4>& mtxs,
		const std::vector<TextureResource>& texs,
		const EnvmapResource& envmapRsc)
	{
		idaten::PathTracing::update(
			gltex,
			width, height,
			camera,
			shapes,
			mtrls,
			lights,
			nodes,
			prims,
			vtxs,
			mtxs,
			texs, envmapRsc);

		// TODO
		m_aovs[0].init((width << 1) * (height << 1));
		m_aovs[1].init((width << 1) * (height << 1));
	}

	void PathTracingGeometryRendering::onGenPath(
		int width, int height,
		int sample, int maxSamples,
		int seed,
		hipTextureObject_t texVtxPos)
	{
		idaten::PathTracing::onGenPath(
			width, height,
			sample, maxSamples,
			seed,
			texVtxPos);

		if (sample == 0) {
			renderAOVs(
				width, height,
				sample, maxSamples,
				seed,
				texVtxPos);
		}
	}

	void PathTracingGeometryRendering::renderAOVs(
		int width, int height,
		int sample, int maxSamples,
		int seed,
		hipTextureObject_t texVtxPos)
	{
		int W = width;
		int H = height;

		getRenderAOVSize(W, H);

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(W + block.x - 1) / block.x,
			(H + block.y - 1) / block.y);

		auto& aovs = getCurAOVs();

		renderAOV << <grid, block >> > (
			//renderAOV << <1, 1 >> > (
			aovs.ptr(),
			W, H,
			sample, maxSamples,
			seed,
			cam.ptr(),
			shapeparam.ptr(), shapeparam.num(),
			nodetex.ptr(),
			primparams.ptr(),
			texVtxPos,
			mtxparams.ptr(),
			m_sobolMatrices.ptr());

		checkCudaKernel(renderAOV);
	}

	void PathTracingGeometryRendering::onGather(
		hipSurfaceObject_t outputSurf,
		int width, int height,
		int maxSamples)
	{
		int mwidth = width;
		int mheight = height;

		width <<= 1;
		height <<= 1;

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		auto& aovs = getCurAOVs();

		geometryRender << <grid, block >> > (
		//geometryRender << <1, 1 >> > (
			paths.ptr(),
			aovs.ptr(),
			outputSurf,
			width, height,
			mwidth, mheight);

		checkCudaKernel(geometryRender);
	}
}
