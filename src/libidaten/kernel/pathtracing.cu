#include "hip/hip_runtime.h"
#include "kernel/pathtracing.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/common.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

struct Path {
	aten::ray ray;
	aten::vec3 throughput;
	aten::hitrecord rec;
	aten::sampler sampler;
	bool isHit;
	bool isTerminate;
};

__global__ void genPath(
	Path* paths,
	int width, int height,
	int sample, int maxSamples,
	int seed,
	aten::CameraParameter* camera)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * camera->width + ix;

	auto& path = paths[idx];
	path.sampler.init((iy * height * 4 + ix * 4) * maxSamples + sample + 1 + seed);

	float s = (ix + path.sampler.nextSample()) / (float)(camera->width);
	float t = (iy + path.sampler.nextSample()) / (float)(camera->height);

	auto camsample = AT_NAME::PinholeCamera::sample(*camera, s, t, nullptr);

	path.ray = camsample.r;
	path.throughput = aten::vec3(1);
	path.isHit = false;
	path.isTerminate = false;
}

__global__ void hitTest(
	Path* paths,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	aten::BVHNode* nodes,
	aten::PrimitiveParamter* prims,
	aten::vertex* vertices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
	}
	
	aten::hitrecord rec;
	bool isHit = intersectBVH(&ctxt, path.ray, AT_MATH_EPSILON, AT_MATH_INF, &rec);

	path.isHit = isHit;
	path.rec = rec;
}

__global__ void shade(
	hipSurfaceObject_t outSurface,
	Path* paths,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	aten::BVHNode* nodes,
	aten::PrimitiveParamter* prims,
	aten::vertex* vertices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];

	if (!path.isHit) {
		return;
	}
	if (path.isTerminate) {
		return;
	}

	aten::vec3 contrib(0);

	const aten::MaterialParameter* mtrl = &ctxt.mtrls[path.rec.mtrlid];

	if (mtrl->attrib.isEmissive) {
		contrib = path.throughput * mtrl->baseColor;

		path.isTerminate = true;
		//p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
		surf2Dwrite(make_float4(contrib.x, contrib.y, contrib.z, 1), outSurface, ix * sizeof(float4), iy, hipBoundaryModeTrap);
		
		return;
	}

	// �����ʒu�̖@��.
	// ���̂���̃��C�̓��o���l��.
	const aten::vec3 orienting_normal = dot(path.rec.normal, path.ray.dir) < 0.0 ? path.rec.normal : -path.rec.normal;

	AT_NAME::MaterialSampling sampling;
			
	sampleMaterial(
		&sampling,
		mtrl,
		orienting_normal, 
		path.ray.dir,
		path.rec.normal,
		&path.sampler,
		path.rec.u, path.rec.v);

	auto nextDir = normalize(sampling.dir);

	path.throughput *= sampling.bsdf;

	// Make next ray.
	path.ray = aten::ray(path.rec.p, nextDir);
}

namespace idaten {
	void PathTracing::prepare()
	{
		addFuncs();
	}

	void PathTracing::update(
		GLuint gltex,
		int width, int height,
		const aten::CameraParameter& camera,
		const std::vector<aten::ShapeParameter>& shapes,
		const std::vector<aten::MaterialParameter>& mtrls,
		const std::vector<aten::LightParameter>& lights,
		const std::vector<aten::BVHNode>& nodes,
		const std::vector<aten::PrimitiveParamter>& prims,
		const std::vector<aten::vertex>& vtxs)
	{
#if 0
		size_t size_stack = 0;
		checkCudaErrors(cudaThreadGetLimit(&size_stack, hipLimitStackSize));
		checkCudaErrors(cudaThreadSetLimit(hipLimitStackSize, 12928));
		checkCudaErrors(cudaThreadGetLimit(&size_stack, hipLimitStackSize));

		AT_PRINTF("Stack size %d\n", size_stack);
#endif

#if 0
		dst.init(sizeof(float4) * width * height);
#else
		glimg.init(gltex, CudaGLRscRegisterType::WriteOnly);
#endif

		cam.init(sizeof(camera));
		cam.writeByNum(&camera, 1);

		shapeparam.init(shapes.size());
		shapeparam.writeByNum(&shapes[0], shapes.size());

		mtrlparam.init(mtrls.size());
		mtrlparam.writeByNum(&mtrls[0], mtrls.size());

		lightparam.init(lights.size());
		lightparam.writeByNum(&lights[0], lights.size());

		nodeparam.init(nodes.size());
		nodeparam.writeByNum(&nodes[0], nodes.size());

		if (!prims.empty()) {
			primparams.init(prims.size());
			primparams.writeByNum(&prims[0], prims.size());
		}

		if (!vtxs.empty()) {
			vtxparams.init(vtxs.size());
			vtxparams.writeByNum(&vtxs[0], vtxs.size());
		}
	}

#include "misc/timer.h"
	aten::SystemTime getSystemTime()
	{
		SYSTEMTIME time;
		::GetSystemTime(&time);

		aten::SystemTime ret;
		ret.year = time.wYear;
		ret.month = time.wMonth;
		ret.dayOfWeek = time.wDayOfWeek;
		ret.day = time.wDay;
		ret.hour = time.wHour;
		ret.minute = time.wMinute;
		ret.second = time.wSecond;
		ret.milliSeconds = time.wMilliseconds;

		return std::move(ret);
	}

	void PathTracing::render(
		aten::vec4* image,
		int width, int height)
	{
		dim3 block(16, 16);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int depth = 0;

		idaten::TypedCudaMemory<Path> paths;
		paths.init(width * height);

		CudaGLResourceMap rscmap(&glimg);
		auto outputSurf = glimg.bindToWrite();

		static const int maxSamples = 5;

		auto time = getSystemTime();

		for (int i = 0; i < maxSamples; i++) {
			genPath << <grid, block >> > (
				paths.ptr(),
				width, height,
				i, maxSamples,
				time.milliSeconds,
				cam.ptr());

			//checkCudaErrors(hipDeviceSynchronize());

			while (depth < 5) {
				hitTest << <grid, block >> > (
					paths.ptr(),
					width, height,
					shapeparam.ptr(), shapeparam.num(),
					mtrlparam.ptr(),
					lightparam.ptr(), lightparam.num(),
					nodeparam.ptr(),
					primparams.ptr(),
					vtxparams.ptr());

				auto err = hipGetLastError();
				if (err != hipSuccess) {
					AT_PRINTF("Cuda Kernel Err(hitTest) [%s]\n", hipGetErrorString(err));
				}

				//checkCudaErrors(hipDeviceSynchronize());

				shade << <grid, block >> > (
					//(float4*)dst.ptr(),
					outputSurf,
					paths.ptr(),
					width, height,
					shapeparam.ptr(), shapeparam.num(),
					mtrlparam.ptr(),
					lightparam.ptr(), lightparam.num(),
					nodeparam.ptr(),
					primparams.ptr(),
					vtxparams.ptr());

				err = hipGetLastError();
				if (err != hipSuccess) {
					AT_PRINTF("Cuda Kernel Err(raytracing) [%s]\n", hipGetErrorString(err));
				}

				//checkCudaErrors(hipDeviceSynchronize());

				depth++;
			}

			checkCudaErrors(hipDeviceSynchronize());
		}
		//dst.read(image, sizeof(aten::vec4) * width * height);
	}
}