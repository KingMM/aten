#include "hip/hip_runtime.h"
#include "kernel/pathtracing.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/bvh.cuh"
#include "kernel/common.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

struct ShadowRay : public aten::ray {
	aten::vec3 lightcontrib;
	real distToLight;
	int targetLightId;

	struct {
		uint32_t isActive : 1;
	};
};

struct Path {
	aten::vec3 throughput;
	aten::vec3 contrib;
	aten::hitrecord rec;
	aten::sampler sampler;
	
	int mtrlid;

	real pdfb;

	bool isHit;
	bool isTerminate;
};

#define BLOCK_SIZE	(16)
#define BLOCK_SIZE2	(BLOCK_SIZE * BLOCK_SIZE)

inline AT_DEVICE_API int getIdx(int ix, int iy, int width)
{
	int X = ix / BLOCK_SIZE;
	int Y = iy / BLOCK_SIZE;

	//int base = Y * BLOCK_SIZE2 * (width / BLOCK_SIZE) + X * BLOCK_SIZE2;

	int XB = X * BLOCK_SIZE;
	int YB = Y * BLOCK_SIZE;

	int base = YB * width + XB * BLOCK_SIZE;

	const auto idx = base + (iy - YB) * BLOCK_SIZE + (ix - XB);

	return idx;
}

__global__ void genPath(
	Path* paths,
	aten::ray* rays,
	int width, int height,
	int sample, int maxSamples,
	int seed,
	aten::CameraParameter* camera)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.sampler.init((iy * height * 4 + ix * 4) * maxSamples + sample + 1 + seed);

	float s = (ix + path.sampler.nextSample()) / (float)(camera->width);
	float t = (iy + path.sampler.nextSample()) / (float)(camera->height);

	AT_NAME::CameraSampleResult camsample;
	AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

	rays[idx] = camsample.r;

	path.throughput = aten::make_float3(1);
	path.mtrlid = -1;
	path.pdfb = 0.0f;
	path.isHit = false;
	path.isTerminate = false;

	// Accumulate value, so do not reset.
	//path.contrib = aten::make_float3(0);
}

__global__ void hitTest(
	Path* paths,
	aten::ray* rays,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vertices,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
		ctxt.matrices = matrices;
	}
	
	aten::hitrecord rec;
	float t = AT_MATH_INF;
	bool isHit = intersectBVH(&ctxt, rays[idx], AT_MATH_EPSILON, AT_MATH_INF, &rec, t);

	path.isHit = isHit;
	path.rec = rec;
}

__global__ void shadeMiss(
	hipSurfaceObject_t outSurface,
	Path* paths,
	int width, int height)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];

	if (!path.isTerminate && !path.isHit) {
		// TODO
		path.contrib = aten::make_float3(0);
		path.isTerminate = true;
	}
}

__global__ void shade(
	hipSurfaceObject_t outSurface,
	Path* paths,
	aten::ray* rays,
	ShadowRay* shadowRays,
	int width, int height,
	int depth, int rrDepth,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vertices,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
		ctxt.matrices = matrices;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& path = paths[idx];
	const auto& ray = rays[idx];

	shadowRays[idx].isActive = false;

	if (!path.isHit) {
		return;
	}
	if (path.isTerminate) {
		return;
	}

	aten::MaterialParameter* mtrl = &ctxt.mtrls[path.rec.mtrlid];
	aten::MaterialParameter* prevMtrl = (path.mtrlid >= 0 ? &ctxt.mtrls[path.mtrlid] : nullptr);

	// �����ʒu�̖@��.
	// ���̂���̃��C�̓��o���l��.
	const aten::vec3 orienting_normal = dot(path.rec.normal, ray.dir) < 0.0 ? path.rec.normal : -path.rec.normal;

	// TODO
	// Apply normal map.

	// Implicit conection to light.
	if (mtrl->attrib.isEmissive) {
		if (depth == 0) {
			// Ray hits the light directly.
			path.contrib = mtrl->baseColor;
			path.isTerminate = true;
			return;
		}
		else if (prevMtrl && prevMtrl->attrib.isSingular) {
			auto emit = prevMtrl->baseColor;
			path.contrib += path.throughput * emit;
			path.isTerminate = true;
			return;
		}
		else {
			auto cosLight = dot(orienting_normal, -ray.dir);
			auto dist2 = (path.rec.p - ray.org).squared_length();

			if (cosLight >= 0) {
				auto pdfLight = 1 / path.rec.area;

				// Convert pdf area to sradian.
				// http://www.slideshare.net/h013/edubpt-v100
				// p31 - p35
				pdfLight = pdfLight * dist2 / cosLight;

				auto misW = path.pdfb / (pdfLight + path.pdfb);

				auto emit = mtrl->baseColor;

				path.contrib += path.throughput * misW * emit;

				// When ray hit the light, tracing will finish.
				path.isTerminate = true;
				return;
			}
		}
	}

	// Explicit conection to light.
	if (!mtrl->attrib.isSingular)
	{
		real lightSelectPdf = 1;
		aten::LightSampleResult sampleres;

		// TODO
		int lightidx = aten::cmpMin<int>(path.sampler.nextSample() * lightnum, lightnum - 1);
		lightSelectPdf = 1.0f / lightnum;

		auto light = ctxt.lights[lightidx];
		if (light.object.idx >= 0) {
			light.object.ptr = &ctxt.shapes[light.object.idx];
		}

		sampleLight(&sampleres, &ctxt, &light, path.rec.p, &path.sampler);

		const auto& posLight = sampleres.pos;
		const auto& nmlLight = sampleres.nml;
		real pdfLight = sampleres.pdf;

		auto lightobj = sampleres.obj;

		auto dirToLight = normalize(sampleres.dir);

		auto cosShadow = dot(orienting_normal, dirToLight);

		real pdfb = samplePDF(mtrl, orienting_normal, ray.dir, dirToLight, path.rec.u, path.rec.v);
		auto bsdf = sampleBSDF(mtrl, orienting_normal, ray.dir, dirToLight, path.rec.u, path.rec.v);

		bsdf *= path.throughput;

		// Get light color.
		auto emit = sampleres.finalColor;

		shadowRays[idx].org = path.rec.p;
		shadowRays[idx].dir = dirToLight;
		shadowRays[idx].lightcontrib = aten::make_float3(0);
		shadowRays[idx].distToLight = sampleres.dir.length();
		shadowRays[idx].targetLightId = lightidx;

		if (light.attrib.isSingular || light.attrib.isInfinite) {
			if (pdfLight > real(0)) {
				// TODO
				// �W�I���g���^�[���̈����ɂ���.
				// singular light �̏ꍇ�́AfinalColor �ɋ����̏��Z���܂܂�Ă���.
				// inifinite light �̏ꍇ�́A���������ɂȂ�ApdfLight�Ɋ܂܂�鋗�������Ƒł����������H.
				// �i�ł����������̂ŁApdfLight�ɂ͋��������͊܂�ł��Ȃ��j.
				auto misW = pdfLight / (pdfb + pdfLight);
				shadowRays[idx].lightcontrib = (misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf;
				shadowRays[idx].isActive = true;
			}
		}
		else {
			auto cosLight = dot(nmlLight, -dirToLight);

			if (cosShadow >= 0 && cosLight >= 0) {
				auto dist2 = sampleres.dir.squared_length();
				auto G = cosShadow * cosLight / dist2;

				if (pdfb > real(0) && pdfLight > real(0)) {
					// Convert pdf from steradian to area.
					// http://www.slideshare.net/h013/edubpt-v100
					// p31 - p35
					pdfb = pdfb * cosLight / dist2;

					auto misW = pdfLight / (pdfb + pdfLight);

					shadowRays[idx].lightcontrib = (misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf;
					shadowRays[idx].isActive = true;
				}
			}
		}
	}

	real russianProb = real(1);

	if (depth > rrDepth) {
		auto t = normalize(path.throughput);
		auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

		russianProb = path.sampler.nextSample();

		if (russianProb >= p) {
			path.contrib = aten::make_float3(0);
			path.isTerminate = true;
		}
		else {
			russianProb = p;
		}
	}
			
	AT_NAME::MaterialSampling sampling;

	sampleMaterial(
		&sampling,
		mtrl,
		orienting_normal,
		ray.dir,
		path.rec.normal,
		&path.sampler,
		path.rec.u, path.rec.v);

	auto nextDir = normalize(sampling.dir);
	auto pdfb = sampling.pdf;
	auto bsdf = sampling.bsdf;

	real c = 1;
	if (!mtrl->attrib.isSingular) {
		// TODO
		// AMD�̂�abs���Ă��邪....
		//c = aten::abs(dot(orienting_normal, nextDir));
		c = dot(orienting_normal, nextDir);
	}

	if (pdfb > 0 && c > 0) {
		path.throughput *= bsdf * c / pdfb;
		path.throughput /= russianProb;
	}
	else {
		path.isTerminate = true;
	}

	// Make next ray.
	rays[idx] = aten::ray(path.rec.p, nextDir);

	path.mtrlid = path.rec.mtrlid;
	path.pdfb = pdfb;
}

__global__ void hitShadowRay(
	Path* paths,
	ShadowRay* shadowRays,
	int width, int height,
	aten::ShapeParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vertices,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vertices = vertices;
		ctxt.matrices = matrices;
	}

	const auto idx = getIdx(ix, iy, width);

	auto& shadowRay = shadowRays[idx];

	if (shadowRay.isActive) {
		auto& path = paths[idx];

		aten::hitrecord rec;
		float t = AT_MATH_INF;
		bool isHit = intersectBVH(&ctxt, shadowRay, AT_MATH_EPSILON, AT_MATH_INF, &rec, t);

		auto light = ctxt.lights[shadowRay.targetLightId];
		if (light.object.idx >= 0) {
			light.object.ptr = &ctxt.shapes[light.object.idx];
		}

		real distHitObjToRayOrg = (rec.p - shadowRay.org).length();

		auto obj = &ctxt.shapes[rec.objid];
		
		shadowRay.isActive = AT_NAME::scene::hitLight(
			isHit, 
			light.attrib,
			light.object.ptr,
			shadowRay.distToLight,
			distHitObjToRayOrg,
			t,
			obj);

		if (shadowRay.isActive) {
			path.contrib += shadowRay.lightcontrib;
		}
	}
}

__global__ void gather(
	hipSurfaceObject_t outSurface,
	Path* paths,
	int width, int height,
	int sample)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = getIdx(ix, iy, width);

	const auto& path = paths[idx];

	float4 data;
	surf2Dread(&data, outSurface, ix * sizeof(float4), iy);

	// First data.w value is 0.
	int n = data.w;
	data = n * data + make_float4(path.contrib.x, path.contrib.y, path.contrib.z, 0) / sample;
	data /= (n + 1);
	data.w = n + 1;

	surf2Dwrite(
		data,
		outSurface,
		ix * sizeof(float4), iy,
		hipBoundaryModeTrap);
}

namespace idaten {
	void PathTracing::prepare()
	{
		addFuncs();
	}

#include "misc/timer.h"
	aten::SystemTime getSystemTime()
	{
		SYSTEMTIME time;
		::GetSystemTime(&time);

		aten::SystemTime ret;
		ret.year = time.wYear;
		ret.month = time.wMonth;
		ret.dayOfWeek = time.wDayOfWeek;
		ret.day = time.wDay;
		ret.hour = time.wHour;
		ret.minute = time.wMinute;
		ret.second = time.wSecond;
		ret.milliSeconds = time.wMilliseconds;

		return std::move(ret);
	}

	static bool doneSetStackSize = false;

	void PathTracing::render(
		aten::vec4* image,
		int width, int height)
	{
#ifdef __AT_DEBUG__
		if (!doneSetStackSize) {
			size_t val = 0;
			cudaThreadGetLimit(&val, hipLimitStackSize);
			cudaThreadSetLimit(hipLimitStackSize, val * 2);
			doneSetStackSize = true;
		}
#endif

		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int depth = 0;

		idaten::TypedCudaMemory<Path> paths;
		paths.init(width * height);

		idaten::TypedCudaMemory<aten::ray> rays;
		rays.init(width * height);

		idaten::TypedCudaMemory<ShadowRay> shadowRays;
		shadowRays.init(width * height);

		CudaGLResourceMap rscmap(&glimg);
		auto outputSurf = glimg.bind();

		auto vtxTex = vtxparams.bind();

		std::vector<hipTextureObject_t> tmp;
		for (int i = 0; i < nodeparam.size(); i++) {
			auto nodeTex = nodeparam[i].bind();
			tmp.push_back(nodeTex);
		}
		nodetex.writeByNum(&tmp[0], tmp.size());

		static const int maxSamples = 1;
		static const int maxDepth = 5;
		static const int rrDepth = 3;

		auto time = getSystemTime();

		for (int i = 0; i < maxSamples; i++) {
#if 1
			genPath << <grid, block >> > (
			//genPath << <1, 1 >> > (
				paths.ptr(),
				rays.ptr(),
				width, height,
				i, maxSamples,
				time.milliSeconds,
				cam.ptr());

			depth = 0;

			while (depth < maxDepth) {
				hitTest << <grid, block >> > (
				//hitTest << <1, 1 >> > (
					paths.ptr(),
					rays.ptr(),
					width, height,
					shapeparam.ptr(), shapeparam.num(),
					mtrlparam.ptr(),
					lightparam.ptr(), lightparam.num(),
					nodetex.ptr(),
					primparams.ptr(),
					vtxTex,
					mtxparams.ptr());

				auto err = hipGetLastError();
				if (err != hipSuccess) {
					AT_PRINTF("Cuda Kernel Err(hitTest) [%s]\n", hipGetErrorString(err));
				}

				shadeMiss << <grid, block >> > (
				//shadeMiss << <1, 1 >> > (
					outputSurf,
					paths.ptr(),
					width, height);

				shade << <grid, block >> > (
				//shade << <1, 1 >> > (
					outputSurf,
					paths.ptr(),
					rays.ptr(),
					shadowRays.ptr(),
					width, height,
					depth, rrDepth,
					shapeparam.ptr(), shapeparam.num(),
					mtrlparam.ptr(),
					lightparam.ptr(), lightparam.num(),
					nodetex.ptr(),
					primparams.ptr(),
					vtxTex,
					mtxparams.ptr());

				err = hipGetLastError();
				if (err != hipSuccess) {
					AT_PRINTF("Cuda Kernel Err(shade) [%s]\n", hipGetErrorString(err));
				}

				hitShadowRay << <grid, block >> > (
				//hitShadowRay << <1, 1 >> > (
					paths.ptr(),
					shadowRays.ptr(),
					width, height,
					shapeparam.ptr(), shapeparam.num(),
					mtrlparam.ptr(),
					lightparam.ptr(), lightparam.num(),
					nodetex.ptr(),
					primparams.ptr(),
					vtxTex,
					mtxparams.ptr());

				err = hipGetLastError();
				if (err != hipSuccess) {
					AT_PRINTF("Cuda Kernel Err(hitShadowRay) [%s]\n", hipGetErrorString(err));
				}

				depth++;
			}
#endif
		}

		gather << <grid, block >> > (
		//gather << <1, 1 >> > (
			outputSurf,
			paths.ptr(),
			width, height,
			maxSamples);

		vtxparams.unbind();
		for (int i = 0; i < nodeparam.size(); i++) {
			nodeparam[i].unbind();
		}
		nodetex.reset();

		//dst.read(image, sizeof(aten::vec4) * width * height);
	}
}
